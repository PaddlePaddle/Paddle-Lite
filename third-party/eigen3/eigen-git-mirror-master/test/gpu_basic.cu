#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2015-2016 Gael Guennebaud <gael.guennebaud@inria.fr>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

// workaround issue between gcc >= 4.7 and cuda 5.5
#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
  #undef _GLIBCXX_ATOMIC_BUILTINS
  #undef _GLIBCXX_USE_INT128
#endif

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int

#include "main.h"
#include "gpu_common.h"

// Check that dense modules can be properly parsed by nvcc
#include <Eigen/Dense>

// struct Foo{
//   EIGEN_DEVICE_FUNC
//   void operator()(int i, const float* mats, float* vecs) const {
//     using namespace Eigen;
//   //   Matrix3f M(data);
//   //   Vector3f x(data+9);
//   //   Map<Vector3f>(data+9) = M.inverse() * x;
//     Matrix3f M(mats+i/16);
//     Vector3f x(vecs+i*3);
//   //   using std::min;
//   //   using std::sqrt;
//     Map<Vector3f>(vecs+i*3) << x.minCoeff(), 1, 2;// / x.dot(x);//(M.inverse() *  x) / x.x();
//     //x = x*2 + x.y() * x + x * x.maxCoeff() - x / x.sum();
//   }
// };

template<typename T>
struct coeff_wise {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T x1(in+i);
    T x2(in+i+1);
    T x3(in+i+2);
    Map<T> res(out+i*T::MaxSizeAtCompileTime);
    
    res.array() += (in[0] * x1 + x2).array() * x3.array();
  }
};

template<typename T>
struct replicate {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T x1(in+i);
    int step   = x1.size() * 4;
    int stride = 3 * step;
    
    typedef Map<Array<typename T::Scalar,Dynamic,Dynamic> > MapType;
    MapType(out+i*stride+0*step, x1.rows()*2, x1.cols()*2) = x1.replicate(2,2);
    MapType(out+i*stride+1*step, x1.rows()*3, x1.cols()) = in[i] * x1.colwise().replicate(3);
    MapType(out+i*stride+2*step, x1.rows(), x1.cols()*3) = in[i] * x1.rowwise().replicate(3);
  }
};

template<typename T>
struct redux {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    int N = 10;
    T x1(in+i);
    out[i*N+0] = x1.minCoeff();
    out[i*N+1] = x1.maxCoeff();
    out[i*N+2] = x1.sum();
    out[i*N+3] = x1.prod();
    out[i*N+4] = x1.matrix().squaredNorm();
    out[i*N+5] = x1.matrix().norm();
    out[i*N+6] = x1.colwise().sum().maxCoeff();
    out[i*N+7] = x1.rowwise().maxCoeff().sum();
    out[i*N+8] = x1.matrix().colwise().squaredNorm().sum();
  }
};

template<typename T1, typename T2>
struct prod_test {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T1::Scalar* in, typename T1::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T1::Scalar, T1::RowsAtCompileTime, T2::ColsAtCompileTime> T3;
    T1 x1(in+i);
    T2 x2(in+i+1);
    Map<T3> res(out+i*T3::MaxSizeAtCompileTime);
    res += in[i] * x1 * x2;
  }
};

template<typename T1, typename T2>
struct diagonal {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T1::Scalar* in, typename T1::Scalar* out) const
  {
    using namespace Eigen;
    T1 x1(in+i);
    Map<T2> res(out+i*T2::MaxSizeAtCompileTime);
    res += x1.diagonal();
  }
};

template<typename T>
struct eigenvalues_direct {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T::Scalar, T::RowsAtCompileTime, 1> Vec;
    T M(in+i);
    Map<Vec> res(out+i*Vec::MaxSizeAtCompileTime);
    T A = M*M.adjoint();
    SelfAdjointEigenSolver<T> eig;
    eig.computeDirect(A);
    res = eig.eigenvalues();
  }
};

template<typename T>
struct eigenvalues {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T::Scalar, T::RowsAtCompileTime, 1> Vec;
    T M(in+i);
    Map<Vec> res(out+i*Vec::MaxSizeAtCompileTime);
    T A = M*M.adjoint();
    SelfAdjointEigenSolver<T> eig;
    eig.compute(A);
    res = eig.eigenvalues();
  }
};

template<typename T>
struct matrix_inverse {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T M(in+i);
    Map<T> res(out+i*T::MaxSizeAtCompileTime);
    res = M.inverse();
  }
};

EIGEN_DECLARE_TEST(gpu_basic)
{
  ei_test_init_gpu();
  
  int nthreads = 100;
  Eigen::VectorXf in, out;
  
  #if !defined(__CUDA_ARCH__) && !defined(__HIP_DEVICE_COMPILE__)
  int data_size = nthreads * 512;
  in.setRandom(data_size);
  out.setRandom(data_size);
  #endif
  
  CALL_SUBTEST( run_and_compare_to_gpu(coeff_wise<Vector3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(coeff_wise<Array44f>(), nthreads, in, out) );

#if !defined(EIGEN_USE_HIP)
  // FIXME
  // These subtests result in a compile failure on the HIP platform
  //
  //  eigen-upstream/Eigen/src/Core/Replicate.h:61:65: error:
  //           base class 'internal::dense_xpr_base<Replicate<Array<float, 4, 1, 0, 4, 1>, -1, -1> >::type'
  //           (aka 'ArrayBase<Eigen::Replicate<Eigen::Array<float, 4, 1, 0, 4, 1>, -1, -1> >') has protected default constructor
  CALL_SUBTEST( run_and_compare_to_gpu(replicate<Array4f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(replicate<Array33f>(), nthreads, in, out) );
#endif
  
  CALL_SUBTEST( run_and_compare_to_gpu(redux<Array4f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(redux<Matrix3f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(prod_test<Matrix3f,Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(prod_test<Matrix4f,Vector4f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(diagonal<Matrix3f,Vector3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(diagonal<Matrix4f,Vector4f>(), nthreads, in, out) );

  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix2f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix4f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues_direct<Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues_direct<Matrix2f>(), nthreads, in, out) );

#if defined(__NVCC__)
  // FIXME
  // These subtests compiles only with nvcc and fail with HIPCC and clang-cuda
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues<Matrix4f>(), nthreads, in, out) );
  typedef Matrix<float,6,6> Matrix6f;
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues<Matrix6f>(), nthreads, in, out) );
#endif
}
