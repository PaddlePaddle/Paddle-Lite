#define EIGEN_USE_GPU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "tensor_benchmarks.h"

// Simple functions
#define BM_FuncGPU(FUNC)                                                       \
  static void BM_##FUNC(int iters, int N) {                                    \
    StopBenchmarkTiming();                                                     \
    Eigen::CudaStreamDevice stream;                                            \
    Eigen::GpuDevice device(&stream);                                          \
    BenchmarkSuite<Eigen::GpuDevice, float> suite(device, N);                  \
    hipDeviceSynchronize();                                                   \
    suite.FUNC(iters);                                                         \
  }                                                                            \
  BENCHMARK_RANGE(BM_##FUNC, 10, 5000);

BM_FuncGPU(memcpy);
BM_FuncGPU(typeCasting);
BM_FuncGPU(random);
BM_FuncGPU(slicing);
BM_FuncGPU(rowChip);
BM_FuncGPU(colChip);
BM_FuncGPU(shuffling);
BM_FuncGPU(padding);
BM_FuncGPU(striding);
BM_FuncGPU(broadcasting);
BM_FuncGPU(coeffWiseOp);
BM_FuncGPU(algebraicFunc);
BM_FuncGPU(transcendentalFunc);
BM_FuncGPU(rowReduction);
BM_FuncGPU(colReduction);
BM_FuncGPU(fullReduction);


// Contractions
#define BM_FuncWithInputDimsGPU(FUNC, D1, D2, D3)                              \
  static void BM_##FUNC##_##D1##x##D2##x##D3(int iters, int N) {               \
    StopBenchmarkTiming();                                                     \
    Eigen::CudaStreamDevice stream;                                            \
    Eigen::GpuDevice device(&stream);                                          \
    BenchmarkSuite<Eigen::GpuDevice, float> suite(device, D1, D2, D3);         \
    hipDeviceSynchronize();                                                   \
    suite.FUNC(iters);                                                         \
  }                                                                            \
  BENCHMARK_RANGE(BM_##FUNC##_##D1##x##D2##x##D3, 10, 5000);


BM_FuncWithInputDimsGPU(contraction, N, N, N);
BM_FuncWithInputDimsGPU(contraction, 64, N, N);
BM_FuncWithInputDimsGPU(contraction, N, 64, N);
BM_FuncWithInputDimsGPU(contraction, N, N, 64);


// Convolutions
#define BM_FuncWithKernelDimsGPU(FUNC, DIM1, DIM2)                             \
  static void BM_##FUNC##_##DIM1##x##DIM2(int iters, int N) {                  \
    StopBenchmarkTiming();                                                     \
    Eigen::CudaStreamDevice stream;                                            \
    Eigen::GpuDevice device(&stream);                                          \
    BenchmarkSuite<Eigen::GpuDevice, float> suite(device, N);                  \
    hipDeviceSynchronize();                                                   \
    suite.FUNC(iters, DIM1, DIM2);                                             \
  }                                                                            \
  BENCHMARK_RANGE(BM_##FUNC##_##DIM1##x##DIM2, 128, 5000);

BM_FuncWithKernelDimsGPU(convolution, 7, 1);
BM_FuncWithKernelDimsGPU(convolution, 1, 7);
BM_FuncWithKernelDimsGPU(convolution, 7, 4);
BM_FuncWithKernelDimsGPU(convolution, 4, 7);
BM_FuncWithKernelDimsGPU(convolution, 7, 64);
BM_FuncWithKernelDimsGPU(convolution, 64, 7);
