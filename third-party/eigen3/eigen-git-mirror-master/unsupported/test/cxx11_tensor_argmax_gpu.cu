#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.


#define EIGEN_TEST_NO_LONGDOUBLE

#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <unsupported/Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

using Eigen::Tensor;

template <int Layout>
void test_gpu_simple_argmax()
{
  Tensor<double, 3, Layout> in(Eigen::array<DenseIndex, 3>(72,53,97));
  Tensor<DenseIndex, 1, Layout> out_max(Eigen::array<DenseIndex, 1>(1));
  Tensor<DenseIndex, 1, Layout> out_min(Eigen::array<DenseIndex, 1>(1));
  in.setRandom();
  in *= in.constant(100.0);
  in(0, 0, 0) = -1000.0;
  in(71, 52, 96) = 1000.0;

  std::size_t in_bytes = in.size() * sizeof(double);
  std::size_t out_bytes = out_max.size() * sizeof(DenseIndex);

  double* d_in;
  DenseIndex* d_out_max;
  DenseIndex* d_out_min;
  gpuMalloc((void**)(&d_in), in_bytes);
  gpuMalloc((void**)(&d_out_max), out_bytes);
  gpuMalloc((void**)(&d_out_min), out_bytes);

  gpuMemcpy(d_in, in.data(), in_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<double, 3, Layout>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, Layout>, Aligned > gpu_out_max(d_out_max, Eigen::array<DenseIndex, 1>(1));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, Layout>, Aligned > gpu_out_min(d_out_min, Eigen::array<DenseIndex, 1>(1));

  gpu_out_max.device(gpu_device) = gpu_in.argmax();
  gpu_out_min.device(gpu_device) = gpu_in.argmin();

  assert(gpuMemcpyAsync(out_max.data(), d_out_max, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuMemcpyAsync(out_min.data(), d_out_min, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  VERIFY_IS_EQUAL(out_max(Eigen::array<DenseIndex, 1>(0)), 72*53*97 - 1);
  VERIFY_IS_EQUAL(out_min(Eigen::array<DenseIndex, 1>(0)), 0);

  gpuFree(d_in);
  gpuFree(d_out_max);
  gpuFree(d_out_min);
}

template <int DataLayout>
void test_gpu_argmax_dim()
{
  Tensor<float, 4, DataLayout> tensor(2,3,5,7);
  std::vector<int> dims;
  dims.push_back(2); dims.push_back(3); dims.push_back(5); dims.push_back(7);

  for (int dim = 0; dim < 4; ++dim) {
    tensor.setRandom();
    tensor = (tensor + tensor.constant(0.5)).log();

    array<DenseIndex, 3> out_shape;
    for (int d = 0; d < 3; ++d) out_shape[d] = (d < dim) ? dims[d] : dims[d+1];

    Tensor<DenseIndex, 3, DataLayout> tensor_arg(out_shape);

    array<DenseIndex, 4> ix;
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != 0) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 0, k, l) = 10.0
            tensor(ix) = 10.0;
          }
        }
      }
    }

    std::size_t in_bytes = tensor.size() * sizeof(float);
    std::size_t out_bytes = tensor_arg.size() * sizeof(DenseIndex);

    float* d_in;
    DenseIndex* d_out;
    gpuMalloc((void**)(&d_in), in_bytes);
    gpuMalloc((void**)(&d_out), out_bytes);

    gpuMemcpy(d_in, tensor.data(), in_bytes, gpuMemcpyHostToDevice);

    Eigen::GpuStreamDevice stream;
    Eigen::GpuDevice gpu_device(&stream);

    Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 4>(2, 3, 5, 7));
    Eigen::TensorMap<Eigen::Tensor<DenseIndex, 3, DataLayout>, Aligned > gpu_out(d_out, out_shape);

    gpu_out.device(gpu_device) = gpu_in.argmax(dim);

    assert(gpuMemcpyAsync(tensor_arg.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
    assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

    VERIFY_IS_EQUAL(tensor_arg.size(),
                    size_t(2*3*5*7 / tensor.dimension(dim)));

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the first index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], 0);
    }

    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != tensor.dimension(dim) - 1) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 2, k, l) = 20.0
            tensor(ix) = 20.0;
          }
        }
      }
    }

    gpuMemcpy(d_in, tensor.data(), in_bytes, gpuMemcpyHostToDevice);

    gpu_out.device(gpu_device) = gpu_in.argmax(dim);

    assert(gpuMemcpyAsync(tensor_arg.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
    assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the last index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], tensor.dimension(dim) - 1);
    }

    gpuFree(d_in);
    gpuFree(d_out);
  }
}

template <int DataLayout>
void test_gpu_argmin_dim()
{
  Tensor<float, 4, DataLayout> tensor(2,3,5,7);
  std::vector<int> dims;
  dims.push_back(2); dims.push_back(3); dims.push_back(5); dims.push_back(7);

  for (int dim = 0; dim < 4; ++dim) {
    tensor.setRandom();
    tensor = (tensor + tensor.constant(0.5)).log();

    array<DenseIndex, 3> out_shape;
    for (int d = 0; d < 3; ++d) out_shape[d] = (d < dim) ? dims[d] : dims[d+1];

    Tensor<DenseIndex, 3, DataLayout> tensor_arg(out_shape);

    array<DenseIndex, 4> ix;
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != 0) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 0, k, l) = 10.0
            tensor(ix) = -10.0;
          }
        }
      }
    }

    std::size_t in_bytes = tensor.size() * sizeof(float);
    std::size_t out_bytes = tensor_arg.size() * sizeof(DenseIndex);

    float* d_in;
    DenseIndex* d_out;
    gpuMalloc((void**)(&d_in), in_bytes);
    gpuMalloc((void**)(&d_out), out_bytes);

    gpuMemcpy(d_in, tensor.data(), in_bytes, gpuMemcpyHostToDevice);

    Eigen::GpuStreamDevice stream;
    Eigen::GpuDevice gpu_device(&stream);

    Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 4>(2, 3, 5, 7));
    Eigen::TensorMap<Eigen::Tensor<DenseIndex, 3, DataLayout>, Aligned > gpu_out(d_out, out_shape);

    gpu_out.device(gpu_device) = gpu_in.argmin(dim);

    assert(gpuMemcpyAsync(tensor_arg.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
    assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

    VERIFY_IS_EQUAL(tensor_arg.size(),
                    2*3*5*7 / tensor.dimension(dim));

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect min to be in the first index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], 0);
    }

    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != tensor.dimension(dim) - 1) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 2, k, l) = 20.0
            tensor(ix) = -20.0;
          }
        }
      }
    }

    gpuMemcpy(d_in, tensor.data(), in_bytes, gpuMemcpyHostToDevice);

    gpu_out.device(gpu_device) = gpu_in.argmin(dim);

    assert(gpuMemcpyAsync(tensor_arg.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
    assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the last index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], tensor.dimension(dim) - 1);
    }

    gpuFree(d_in);
    gpuFree(d_out);
  }
}

EIGEN_DECLARE_TEST(cxx11_tensor_argmax_gpu)
{
  CALL_SUBTEST_1(test_gpu_simple_argmax<RowMajor>());
  CALL_SUBTEST_1(test_gpu_simple_argmax<ColMajor>());
  CALL_SUBTEST_2(test_gpu_argmax_dim<RowMajor>());
  CALL_SUBTEST_2(test_gpu_argmax_dim<ColMajor>());
  CALL_SUBTEST_3(test_gpu_argmin_dim<RowMajor>());
  CALL_SUBTEST_3(test_gpu_argmin_dim<ColMajor>());
}
