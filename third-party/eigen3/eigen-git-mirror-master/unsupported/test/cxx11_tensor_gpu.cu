#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <unsupported/Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

#define EIGEN_GPU_TEST_C99_MATH  EIGEN_HAS_CXX11

using Eigen::Tensor;

void test_gpu_nullary() {
  Tensor<float, 1, 0, int> in1(2);
  Tensor<float, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t tensor_bytes = in1.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  gpuMalloc((void**)(&d_in1), tensor_bytes);
  gpuMalloc((void**)(&d_in2), tensor_bytes);
  gpuMemcpy(d_in1, in1.data(), tensor_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), tensor_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(3.14f);
  gpu_in2.device(gpu_device) = gpu_in2.random();

  Tensor<float, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(gpuMemcpyAsync(new1.data(), d_in1, tensor_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuMemcpyAsync(new2.data(), d_in2, tensor_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);

  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), 3.14f);
    VERIFY_IS_NOT_EQUAL(new2(i), in2(i));
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
}

void test_gpu_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> in2(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> out(Eigen::array<Eigen::DenseIndex, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_in2), in2_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), in2_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
      d_in2, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
      d_out, Eigen::array<Eigen::DenseIndex, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<Eigen::DenseIndex, 1>(i)),
        in1(Eigen::array<Eigen::DenseIndex, 1>(i)) + in2(Eigen::array<Eigen::DenseIndex, 1>(i)));
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
  gpuFree(d_out);
}

void test_gpu_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in3(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_in3;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_in2), in2_bytes);
  gpuMalloc((void**)(&d_in3), in3_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), in2_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in3, in3.data(), in3_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2 * gpu_in3;

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        VERIFY_IS_APPROX(out(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)), in1(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) + in2(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) * in3(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)));
      }
    }
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
  gpuFree(d_in3);
  gpuFree(d_out);
}

void test_gpu_props() {
  Tensor<float, 1> in1(200);
  Tensor<bool, 1> out(200);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(bool);

  float* d_in1;
  bool* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, 200);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_out(
      d_out, 200);

  gpu_out.device(gpu_device) = (gpu_in1.isnan)();

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 200; ++i) {
    VERIFY_IS_EQUAL(out(i), (std::isnan)(in1(i)));
  }

  gpuFree(d_in1);
  gpuFree(d_out);
}

void test_gpu_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<Eigen::DenseIndex, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.maximum(reduction_axis);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float expected = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          expected =
              std::max<float>(expected, in1(i, k, j, l));
        }
      }
      VERIFY_IS_APPROX(out(i,j), expected);
    }
  }

  gpuFree(d_in1);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_contraction()
{
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 4, DataLayout> t_left(6, 50, 3, 31);
  Tensor<float, 5, DataLayout> t_right(Eigen::array<Eigen::DenseIndex, 5>(3, 31, 7, 20, 1));
  Tensor<float, 5, DataLayout> t_result(Eigen::array<Eigen::DenseIndex, 5>(6, 50, 7, 20, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  gpuMalloc((void**)(&d_t_left), t_left_bytes);
  gpuMalloc((void**)(&d_t_right), t_right_bytes);
  gpuMalloc((void**)(&d_t_result), t_result_bytes);

  gpuMemcpy(d_t_left, t_left.data(), t_left_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_t_right, t_right.data(), t_right_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_t_left(d_t_left, 6, 50, 3, 31);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_right(d_t_right, 3, 31, 7, 20, 1);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_result(d_t_result, 6, 50, 7, 20, 1);

  typedef Eigen::Map<Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> > MapXf;
  MapXf m_left(t_left.data(), 300, 93);
  MapXf m_right(t_right.data(), 93, 140);
  Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> m_result(300, 140);

  typedef Tensor<float, 1>::DimensionPair DimPair;
  Eigen::array<DimPair, 2> dims;
  dims[0] = DimPair(2, 0);
  dims[1] = DimPair(3, 1);

  m_result = m_left * m_right;
  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);

  gpuMemcpy(t_result.data(), d_t_result, t_result_bytes, gpuMemcpyDeviceToHost);

  for (DenseIndex i = 0; i < t_result.size(); i++) {
    if (fabs(t_result.data()[i] - m_result.data()[i]) >= 1e-4f) {
      std::cout << "mismatch detected at index " << i << ": " << t_result.data()[i] << " vs " <<  m_result.data()[i] << std::endl;
      assert(false);
    }
  }

  gpuFree(d_t_left);
  gpuFree(d_t_right);
  gpuFree(d_t_result);
}

template<int DataLayout>
void test_gpu_convolution_1d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 1, DataLayout> kernel(4);
  Tensor<float, 4, DataLayout> out(74,34,11,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input, 74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 1, DataLayout> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out, 74,34,11,137);

  Eigen::array<Eigen::DenseIndex, 1> dims(1);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 34; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k,l) * kernel(0) + input(i,j+1,k,l) * kernel(1) +
                                 input(i,j+2,k,l) * kernel(2) + input(i,j+3,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

void test_gpu_convolution_inner_dim_col_major_1d()
{
  Tensor<float, 4, ColMajor> input(74,9,11,7);
  Tensor<float, 1, ColMajor> kernel(4);
  Tensor<float, 4, ColMajor> out(71,9,11,7);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_input(d_input,74,9,11,7);
  Eigen::TensorMap<Eigen::Tensor<float, 1, ColMajor> > gpu_kernel(d_kernel,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_out(d_out,71,9,11,7);

  Eigen::array<Eigen::DenseIndex, 1> dims(0);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 71; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 7; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i+0,j,k,l) * kernel(0) + input(i+1,j,k,l) * kernel(1) +
                                 input(i+2,j,k,l) * kernel(2) + input(i+3,j,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

void test_gpu_convolution_inner_dim_row_major_1d()
{
  Tensor<float, 4, RowMajor> input(7,9,11,74);
  Tensor<float, 1, RowMajor> kernel(4);
  Tensor<float, 4, RowMajor> out(7,9,11,71);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_input(d_input, 7,9,11,74);
  Eigen::TensorMap<Eigen::Tensor<float, 1, RowMajor> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_out(d_out, 7,9,11,71);

  Eigen::array<Eigen::DenseIndex, 1> dims(3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 7; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 71; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j,k,l+0) * kernel(0) + input(i,j,k,l+1) * kernel(1) +
                                 input(i,j,k,l+2) * kernel(2) + input(i,j,k,l+3) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_convolution_2d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 2, DataLayout> kernel(3,4);
  Tensor<float, 4, DataLayout> out(74,35,8,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input,74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> > gpu_kernel(d_kernel,3,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out,74,35,8,137);

  Eigen::array<Eigen::DenseIndex, 2> dims(1,2);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k+0,l) * kernel(0,0) +
                                 input(i,j+1,k+0,l) * kernel(1,0) +
                                 input(i,j+2,k+0,l) * kernel(2,0) +
                                 input(i,j+0,k+1,l) * kernel(0,1) +
                                 input(i,j+1,k+1,l) * kernel(1,1) +
                                 input(i,j+2,k+1,l) * kernel(2,1) +
                                 input(i,j+0,k+2,l) * kernel(0,2) +
                                 input(i,j+1,k+2,l) * kernel(1,2) +
                                 input(i,j+2,k+2,l) * kernel(2,2) +
                                 input(i,j+0,k+3,l) * kernel(0,3) +
                                 input(i,j+1,k+3,l) * kernel(1,3) +
                                 input(i,j+2,k+3,l) * kernel(2,3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_convolution_3d()
{
  Tensor<float, 5, DataLayout> input(Eigen::array<Eigen::DenseIndex, 5>(74,37,11,137,17));
  Tensor<float, 3, DataLayout> kernel(3,4,2);
  Tensor<float, 5, DataLayout> out(Eigen::array<Eigen::DenseIndex, 5>(74,35,8,136,17));
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;    
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_input(d_input,74,37,11,137,17);
  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> > gpu_kernel(d_kernel,3,4,2);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_out(d_out,74,35,8,136,17);

  Eigen::array<Eigen::DenseIndex, 3> dims(1,2,3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 136; ++l) {
          for (int m = 0; m < 17; ++m) {
            const float result = out(i,j,k,l,m);
            const float expected = input(i,j+0,k+0,l+0,m) * kernel(0,0,0) +
                                   input(i,j+1,k+0,l+0,m) * kernel(1,0,0) +
                                   input(i,j+2,k+0,l+0,m) * kernel(2,0,0) +
                                   input(i,j+0,k+1,l+0,m) * kernel(0,1,0) +
                                   input(i,j+1,k+1,l+0,m) * kernel(1,1,0) +
                                   input(i,j+2,k+1,l+0,m) * kernel(2,1,0) +
                                   input(i,j+0,k+2,l+0,m) * kernel(0,2,0) +
                                   input(i,j+1,k+2,l+0,m) * kernel(1,2,0) +
                                   input(i,j+2,k+2,l+0,m) * kernel(2,2,0) +
                                   input(i,j+0,k+3,l+0,m) * kernel(0,3,0) +
                                   input(i,j+1,k+3,l+0,m) * kernel(1,3,0) +
                                   input(i,j+2,k+3,l+0,m) * kernel(2,3,0) +
                                   input(i,j+0,k+0,l+1,m) * kernel(0,0,1) +
                                   input(i,j+1,k+0,l+1,m) * kernel(1,0,1) +
                                   input(i,j+2,k+0,l+1,m) * kernel(2,0,1) +
                                   input(i,j+0,k+1,l+1,m) * kernel(0,1,1) +
                                   input(i,j+1,k+1,l+1,m) * kernel(1,1,1) +
                                   input(i,j+2,k+1,l+1,m) * kernel(2,1,1) +
                                   input(i,j+0,k+2,l+1,m) * kernel(0,2,1) +
                                   input(i,j+1,k+2,l+1,m) * kernel(1,2,1) +
                                   input(i,j+2,k+2,l+1,m) * kernel(2,2,1) +
                                   input(i,j+0,k+3,l+1,m) * kernel(0,3,1) +
                                   input(i,j+1,k+3,l+1,m) * kernel(1,3,1) +
                                   input(i,j+2,k+3,l+1,m) * kernel(2,3,1);
            VERIFY_IS_APPROX(result, expected);
          }
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}


#if EIGEN_GPU_TEST_C99_MATH
template <typename Scalar>
void test_gpu_lgamma(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.lgamma();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::lgamma)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}
#endif

template <typename Scalar>
void test_gpu_digamma()
{
  Tensor<Scalar, 1> in(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in(0) = Scalar(1);
  in(1) = Scalar(1.5);
  in(2) = Scalar(4);
  in(3) = Scalar(-10.5);
  in(4) = Scalar(10000.5);
  in(5) = Scalar(0);
  in(6) = Scalar(-1);

  expected_out(0) = Scalar(-0.5772156649015329);
  expected_out(1) = Scalar(0.03648997397857645);
  expected_out(2) = Scalar(1.2561176684318);
  expected_out(3) = Scalar(2.398239129535781);
  expected_out(4) = Scalar(9.210340372392849);
  expected_out(5) = std::numeric_limits<Scalar>::infinity();
  expected_out(6) = std::numeric_limits<Scalar>::infinity();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in.digamma();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 5; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }
  for (int i = 5; i < 7; ++i) {
    VERIFY_IS_EQUAL(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_zeta()
{
  Tensor<Scalar, 1> in_x(6);
  Tensor<Scalar, 1> in_q(6);
  Tensor<Scalar, 1> out(6);
  Tensor<Scalar, 1> expected_out(6);
  out.setZero();

  in_x(0) = Scalar(1);
  in_x(1) = Scalar(1.5);
  in_x(2) = Scalar(4);
  in_x(3) = Scalar(-10.5);
  in_x(4) = Scalar(10000.5);
  in_x(5) = Scalar(3);
  
  in_q(0) = Scalar(1.2345);
  in_q(1) = Scalar(2);
  in_q(2) = Scalar(1.5);
  in_q(3) = Scalar(3);
  in_q(4) = Scalar(1.0001);
  in_q(5) = Scalar(-2.5);

  expected_out(0) = std::numeric_limits<Scalar>::infinity();
  expected_out(1) = Scalar(1.61237534869);
  expected_out(2) = Scalar(0.234848505667);
  expected_out(3) = Scalar(1.03086757337e-5);
  expected_out(4) = Scalar(0.367879440865);
  expected_out(5) = Scalar(0.054102025820864097);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_q;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_q), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_q, in_q.data(), bytes, gpuMemcpyHostToDevice);
  
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_q(d_in_q, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 6);

  gpu_out.device(gpu_device) = gpu_in_x.zeta(gpu_in_q);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  VERIFY_IS_EQUAL(out(0), expected_out(0));
  VERIFY((std::isnan)(out(3)));

  for (int i = 1; i < 6; ++i) {
    if (i != 3) {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_in_q);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_polygamma()
{
  Tensor<Scalar, 1> in_x(7);
  Tensor<Scalar, 1> in_n(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in_n(0) = Scalar(1);
  in_n(1) = Scalar(1);
  in_n(2) = Scalar(1);
  in_n(3) = Scalar(17);
  in_n(4) = Scalar(31);
  in_n(5) = Scalar(28);
  in_n(6) = Scalar(8);
  
  in_x(0) = Scalar(2);
  in_x(1) = Scalar(3);
  in_x(2) = Scalar(25.5);
  in_x(3) = Scalar(4.7);
  in_x(4) = Scalar(11.8);
  in_x(5) = Scalar(17.7);
  in_x(6) = Scalar(30.2);

  expected_out(0) = Scalar(0.644934066848);
  expected_out(1) = Scalar(0.394934066848);
  expected_out(2) = Scalar(0.0399946696496);
  expected_out(3) = Scalar(293.334565435);
  expected_out(4) = Scalar(0.445487887616);
  expected_out(5) = Scalar(-2.47810300902e-07);
  expected_out(6) = Scalar(-8.29668781082e-09);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_n;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_n), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_n, in_n.data(), bytes, gpuMemcpyHostToDevice);
  
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_n(d_in_n, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in_n.polygamma(gpu_in_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 7; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in_x);
  gpuFree(d_in_n);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igamma()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igamma_s[][6] = {{0.0, nan, nan, nan, nan, nan},
                          {0.0, 0.6321205588285578, 0.7768698398515702,
                           0.9816843611112658, 9.999500016666262e-05, 1.0},
                          {0.0, 0.4275932955291202, 0.608374823728911,
                           0.9539882943107686, 7.522076445089201e-07, 1.0},
                          {0.0, 0.01898815687615381, 0.06564245437845008,
                           0.5665298796332909, 4.166333347221828e-18, 1.0},
                          {0.0, 0.9999780593618628, 0.9999899967080838,
                           0.9999996219837988, 0.9991370418689945, 1.0},
                          {0.0, 0.0, 0.0, 0.0, 0.0, 0.5042041932513908}};



  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  assert(gpuMalloc((void**)(&d_a), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_x), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_out), bytes) == gpuSuccess);

  gpuMemcpy(d_a, a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igamma(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igamma_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igamma_s[i][j]);
      }
    }
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igammac()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igammac_s[][6] = {{nan, nan, nan, nan, nan, nan},
                           {1.0, 0.36787944117144233, 0.22313016014842982,
                            0.018315638888734182, 0.9999000049998333, 0.0},
                           {1.0, 0.5724067044708798, 0.3916251762710878,
                            0.04601170568923136, 0.9999992477923555, 0.0},
                           {1.0, 0.9810118431238462, 0.9343575456215499,
                            0.4334701203667089, 1.0, 0.0},
                           {1.0, 2.1940638138146658e-05, 1.0003291916285e-05,
                            3.7801620118431334e-07, 0.0008629581310054535,
                            0.0},
                           {1.0, 1.0, 1.0, 1.0, 1.0, 0.49579580674813944}};

  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_a), bytes);
  gpuMalloc((void**)(&d_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_a, a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igammac(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igammac_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igammac_s[i][j]);
      }
    }
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

#if EIGEN_GPU_TEST_C99_MATH
template <typename Scalar>
void test_gpu_erf(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  assert(gpuMalloc((void**)(&d_in), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_out), bytes) == gpuSuccess);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erf();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erf)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_erfc(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erfc();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erfc)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}
#endif
template <typename Scalar>
void test_gpu_ndtri()
{
  Tensor<Scalar, 1> in_x(8);
  Tensor<Scalar, 1> out(8);
  Tensor<Scalar, 1> expected_out(8);
  out.setZero();

  in_x(0) = Scalar(1);
  in_x(1) = Scalar(0.);
  in_x(2) = Scalar(0.5);
  in_x(3) = Scalar(0.2);
  in_x(4) = Scalar(0.8);
  in_x(5) = Scalar(0.9);
  in_x(6) = Scalar(0.1);
  in_x(7) = Scalar(0.99);
  in_x(8) = Scalar(0.01);

  expected_out(0) = std::numeric_limits<Scalar>::infinity();
  expected_out(1) = -std::numeric_limits<Scalar>::infinity();
  expected_out(2) = Scalar(0.0);
  expected_out(3) = Scalar(-0.8416212335729142);
  expected_out(4) = Scalar(0.8416212335729142);
  expected_out(5) = Scalar(1.2815515655446004);
  expected_out(6) = Scalar(-1.2815515655446004);
  expected_out(7) = Scalar(2.3263478740408408);
  expected_out(8) = Scalar(-2.3263478740408408);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 6);

  gpu_out.device(gpu_device) = gpu_in_x.ndtri();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  VERIFY_IS_EQUAL(out(0), expected_out(0));
  VERIFY((std::isnan)(out(3)));

  for (int i = 1; i < 6; ++i) {
    if (i != 3) {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_betainc()
{
  Tensor<Scalar, 1> in_x(125);
  Tensor<Scalar, 1> in_a(125);
  Tensor<Scalar, 1> in_b(125);
  Tensor<Scalar, 1> out(125);
  Tensor<Scalar, 1> expected_out(125);
  out.setZero();

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();

  Array<Scalar, 1, Dynamic> x(125);
  Array<Scalar, 1, Dynamic> a(125);
  Array<Scalar, 1, Dynamic> b(125);
  Array<Scalar, 1, Dynamic> v(125);

  a << 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999;

  b << 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999;

  x << -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1,
      0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1,
      -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1;

  v << nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, 0.47972119876364683, 0.5, 0.5202788012363533, nan, nan,
      0.9518683957740043, 0.9789663010413743, 0.9931729188073435, nan, nan,
      0.999995949033062, 0.9999999999993698, 0.9999999999999999, nan, nan,
      0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan, nan, nan,
      nan, nan, nan, nan, 0.006827081192655869, 0.0210336989586256,
      0.04813160422599567, nan, nan, 0.20014344256217678, 0.5000000000000001,
      0.7998565574378232, nan, nan, 0.9991401428435834, 0.999999999698403,
      0.9999999999999999, nan, nan, 0.9999999999999999, 0.9999999999999999,
      0.9999999999999999, nan, nan, nan, nan, nan, nan, nan,
      1.0646600232370887e-25, 6.301722877826246e-13, 4.050966937974938e-06, nan,
      nan, 7.864342668429763e-23, 3.015969667594166e-10, 0.0008598571564165444,
      nan, nan, 6.031987710123844e-08, 0.5000000000000007, 0.9999999396801229,
      nan, nan, 0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan,
      nan, nan, nan, nan, nan, nan, 0.0, 7.029920380986636e-306,
      2.2450728208591345e-101, nan, nan, 0.0, 9.275871147869727e-302,
      1.2232913026152827e-97, nan, nan, 0.0, 3.0891393081932924e-252,
      2.9303043666183996e-60, nan, nan, 2.248913486879199e-196,
      0.5000000000004947, 0.9999999999999999, nan;

  for (int i = 0; i < 125; ++i) {
    in_x(i) = x(i);
    in_a(i) = a(i);
    in_b(i) = b(i);
    expected_out(i) = v(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_a;
  Scalar* d_in_b;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_a), bytes);
  gpuMalloc((void**)(&d_in_b), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_a, in_a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_b, in_b.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_a(d_in_a, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_b(d_in_b, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 125);

  gpu_out.device(gpu_device) = betainc(gpu_in_a, gpu_in_b, gpu_in_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 1; i < 125; ++i) {
    if ((std::isnan)(expected_out(i))) {
      VERIFY((std::isnan)(out(i)));
    } else {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_in_a);
  gpuFree(d_in_b);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_i0e()
{
  Tensor<Scalar, 1> in_x(21);
  Tensor<Scalar, 1> out(21);
  Tensor<Scalar, 1> expected_out(21);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(21);
  Array<Scalar, 1, Dynamic> expected_out_array(21);

  in_x_array << -20.0, -18.0, -16.0, -14.0, -12.0, -10.0, -8.0, -6.0, -4.0,
      -2.0, 0.0, 2.0, 4.0, 6.0, 8.0, 10.0, 12.0, 14.0, 16.0, 18.0, 20.0;

  expected_out_array << 0.0897803118848, 0.0947062952128, 0.100544127361,
      0.107615251671, 0.116426221213, 0.127833337163, 0.143431781857,
      0.16665743264, 0.207001921224, 0.308508322554, 1.0, 0.308508322554,
      0.207001921224, 0.16665743264, 0.143431781857, 0.127833337163,
      0.116426221213, 0.107615251671, 0.100544127361, 0.0947062952128,
      0.0897803118848;

  for (int i = 0; i < 21; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 21);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 21);

  gpu_out.device(gpu_device) = gpu_in.bessel_i0e();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 21; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_i1e()
{
  Tensor<Scalar, 1> in_x(21);
  Tensor<Scalar, 1> out(21);
  Tensor<Scalar, 1> expected_out(21);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(21);
  Array<Scalar, 1, Dynamic> expected_out_array(21);

  in_x_array << -20.0, -18.0, -16.0, -14.0, -12.0, -10.0, -8.0, -6.0, -4.0,
      -2.0, 0.0, 2.0, 4.0, 6.0, 8.0, 10.0, 12.0, 14.0, 16.0, 18.0, 20.0;

  expected_out_array << -0.0875062221833, -0.092036796872, -0.0973496147565,
      -0.103697667463, -0.11146429929, -0.121262681384, -0.134142493293,
      -0.152051459309, -0.178750839502, -0.215269289249, 0.0, 0.215269289249,
      0.178750839502, 0.152051459309, 0.134142493293, 0.121262681384,
      0.11146429929, 0.103697667463, 0.0973496147565, 0.092036796872,
      0.0875062221833;

  for (int i = 0; i < 21; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 21);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 21);

  gpu_out.device(gpu_device) = gpu_in.bessel_i1e();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 21; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igamma_der_a()
{
  Tensor<Scalar, 1> in_x(30);
  Tensor<Scalar, 1> in_a(30);
  Tensor<Scalar, 1> out(30);
  Tensor<Scalar, 1> expected_out(30);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_a_array(30);
  Array<Scalar, 1, Dynamic> in_x_array(30);
  Array<Scalar, 1, Dynamic> expected_out_array(30);

  // See special_functions.cpp for the Python code that generates the test data.

  in_a_array << 0.01, 0.01, 0.01, 0.01, 0.01, 0.1, 0.1, 0.1, 0.1, 0.1, 1.0, 1.0,
      1.0, 1.0, 1.0, 10.0, 10.0, 10.0, 10.0, 10.0, 100.0, 100.0, 100.0, 100.0,
      100.0, 1000.0, 1000.0, 1000.0, 1000.0, 1000.0;

  in_x_array << 1.25668890405e-26, 1.17549435082e-38, 1.20938905072e-05,
      1.17549435082e-38, 1.17549435082e-38, 5.66572070696e-16, 0.0132865061065,
      0.0200034203853, 6.29263709118e-17, 1.37160367764e-06, 0.333412038288,
      1.18135687766, 0.580629033777, 0.170631439426, 0.786686768458,
      7.63873279537, 13.1944344379, 11.896042354, 10.5830172417, 10.5020942233,
      92.8918587747, 95.003720371, 86.3715926467, 96.0330217672, 82.6389930677,
      968.702906754, 969.463546828, 1001.79726022, 955.047416547, 1044.27458568;

  expected_out_array << -32.7256441441, -36.4394150514, -9.66467612263,
      -36.4394150514, -36.4394150514, -1.0891900302, -2.66351229645,
      -2.48666868596, -0.929700494428, -3.56327722764, -0.455320135314,
      -0.391437214323, -0.491352055991, -0.350454834292, -0.471773162921,
      -0.104084440522, -0.0723646747909, -0.0992828975532, -0.121638215446,
      -0.122619605294, -0.0317670267286, -0.0359974812869, -0.0154359225363,
      -0.0375775365921, -0.00794899153653, -0.00777303219211, -0.00796085782042,
      -0.0125850719397, -0.00455500206958, -0.00476436993148;

  for (int i = 0; i < 30; ++i) {
    in_x(i) = in_x_array(i);
    in_a(i) = in_a_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_a), bytes);
  gpuMalloc((void**)(&d_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_a, in_a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_a(d_a, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_x(d_x, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 30);

  gpu_out.device(gpu_device) = gpu_a.igamma_der_a(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 30; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_gamma_sample_der_alpha()
{
  Tensor<Scalar, 1> in_alpha(30);
  Tensor<Scalar, 1> in_sample(30);
  Tensor<Scalar, 1> out(30);
  Tensor<Scalar, 1> expected_out(30);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_alpha_array(30);
  Array<Scalar, 1, Dynamic> in_sample_array(30);
  Array<Scalar, 1, Dynamic> expected_out_array(30);

  // See special_functions.cpp for the Python code that generates the test data.

  in_alpha_array << 0.01, 0.01, 0.01, 0.01, 0.01, 0.1, 0.1, 0.1, 0.1, 0.1, 1.0,
      1.0, 1.0, 1.0, 1.0, 10.0, 10.0, 10.0, 10.0, 10.0, 100.0, 100.0, 100.0,
      100.0, 100.0, 1000.0, 1000.0, 1000.0, 1000.0, 1000.0;

  in_sample_array << 1.25668890405e-26, 1.17549435082e-38, 1.20938905072e-05,
      1.17549435082e-38, 1.17549435082e-38, 5.66572070696e-16, 0.0132865061065,
      0.0200034203853, 6.29263709118e-17, 1.37160367764e-06, 0.333412038288,
      1.18135687766, 0.580629033777, 0.170631439426, 0.786686768458,
      7.63873279537, 13.1944344379, 11.896042354, 10.5830172417, 10.5020942233,
      92.8918587747, 95.003720371, 86.3715926467, 96.0330217672, 82.6389930677,
      968.702906754, 969.463546828, 1001.79726022, 955.047416547, 1044.27458568;

  expected_out_array << 7.42424742367e-23, 1.02004297287e-34, 0.0130155240738,
      1.02004297287e-34, 1.02004297287e-34, 1.96505168277e-13, 0.525575786243,
      0.713903991771, 2.32077561808e-14, 0.000179348049886, 0.635500453302,
      1.27561284917, 0.878125852156, 0.41565819538, 1.03606488534,
      0.885964824887, 1.16424049334, 1.10764479598, 1.04590810812,
      1.04193666963, 0.965193152414, 0.976217589464, 0.93008035061,
      0.98153216096, 0.909196397698, 0.98434963993, 0.984738050206,
      1.00106492525, 0.97734200649, 1.02198794179;

  for (int i = 0; i < 30; ++i) {
    in_alpha(i) = in_alpha_array(i);
    in_sample(i) = in_sample_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_alpha.size() * sizeof(Scalar);

  Scalar* d_alpha;
  Scalar* d_sample;
  Scalar* d_out;
  gpuMalloc((void**)(&d_alpha), bytes);
  gpuMalloc((void**)(&d_sample), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_alpha, in_alpha.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_sample, in_sample.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_alpha(d_alpha, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_sample(d_sample, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 30);

  gpu_out.device(gpu_device) = gpu_alpha.gamma_sample_der_alpha(gpu_sample);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 30; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_alpha);
  gpuFree(d_sample);
  gpuFree(d_out);
}

EIGEN_DECLARE_TEST(cxx11_tensor_gpu)
{
  CALL_SUBTEST_1(test_gpu_nullary());
  CALL_SUBTEST_1(test_gpu_elementwise_small());
  CALL_SUBTEST_1(test_gpu_elementwise());
  CALL_SUBTEST_1(test_gpu_props());
  CALL_SUBTEST_1(test_gpu_reduction());
  CALL_SUBTEST_2(test_gpu_contraction<ColMajor>());
  CALL_SUBTEST_2(test_gpu_contraction<RowMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_1d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_1d<RowMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_inner_dim_col_major_1d());
  CALL_SUBTEST_3(test_gpu_convolution_inner_dim_row_major_1d());
  CALL_SUBTEST_3(test_gpu_convolution_2d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_2d<RowMajor>());
#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.
// they hang..need to investigate and fix
  CALL_SUBTEST_3(test_gpu_convolution_3d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_3d<RowMajor>());
#endif

#if EIGEN_GPU_TEST_C99_MATH
  // std::erf, std::erfc, and so on where only added in c++11. We use them
  // as a golden reference to validate the results produced by Eigen. Therefore
  // we can only run these tests if we use a c++11 compiler.
  CALL_SUBTEST_4(test_gpu_lgamma<float>(1.0f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_lgamma<double>(1.0));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(100.0));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(0.01));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(0.001));

  CALL_SUBTEST_4(test_gpu_erf<float>(1.0f));
  CALL_SUBTEST_4(test_gpu_erf<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_erf<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_erf<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_erfc<float>(1.0f));
  // CALL_SUBTEST(test_gpu_erfc<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_erfc<float>(5.0f)); // GPU erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_gpu_erfc<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_erfc<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_erf<double>(1.0));
  CALL_SUBTEST_4(test_gpu_erf<double>(100.0));
  CALL_SUBTEST_4(test_gpu_erf<double>(0.01));
  CALL_SUBTEST_4(test_gpu_erf<double>(0.001));

  CALL_SUBTEST_4(test_gpu_erfc<double>(1.0));
  // CALL_SUBTEST(test_gpu_erfc<double>(100.0));
  CALL_SUBTEST_4(test_gpu_erfc<double>(5.0)); // GPU erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_gpu_erfc<double>(0.01));
  CALL_SUBTEST_4(test_gpu_erfc<double>(0.001));

#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.

  CALL_SUBTEST_5(test_gpu_ndtri<float>());
  CALL_SUBTEST_5(test_gpu_ndtri<double>());

  CALL_SUBTEST_5(test_gpu_digamma<float>());
  CALL_SUBTEST_5(test_gpu_digamma<double>());

  CALL_SUBTEST_5(test_gpu_polygamma<float>());
  CALL_SUBTEST_5(test_gpu_polygamma<double>());

  CALL_SUBTEST_5(test_gpu_zeta<float>());
  CALL_SUBTEST_5(test_gpu_zeta<double>());
#endif

  CALL_SUBTEST_5(test_gpu_igamma<float>());
  CALL_SUBTEST_5(test_gpu_igammac<float>());

  CALL_SUBTEST_5(test_gpu_igamma<double>());
  CALL_SUBTEST_5(test_gpu_igammac<double>());

#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.
  CALL_SUBTEST_6(test_gpu_betainc<float>());
  CALL_SUBTEST_6(test_gpu_betainc<double>());

  CALL_SUBTEST_6(test_gpu_i0e<float>());
  CALL_SUBTEST_6(test_gpu_i0e<double>());

  CALL_SUBTEST_6(test_gpu_i1e<float>());
  CALL_SUBTEST_6(test_gpu_i1e<double>());

  CALL_SUBTEST_6(test_gpu_i1e<float>());
  CALL_SUBTEST_6(test_gpu_i1e<double>());

  CALL_SUBTEST_6(test_gpu_igamma_der_a<float>());
  CALL_SUBTEST_6(test_gpu_igamma_der_a<double>());

  CALL_SUBTEST_6(test_gpu_gamma_sample_der_alpha<float>());
  CALL_SUBTEST_6(test_gpu_gamma_sample_der_alpha<double>());
#endif

#endif
}
