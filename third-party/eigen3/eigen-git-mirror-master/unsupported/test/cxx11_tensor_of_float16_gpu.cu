#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>


using Eigen::Tensor;

template<typename>
void test_gpu_numext() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  bool* d_res_half = (bool*)gpu_device.allocate(num_elem * sizeof(bool));
  bool* d_res_float = (bool*)gpu_device.allocate(num_elem * sizeof(bool));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_res_float.device(gpu_device) = gpu_float.unaryExpr(Eigen::internal::scalar_isnan_op<float>());
  gpu_res_half.device(gpu_device) = gpu_float.cast<Eigen::half>().unaryExpr(Eigen::internal::scalar_isnan_op<Eigen::half>());

  Tensor<bool, 1> half_prec(num_elem);
  Tensor<bool, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(bool));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(bool));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking numext " << i << std::endl;
    VERIFY_IS_EQUAL(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}


#ifdef EIGEN_HAS_GPU_FP16

template<typename>
void test_gpu_conversion() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;
  
  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  float* d_conv = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_half(
      d_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_conv(
      d_conv, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random();
  gpu_half.device(gpu_device) = gpu_float.cast<Eigen::half>();
  gpu_conv.device(gpu_device) = gpu_half.cast<float>();

  Tensor<float, 1> initial(num_elem);
  Tensor<float, 1> final(num_elem);
  gpu_device.memcpyDeviceToHost(initial.data(), d_float, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(final.data(), d_conv, num_elem*sizeof(float));

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(initial(i), final(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_half);
  gpu_device.deallocate(d_conv);
}

template<typename>
void test_gpu_unary() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_res_float.device(gpu_device) = gpu_float.abs();
  gpu_res_half.device(gpu_device) = gpu_float.cast<Eigen::half>().abs().cast<float>();

  Tensor<float, 1> half_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking unary " << i << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}

template<typename>
void test_gpu_elementwise() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float1(
      d_float1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float2(
      d_float2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  gpu_float1.device(gpu_device) = gpu_float1.random();
  gpu_float2.device(gpu_device) = gpu_float2.random();
  gpu_res_float.device(gpu_device) = (gpu_float1 + gpu_float2) * gpu_float1;
  gpu_res_half.device(gpu_device) = ((gpu_float1.cast<Eigen::half>() + gpu_float2.cast<Eigen::half>()) * gpu_float1.cast<Eigen::half>()).cast<float>();

  Tensor<float, 1> half_prec(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking elemwise " << i << ": full prec = " << full_prec(i) << " vs half prec = " << half_prec(i) << std::endl;
    VERIFY_IS_APPROX(static_cast<Eigen::half>(full_prec(i)), static_cast<Eigen::half>(half_prec(i)));
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}

template<typename>
void test_gpu_trancendental() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float3 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_res1_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res1_float = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res2_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res2_float = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res3_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res3_float = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float1(d_float1, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float2(d_float2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float3(d_float3, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res1_half(d_res1_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res1_float(d_res1_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res2_half(d_res2_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res2_float(d_res2_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res3_half(d_res3_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res3_float(d_res3_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res4_half(d_res3_half, num_elem);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res4_float(d_res3_float, num_elem);

  gpu_float1.device(gpu_device) = gpu_float1.random() - gpu_float1.constant(0.5f);
  gpu_float2.device(gpu_device) = gpu_float2.random() + gpu_float1.constant(0.5f);
  gpu_float3.device(gpu_device) = gpu_float3.random();
  gpu_res1_float.device(gpu_device) = gpu_float1.exp().cast<Eigen::half>();
  gpu_res2_float.device(gpu_device) = gpu_float2.log().cast<Eigen::half>();
  gpu_res3_float.device(gpu_device) = gpu_float3.log1p().cast<Eigen::half>();
  gpu_res4_float.device(gpu_device) = gpu_float3.expm1().cast<Eigen::half>();

  gpu_res1_half.device(gpu_device) = gpu_float1.cast<Eigen::half>();
  gpu_res1_half.device(gpu_device) = gpu_res1_half.exp();

  gpu_res2_half.device(gpu_device) = gpu_float2.cast<Eigen::half>();
  gpu_res2_half.device(gpu_device) = gpu_res2_half.log();

  gpu_res3_half.device(gpu_device) = gpu_float3.cast<Eigen::half>();
  gpu_res3_half.device(gpu_device) = gpu_res3_half.log1p();

  gpu_res3_half.device(gpu_device) = gpu_float3.cast<Eigen::half>();
  gpu_res3_half.device(gpu_device) = gpu_res3_half.expm1();

  Tensor<float, 1> input1(num_elem);
  Tensor<Eigen::half, 1> half_prec1(num_elem);
  Tensor<Eigen::half, 1> full_prec1(num_elem);
  Tensor<float, 1> input2(num_elem);
  Tensor<Eigen::half, 1> half_prec2(num_elem);
  Tensor<Eigen::half, 1> full_prec2(num_elem);
  Tensor<float, 1> input3(num_elem);
  Tensor<Eigen::half, 1> half_prec3(num_elem);
  Tensor<Eigen::half, 1> full_prec3(num_elem);
  gpu_device.memcpyDeviceToHost(input1.data(), d_float1, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(input2.data(), d_float2, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(input3.data(), d_float3, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(half_prec1.data(), d_res1_half, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec1.data(), d_res1_float, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(half_prec2.data(), d_res2_half, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec2.data(), d_res2_float, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(half_prec3.data(), d_res3_half, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec3.data(), d_res3_float, num_elem*sizeof(Eigen::half));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking elemwise exp " << i << " input = " << input1(i) << " full = " << full_prec1(i) << " half = " << half_prec1(i) << std::endl;
    VERIFY_IS_APPROX(full_prec1(i), half_prec1(i));
  }
  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking elemwise log " << i << " input = " << input2(i) << " full = " << full_prec2(i) << " half = " << half_prec2(i) << std::endl;
    if(std::abs(input2(i)-1.f)<0.05f) // log lacks accuracy nearby 1
      VERIFY_IS_APPROX(full_prec2(i)+Eigen::half(0.1f), half_prec2(i)+Eigen::half(0.1f));
    else
      VERIFY_IS_APPROX(full_prec2(i), half_prec2(i));
  }
  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking elemwise plog1 " << i << " input = " << input3(i) << " full = " << full_prec3(i) << " half = " << half_prec3(i) << std::endl;
    VERIFY_IS_APPROX(full_prec3(i), half_prec3(i));
  }
  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_float3);
  gpu_device.deallocate(d_res1_half);
  gpu_device.deallocate(d_res1_float);
  gpu_device.deallocate(d_res2_half);
  gpu_device.deallocate(d_res2_float);
  gpu_device.deallocate(d_res3_float);
  gpu_device.deallocate(d_res3_half);
}

template<typename>
void test_gpu_contractions() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int rows = 23;
  int cols = 23;
  int num_elem = rows*cols;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_res_half = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));
  Eigen::half* d_res_float = (Eigen::half*)gpu_device.allocate(num_elem * sizeof(Eigen::half));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(
      d_float1, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(
      d_float2, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 2>, Eigen::Aligned> gpu_res_half(
      d_res_half, rows, cols);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 2>, Eigen::Aligned> gpu_res_float(
      d_res_float, rows, cols);

  gpu_float1.device(gpu_device) = gpu_float1.random() - gpu_float1.constant(0.5f);
  gpu_float2.device(gpu_device) = gpu_float2.random() - gpu_float2.constant(0.5f);

  typedef Tensor<float, 2>::DimensionPair DimPair;
  Eigen::array<DimPair, 1> dims(DimPair(1, 0));
  gpu_res_float.device(gpu_device) = gpu_float1.contract(gpu_float2, dims).cast<Eigen::half>();
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().contract(gpu_float2.cast<Eigen::half>(), dims);

  Tensor<Eigen::half, 2> half_prec(rows, cols);
  Tensor<Eigen::half, 2> full_prec(rows, cols);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, num_elem*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(Eigen::half));
  gpu_device.synchronize();

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      std::cout << "Checking contract " << i << " " << j << full_prec(i, j) << " " << half_prec(i, j) << std::endl;
      if (numext::abs(full_prec(i, j) - half_prec(i, j)) > Eigen::half(1e-2f)) {
        VERIFY_IS_APPROX(full_prec(i, j), half_prec(i, j));
      }
    }
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}

template<typename>
void test_gpu_reductions(int size1, int size2, int redux) {

   std::cout << "Reducing " << size1 << " by " << size2
             << " tensor along dim " << redux << std::endl; 

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = size1*size2;
  int result_size = (redux == 1 ? size1 : size2);

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_res_half = (Eigen::half*)gpu_device.allocate(result_size * sizeof(Eigen::half));
  Eigen::half* d_res_float = (Eigen::half*)gpu_device.allocate(result_size * sizeof(Eigen::half));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(
      d_float1, size1, size2);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(
      d_float2, size1, size2);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res_half(
      d_res_half, result_size);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, result_size);

  gpu_float1.device(gpu_device) = gpu_float1.random() * 2.0f;
  gpu_float2.device(gpu_device) = gpu_float2.random() * 2.0f;

  Eigen::array<int, 1> redux_dim = {{redux}};
  gpu_res_float.device(gpu_device) = gpu_float1.sum(redux_dim).cast<Eigen::half>();
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().sum(redux_dim);

  Tensor<Eigen::half, 1> half_prec(result_size);
  Tensor<Eigen::half, 1> full_prec(result_size);
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, result_size*sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, result_size*sizeof(Eigen::half));
  gpu_device.synchronize();

  for (int i = 0; i < result_size; ++i) {
    std::cout << "EXPECTED " << full_prec(i) << " GOT " << half_prec(i) << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec(i));
  }

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}

template<typename>
void test_gpu_reductions() {
  test_gpu_reductions<void>(13, 13, 0);
  test_gpu_reductions<void>(13, 13, 1);

  test_gpu_reductions<void>(35, 36, 0);
  test_gpu_reductions<void>(35, 36, 1);

  test_gpu_reductions<void>(36, 35, 0);
  test_gpu_reductions<void>(36, 35, 1);
}

template<typename>
void test_gpu_full_reductions() {
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int size = 13;
  int num_elem = size*size;

  float* d_float1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_float2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen::half* d_res_half = (Eigen::half*)gpu_device.allocate(1 * sizeof(Eigen::half));
  Eigen::half* d_res_float = (Eigen::half*)gpu_device.allocate(1 * sizeof(Eigen::half));

  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float1(
      d_float1, size, size);
  Eigen::TensorMap<Eigen::Tensor<float, 2>, Eigen::Aligned> gpu_float2(
      d_float2, size, size);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 0>, Eigen::Aligned> gpu_res_half(
      d_res_half);
  Eigen::TensorMap<Eigen::Tensor<Eigen::half, 0>, Eigen::Aligned> gpu_res_float(
      d_res_float);

  gpu_float1.device(gpu_device) = gpu_float1.random();
  gpu_float2.device(gpu_device) = gpu_float2.random();

  gpu_res_float.device(gpu_device) = gpu_float1.sum().cast<Eigen::half>();
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().sum();

  Tensor<Eigen::half, 0> half_prec;
  Tensor<Eigen::half, 0> full_prec;
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, sizeof(Eigen::half));
  gpu_device.synchronize();

  VERIFY_IS_APPROX(full_prec(), half_prec());

  gpu_res_float.device(gpu_device) = gpu_float1.maximum().cast<Eigen::half>();
  gpu_res_half.device(gpu_device) = gpu_float1.cast<Eigen::half>().maximum();
  gpu_device.memcpyDeviceToHost(half_prec.data(), d_res_half, sizeof(Eigen::half));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, sizeof(Eigen::half));
  gpu_device.synchronize();

  VERIFY_IS_APPROX(full_prec(), half_prec());

  gpu_device.deallocate(d_float1);
  gpu_device.deallocate(d_float2);
  gpu_device.deallocate(d_res_half);
  gpu_device.deallocate(d_res_float);
}

template<typename>
void test_gpu_forced_evals() {

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half1 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_half2 = (float*)gpu_device.allocate(num_elem * sizeof(float));
  float* d_res_float = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_float(
      d_float, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_half1(
      d_res_half1, num_elem);
 Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Unaligned> gpu_res_half2(
      d_res_half2, num_elem);
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_res_float(
      d_res_float, num_elem);

  Eigen::array<int, 1> no_bcast;
  no_bcast[0] = 1;

  gpu_float.device(gpu_device) = gpu_float.random() - gpu_float.constant(0.5f);
  gpu_res_float.device(gpu_device) = gpu_float.abs();
  gpu_res_half1.device(gpu_device) = gpu_float.cast<Eigen::half>().abs().eval().cast<float>();
  gpu_res_half2.device(gpu_device) = gpu_float.cast<Eigen::half>().abs().broadcast(no_bcast).eval().cast<float>();

  Tensor<float, 1> half_prec1(num_elem);
  Tensor<float, 1> half_prec2(num_elem);
  Tensor<float, 1> full_prec(num_elem);
  gpu_device.memcpyDeviceToHost(half_prec1.data(), d_res_half1, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(half_prec2.data(), d_res_half1, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(full_prec.data(), d_res_float, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    std::cout << "Checking forced eval " << i << full_prec(i) << " vs " << half_prec1(i) << " vs " << half_prec2(i) << std::endl;
    VERIFY_IS_APPROX(full_prec(i), half_prec1(i));
    VERIFY_IS_APPROX(full_prec(i), half_prec2(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_res_half1);
  gpu_device.deallocate(d_res_half2);
  gpu_device.deallocate(d_res_float);
}
#endif


EIGEN_DECLARE_TEST(cxx11_tensor_of_float16_gpu)
{
  CALL_SUBTEST_1(test_gpu_numext<void>());

#ifdef EIGEN_HAS_GPU_FP16
  CALL_SUBTEST_1(test_gpu_conversion<void>());
  CALL_SUBTEST_1(test_gpu_unary<void>());
  CALL_SUBTEST_1(test_gpu_elementwise<void>());
  CALL_SUBTEST_1(test_gpu_trancendental<void>());
  CALL_SUBTEST_2(test_gpu_contractions<void>());
  CALL_SUBTEST_3(test_gpu_reductions<void>());
  CALL_SUBTEST_4(test_gpu_full_reductions<void>());
  CALL_SUBTEST_5(test_gpu_forced_evals<void>());
#else
  std::cout << "Half floats are not supported by this version of gpu: skipping the test" << std::endl;
#endif
}
