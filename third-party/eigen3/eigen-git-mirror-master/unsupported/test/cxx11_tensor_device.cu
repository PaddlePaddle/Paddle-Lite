#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <unsupported/Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

using Eigen::Tensor;
using Eigen::RowMajor;

// Context for evaluation on cpu
struct CPUContext {
  CPUContext(const Eigen::Tensor<float, 3>& in1, Eigen::Tensor<float, 3>& in2, Eigen::Tensor<float, 3>& out) : in1_(in1), in2_(in2), out_(out), kernel_1d_(2), kernel_2d_(2,2), kernel_3d_(2,2,2) {
    kernel_1d_(0) = 3.14f;
    kernel_1d_(1) = 2.7f;

    kernel_2d_(0,0) = 3.14f;
    kernel_2d_(1,0) = 2.7f;
    kernel_2d_(0,1) = 0.2f;
    kernel_2d_(1,1) = 7.0f;

    kernel_3d_(0,0,0) = 3.14f;
    kernel_3d_(0,1,0) = 2.7f;
    kernel_3d_(0,0,1) = 0.2f;
    kernel_3d_(0,1,1) = 7.0f;
    kernel_3d_(1,0,0) = -1.0f;
    kernel_3d_(1,1,0) = -0.3f;
    kernel_3d_(1,0,1) = -0.7f;
    kernel_3d_(1,1,1) = -0.5f;
  }

  const Eigen::DefaultDevice& device() const { return cpu_device_; }

  const Eigen::Tensor<float, 3>& in1() const { return in1_; }
  const Eigen::Tensor<float, 3>& in2() const { return in2_; }
  Eigen::Tensor<float, 3>& out() { return out_; }
  const Eigen::Tensor<float, 1>& kernel1d() const { return kernel_1d_; }
  const Eigen::Tensor<float, 2>& kernel2d() const { return kernel_2d_; }
  const Eigen::Tensor<float, 3>& kernel3d() const { return kernel_3d_; }

 private:
  const Eigen::Tensor<float, 3>& in1_;
  const Eigen::Tensor<float, 3>& in2_;
  Eigen::Tensor<float, 3>& out_;

  Eigen::Tensor<float, 1> kernel_1d_;
  Eigen::Tensor<float, 2> kernel_2d_;
  Eigen::Tensor<float, 3> kernel_3d_;

  Eigen::DefaultDevice cpu_device_;
};


// Context for evaluation on GPU
struct GPUContext {
  GPUContext(const Eigen::TensorMap<Eigen::Tensor<float, 3> >& in1, Eigen::TensorMap<Eigen::Tensor<float, 3> >& in2, Eigen::TensorMap<Eigen::Tensor<float, 3> >& out) : in1_(in1), in2_(in2), out_(out), gpu_device_(&stream_) {
    assert(gpuMalloc((void**)(&kernel_1d_), 2*sizeof(float)) == gpuSuccess);
    float kernel_1d_val[] = {3.14f, 2.7f};
    assert(gpuMemcpy(kernel_1d_, kernel_1d_val, 2*sizeof(float), gpuMemcpyHostToDevice) == gpuSuccess);

    assert(gpuMalloc((void**)(&kernel_2d_), 4*sizeof(float)) == gpuSuccess);
    float kernel_2d_val[] = {3.14f, 2.7f, 0.2f, 7.0f};
    assert(gpuMemcpy(kernel_2d_, kernel_2d_val, 4*sizeof(float), gpuMemcpyHostToDevice) == gpuSuccess);

    assert(gpuMalloc((void**)(&kernel_3d_), 8*sizeof(float)) == gpuSuccess);
    float kernel_3d_val[] = {3.14f, -1.0f, 2.7f, -0.3f, 0.2f, -0.7f, 7.0f, -0.5f};
    assert(gpuMemcpy(kernel_3d_, kernel_3d_val, 8*sizeof(float), gpuMemcpyHostToDevice) == gpuSuccess);
  }
  ~GPUContext() {
    assert(gpuFree(kernel_1d_) == gpuSuccess);
    assert(gpuFree(kernel_2d_) == gpuSuccess);
    assert(gpuFree(kernel_3d_) == gpuSuccess);
  }

  const Eigen::GpuDevice& device() const { return gpu_device_; }

  const Eigen::TensorMap<Eigen::Tensor<float, 3> >& in1() const { return in1_; }
  const Eigen::TensorMap<Eigen::Tensor<float, 3> >& in2() const { return in2_; }
  Eigen::TensorMap<Eigen::Tensor<float, 3> >& out() { return out_; }
  Eigen::TensorMap<Eigen::Tensor<float, 1> > kernel1d() const { return Eigen::TensorMap<Eigen::Tensor<float, 1> >(kernel_1d_, 2); }
  Eigen::TensorMap<Eigen::Tensor<float, 2> > kernel2d() const { return Eigen::TensorMap<Eigen::Tensor<float, 2> >(kernel_2d_, 2, 2); }
  Eigen::TensorMap<Eigen::Tensor<float, 3> > kernel3d() const { return Eigen::TensorMap<Eigen::Tensor<float, 3> >(kernel_3d_, 2, 2, 2); }

 private:
  const Eigen::TensorMap<Eigen::Tensor<float, 3> >& in1_;
  const Eigen::TensorMap<Eigen::Tensor<float, 3> >& in2_;
  Eigen::TensorMap<Eigen::Tensor<float, 3> >& out_;

  float* kernel_1d_;
  float* kernel_2d_;
  float* kernel_3d_;

  Eigen::GpuStreamDevice stream_;
  Eigen::GpuDevice gpu_device_;
};


// The actual expression to evaluate
template <typename Context>
void test_contextual_eval(Context* context)
{
  context->out().device(context->device()) = context->in1() + context->in2() * 3.14f + context->in1().constant(2.718f);
}

template <typename Context>
void test_forced_contextual_eval(Context* context)
{
  context->out().device(context->device()) = (context->in1() + context->in2()).eval() * 3.14f + context->in1().constant(2.718f);
}

template <typename Context>
void test_compound_assignment(Context* context)
{
  context->out().device(context->device()) = context->in1().constant(2.718f);
  context->out().device(context->device()) += context->in1() + context->in2() * 3.14f;
}


template <typename Context>
void test_contraction(Context* context)
{
  Eigen::array<std::pair<int, int>, 2> dims;
  dims[0] = std::make_pair(1, 1);
  dims[1] = std::make_pair(2, 2);

  Eigen::array<int, 2> shape(40, 50*70);

  Eigen::DSizes<int, 2> indices(0,0);
  Eigen::DSizes<int, 2> sizes(40,40);

  context->out().reshape(shape).slice(indices, sizes).device(context->device()) = context->in1().contract(context->in2(), dims);
}


template <typename Context>
void test_1d_convolution(Context* context)
{
  Eigen::DSizes<int, 3> indices(0,0,0);
  Eigen::DSizes<int, 3> sizes(40,49,70);

  Eigen::array<int, 1> dims(1);
  context->out().slice(indices, sizes).device(context->device()) = context->in1().convolve(context->kernel1d(), dims);
}

template <typename Context>
void test_2d_convolution(Context* context)
{
  Eigen::DSizes<int, 3> indices(0,0,0);
  Eigen::DSizes<int, 3> sizes(40,49,69);

  Eigen::array<int, 2> dims(1,2);
  context->out().slice(indices, sizes).device(context->device()) = context->in1().convolve(context->kernel2d(), dims);
}

template <typename Context>
void test_3d_convolution(Context* context)
{
  Eigen::DSizes<int, 3> indices(0,0,0);
  Eigen::DSizes<int, 3> sizes(39,49,69);

  Eigen::array<int, 3> dims(0,1,2);
  context->out().slice(indices, sizes).device(context->device()) = context->in1().convolve(context->kernel3d(), dims);
}


void test_cpu() {
  Eigen::Tensor<float, 3> in1(40,50,70);
  Eigen::Tensor<float, 3> in2(40,50,70);
  Eigen::Tensor<float, 3> out(40,50,70);

  in1 = in1.random() + in1.constant(10.0f);
  in2 = in2.random() + in2.constant(10.0f);

  CPUContext context(in1, in2, out);
  test_contextual_eval(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), in1(i,j,k) + in2(i,j,k) * 3.14f + 2.718f);
      }
    }
  }

  test_forced_contextual_eval(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), (in1(i,j,k) + in2(i,j,k)) * 3.14f + 2.718f);
      }
    }
  }

  test_compound_assignment(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), in1(i,j,k) + in2(i,j,k) * 3.14f + 2.718f);
      }
    }
  }

  test_contraction(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 40; ++j) {
      const float result = out(i,j,0);
      float expected = 0;
      for (int k = 0; k < 50; ++k) {
        for (int l = 0; l < 70; ++l) {
          expected += in1(i, k, l) * in2(j, k, l);
        }
      }
      VERIFY_IS_APPROX(expected, result);
    }
  }

  test_1d_convolution(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f));
      }
    }
  }

  test_2d_convolution(&context);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 69; ++k) {
        const float result = out(i,j,k);
        const float expected = (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f) +
                               (in1(i,j,k+1) * 0.2f + in1(i,j+1,k+1) * 7.0f);
        if (fabs(expected) < 1e-4f && fabs(result) < 1e-4f) {
          continue;
        }
        VERIFY_IS_APPROX(expected, result);
      }
    }
  }

  test_3d_convolution(&context);
  for (int i = 0; i < 39; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 69; ++k) {
        const float result = out(i,j,k);
        const float expected = (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f +
                                in1(i,j,k+1) * 0.2f + in1(i,j+1,k+1) * 7.0f) +
                               (in1(i+1,j,k) * -1.0f + in1(i+1,j+1,k) * -0.3f +
                                in1(i+1,j,k+1) * -0.7f + in1(i+1,j+1,k+1) * -0.5f);
        if (fabs(expected) < 1e-4f && fabs(result) < 1e-4f) {
          continue;
        }
        VERIFY_IS_APPROX(expected, result);
      }
    }
  }
}

void test_gpu() {
  Eigen::Tensor<float, 3> in1(40,50,70);
  Eigen::Tensor<float, 3> in2(40,50,70);
  Eigen::Tensor<float, 3> out(40,50,70);
  in1 = in1.random() + in1.constant(10.0f);
  in2 = in2.random() + in2.constant(10.0f);

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_in2), in2_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), in2_bytes, gpuMemcpyHostToDevice);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, 40,50,70);
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, 40,50,70);
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, 40,50,70);

  GPUContext context(gpu_in1, gpu_in2, gpu_out);
  test_contextual_eval(&context);
  assert(gpuMemcpy(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), in1(i,j,k) + in2(i,j,k) * 3.14f + 2.718f);
      }
    }
  }

  test_forced_contextual_eval(&context);
  assert(gpuMemcpy(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), (in1(i,j,k) + in2(i,j,k)) * 3.14f + 2.718f);
      }
    }
  }

  test_compound_assignment(&context);
  assert(gpuMemcpy(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 50; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), in1(i,j,k) + in2(i,j,k) * 3.14f + 2.718f);
      }
    }
  }

  test_contraction(&context);
  assert(gpuMemcpy(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 40; ++j) {
      const float result = out(i,j,0);
      float expected = 0;
      for (int k = 0; k < 50; ++k) {
        for (int l = 0; l < 70; ++l) {
          expected += in1(i, k, l) * in2(j, k, l);
        }
      }
      VERIFY_IS_APPROX(expected, result);
    }
  }

  test_1d_convolution(&context);
  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, context.device().stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(context.device().stream()) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 70; ++k) {
        VERIFY_IS_APPROX(out(i,j,k), (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f));
      }
    }
  }

  test_2d_convolution(&context);
  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, context.device().stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(context.device().stream()) == gpuSuccess);
  for (int i = 0; i < 40; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 69; ++k) {
        const float result = out(i,j,k);
        const float expected = (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f +
                                in1(i,j,k+1) * 0.2f + in1(i,j+1,k+1) * 7.0f);
        VERIFY_IS_APPROX(expected, result);
      }
    }
  }

#if !defined(EIGEN_USE_HIP)
// disable this test on the HIP platform
// 3D tensor convolutions seem to hang on the HIP platform

  test_3d_convolution(&context);
  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, context.device().stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(context.device().stream()) == gpuSuccess);
  for (int i = 0; i < 39; ++i) {
    for (int j = 0; j < 49; ++j) {
      for (int k = 0; k < 69; ++k) {
       const float result = out(i,j,k);
        const float expected = (in1(i,j,k) * 3.14f + in1(i,j+1,k) * 2.7f +
                                in1(i,j,k+1) * 0.2f + in1(i,j+1,k+1) * 7.0f +
                                in1(i+1,j,k) * -1.0f + in1(i+1,j+1,k) * -0.3f +
                                in1(i+1,j,k+1) * -0.7f + in1(i+1,j+1,k+1) * -0.5f);
        VERIFY_IS_APPROX(expected, result);
      }
    }
  }

#endif
 
}


EIGEN_DECLARE_TEST(cxx11_tensor_device)
{
  CALL_SUBTEST_1(test_cpu());
  CALL_SUBTEST_2(test_gpu());
}
