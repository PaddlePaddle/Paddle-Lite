#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

using Eigen::Tensor;
typedef Tensor<float, 1>::DimensionPair DimPair;

template<int DataLayout>
void test_gpu_cumsum(int m_size, int k_size, int n_size)
{
  std::cout << "Testing for (" << m_size << "," << k_size << "," << n_size << ")" << std::endl;
  Tensor<float, 3, DataLayout> t_input(m_size, k_size, n_size);
  Tensor<float, 3, DataLayout> t_result(m_size, k_size, n_size);
  Tensor<float, 3, DataLayout> t_result_gpu(m_size, k_size, n_size);

  t_input.setRandom();

  std::size_t t_input_bytes = t_input.size()  * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_input;
  float* d_t_result;

  gpuMalloc((void**)(&d_t_input), t_input_bytes);
  gpuMalloc((void**)(&d_t_result), t_result_bytes);

  gpuMemcpy(d_t_input, t_input.data(), t_input_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> >
      gpu_t_input(d_t_input, Eigen::array<int, 3>(m_size, k_size, n_size));
  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> >
      gpu_t_result(d_t_result, Eigen::array<int, 3>(m_size, k_size, n_size));

  gpu_t_result.device(gpu_device) = gpu_t_input.cumsum(1);
  t_result = t_input.cumsum(1);

  gpuMemcpy(t_result_gpu.data(), d_t_result, t_result_bytes, gpuMemcpyDeviceToHost);
  for (DenseIndex i = 0; i < t_result.size(); i++) {
    if (fabs(t_result(i) - t_result_gpu(i)) < 1e-4f) {
      continue;
    }
    if (Eigen::internal::isApprox(t_result(i), t_result_gpu(i), 1e-4f)) {
      continue;
    }
    std::cout << "mismatch detected at index " << i << ": " << t_result(i)
              << " vs " <<  t_result_gpu(i) << std::endl;
    assert(false);
  }

  gpuFree((void*)d_t_input);
  gpuFree((void*)d_t_result);
}


EIGEN_DECLARE_TEST(cxx11_tensor_scan_gpu)
{
  CALL_SUBTEST_1(test_gpu_cumsum<ColMajor>(128, 128, 128));
  CALL_SUBTEST_2(test_gpu_cumsum<RowMajor>(128, 128, 128));
}
