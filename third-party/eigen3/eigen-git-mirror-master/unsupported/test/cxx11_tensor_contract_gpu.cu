#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
// Copyright (C) 2014 Navdeep Jaitly <ndjaitly@google.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <unsupported/Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

using Eigen::Tensor;
typedef Tensor<float, 1>::DimensionPair DimPair;

template<int DataLayout>
void test_gpu_contraction(int m_size, int k_size, int n_size)
{
  std::cout << "Testing for (" << m_size << "," << k_size << "," << n_size << ")" << std::endl;
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 2, DataLayout> t_left(m_size, k_size);
  Tensor<float, 2, DataLayout> t_right(k_size, n_size);
  Tensor<float, 2, DataLayout> t_result(m_size, n_size);
  Tensor<float, 2, DataLayout> t_result_gpu(m_size, n_size);
  Eigen::array<DimPair, 1> dims(DimPair(1, 0));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  gpuMalloc((void**)(&d_t_left), t_left_bytes);
  gpuMalloc((void**)(&d_t_right), t_right_bytes);
  gpuMalloc((void**)(&d_t_result), t_result_bytes);

  gpuMemcpy(d_t_left, t_left.data(), t_left_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_t_right, t_right.data(), t_right_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> >
      gpu_t_left(d_t_left, Eigen::array<int, 2>(m_size, k_size));
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> >
      gpu_t_right(d_t_right, Eigen::array<int, 2>(k_size, n_size));
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> >
      gpu_t_result(d_t_result, Eigen::array<int, 2>(m_size, n_size));


  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);
  t_result = t_left.contract(t_right, dims);

  gpuMemcpy(t_result_gpu.data(), d_t_result, t_result_bytes, gpuMemcpyDeviceToHost);
  for (DenseIndex i = 0; i < t_result.size(); i++) {
    if (fabs(t_result(i) - t_result_gpu(i)) < 1e-4f) {
      continue;
    }
    if (Eigen::internal::isApprox(t_result(i), t_result_gpu(i), 1e-4f)) {
      continue;
    }
    std::cout << "mismatch detected at index " << i << ": " << t_result(i)
              << " vs " <<  t_result_gpu(i) << std::endl;
    assert(false);
  }

  gpuFree((void*)d_t_left);
  gpuFree((void*)d_t_right);
  gpuFree((void*)d_t_result);
}


template<int DataLayout>
void test_scalar(int m_size, int k_size, int n_size)
{
  std::cout << "Testing for (" << m_size << "," << k_size << "," << n_size << ")" << std::endl;
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 2, DataLayout> t_left(m_size, k_size);
  Tensor<float, 2, DataLayout> t_right(k_size, n_size);
  Tensor<float, 0, DataLayout> t_result;
  Tensor<float, 0, DataLayout> t_result_gpu;
  Eigen::array<DimPair, 2> dims(DimPair(0, 0), DimPair(1, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  gpuMalloc((void**)(&d_t_left), t_left_bytes);
  gpuMalloc((void**)(&d_t_right), t_right_bytes);
  gpuMalloc((void**)(&d_t_result), t_result_bytes);

  gpuMemcpy(d_t_left, t_left.data(), t_left_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_t_right, t_right.data(), t_right_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> >
      gpu_t_left(d_t_left, m_size, k_size);
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> >
      gpu_t_right(d_t_right, k_size, n_size);
  Eigen::TensorMap<Eigen::Tensor<float, 0, DataLayout> >
      gpu_t_result(d_t_result);

  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);
  t_result = t_left.contract(t_right, dims);

  gpuMemcpy(t_result_gpu.data(), d_t_result, t_result_bytes, gpuMemcpyDeviceToHost);
  if (fabs(t_result() - t_result_gpu()) > 1e-4f &&
      !Eigen::internal::isApprox(t_result(), t_result_gpu(), 1e-4f)) {
    std::cout << "mismatch detected: " << t_result()
              << " vs " <<  t_result_gpu() << std::endl;
    assert(false);
  }

  gpuFree((void*)d_t_left);
  gpuFree((void*)d_t_right);
  gpuFree((void*)d_t_result);
}


template<int DataLayout>
void test_gpu_contraction_m() {
  for (int k = 32; k < 256; k++) {
    test_gpu_contraction<ColMajor>(k, 128, 128);
    test_gpu_contraction<RowMajor>(k, 128, 128);
  }
}

template<int DataLayout>
void test_gpu_contraction_k() {
  for (int k = 32; k < 256; k++) {
    test_gpu_contraction<ColMajor>(128, k, 128);
    test_gpu_contraction<RowMajor>(128, k, 128);
  }
}

template<int DataLayout>
void test_gpu_contraction_n() {
  for (int k = 32; k < 256; k++) {
    test_gpu_contraction<ColMajor>(128, 128, k);
    test_gpu_contraction<RowMajor>(128, 128, k);
  }
}


template<int DataLayout>
void test_gpu_contraction_sizes() {
  int m_sizes[] = { 31,  39,   63,   64,   65,
                   127, 129,  255,  257 , 511,
                   512, 513, 1023, 1024, 1025};

  int n_sizes[] = { 31,  39,   63,   64,   65,
                   127, 129,  255,  257,  511,
                   512, 513, 1023, 1024, 1025};

  int k_sizes[] = {  31,   39,  63,  64,   65,
                     95,   96, 127, 129,  255,
                    257,  511, 512, 513, 1023,
                   1024, 1025};

  for (int i = 0; i < 15; i++) {
    for (int j = 0; j < 15; j++) {
      for (int k = 0; k < 17; k++) {
        test_gpu_contraction<DataLayout>(m_sizes[i], n_sizes[j], k_sizes[k]);
      }
    }
  }
}

EIGEN_DECLARE_TEST(cxx11_tensor_contract_gpu)
{
  CALL_SUBTEST_1(test_gpu_contraction<ColMajor>(128, 128, 128));
  CALL_SUBTEST_1(test_gpu_contraction<RowMajor>(128, 128, 128));

  CALL_SUBTEST_1(test_scalar<ColMajor>(128, 128, 128));
  CALL_SUBTEST_1(test_scalar<RowMajor>(128, 128, 128));

  CALL_SUBTEST_2(test_gpu_contraction_m<ColMajor>());
  CALL_SUBTEST_3(test_gpu_contraction_m<RowMajor>());

  CALL_SUBTEST_4(test_gpu_contraction_k<ColMajor>());
  CALL_SUBTEST_5(test_gpu_contraction_k<RowMajor>());

  CALL_SUBTEST_6(test_gpu_contraction_n<ColMajor>());
  CALL_SUBTEST_7(test_gpu_contraction_n<RowMajor>());

#if !defined(EIGEN_USE_HIP)
// disable these subtests for HIP
  CALL_SUBTEST_8(test_gpu_contraction_sizes<ColMajor>());
  CALL_SUBTEST_9(test_gpu_contraction_sizes<RowMajor>());
#endif	
}
