// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE

#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

template<typename T>
void test_cuda_complex_cwise_ops() {
  const int kNumItems = 2;
  std::size_t complex_bytes = kNumItems * sizeof(std::complex<T>);

  std::complex<T>* d_in1;
  std::complex<T>* d_in2;
  std::complex<T>* d_out;
  hipMalloc((void**)(&d_in1), complex_bytes);
  hipMalloc((void**)(&d_in2), complex_bytes);
  hipMalloc((void**)(&d_out), complex_bytes);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<std::complex<T>, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, kNumItems);
  Eigen::TensorMap<Eigen::Tensor<std::complex<T>, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, kNumItems);
  Eigen::TensorMap<Eigen::Tensor<std::complex<T>, 1, 0, int>, Eigen::Aligned> gpu_out(
      d_out, kNumItems);

  const std::complex<T> a(3.14f, 2.7f);
  const std::complex<T> b(-10.6f, 1.4f);

  gpu_in1.device(gpu_device) = gpu_in1.constant(a);
  gpu_in2.device(gpu_device) = gpu_in2.constant(b);

  enum CwiseOp {
    Add = 0,
    Sub,
    Mul,
    Div,
    Neg,
    NbOps
  };

  Tensor<std::complex<T>, 1, 0, int> actual(kNumItems);
  for (int op = Add; op < NbOps; op++) {
    std::complex<T> expected;
    switch (static_cast<CwiseOp>(op)) {
      case Add:
        gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;
        expected = a + b;
        break;
      case Sub:
        gpu_out.device(gpu_device) = gpu_in1 - gpu_in2;
        expected = a - b;
        break;
      case Mul:
        gpu_out.device(gpu_device) = gpu_in1 * gpu_in2;
        expected = a * b;
        break;
      case Div:
        gpu_out.device(gpu_device) = gpu_in1 / gpu_in2;
        expected = a / b;
        break;
      case Neg:
        gpu_out.device(gpu_device) = -gpu_in1;
        expected = -a;
        break;
    }
    assert(hipMemcpyAsync(actual.data(), d_out, complex_bytes, hipMemcpyDeviceToHost,
                           gpu_device.stream()) == hipSuccess);
    assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

    for (int i = 0; i < kNumItems; ++i) {
      VERIFY_IS_APPROX(actual(i), expected);
    }
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}


EIGEN_DECLARE_TEST(test_cxx11_tensor_complex_cwise_ops)
{
  CALL_SUBTEST(test_cuda_complex_cwise_ops<float>());
  CALL_SUBTEST(test_cuda_complex_cwise_ops<double>());
}
