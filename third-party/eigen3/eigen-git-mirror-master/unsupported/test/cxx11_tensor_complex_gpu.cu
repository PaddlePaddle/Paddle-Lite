#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE

#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

void test_cuda_nullary() {
  Tensor<std::complex<float>, 1, 0, int> in1(2);
  Tensor<std::complex<float>, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t float_bytes = in1.size() * sizeof(float);
  std::size_t complex_bytes = in1.size() * sizeof(std::complex<float>);

  std::complex<float>* d_in1;
  std::complex<float>* d_in2;
  float* d_out2;
  hipMalloc((void**)(&d_in1), complex_bytes);
  hipMalloc((void**)(&d_in2), complex_bytes);
  hipMalloc((void**)(&d_out2), float_bytes);
  hipMemcpy(d_in1, in1.data(), complex_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), complex_bytes, hipMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<std::complex<float>, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<std::complex<float>, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_out2(
      d_out2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(std::complex<float>(3.14f, 2.7f));
  gpu_out2.device(gpu_device) = gpu_in2.abs();

  Tensor<std::complex<float>, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(hipMemcpyAsync(new1.data(), d_in1, complex_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipMemcpyAsync(new2.data(), d_out2, float_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);

  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), std::complex<float>(3.14f, 2.7f));
    VERIFY_IS_APPROX(new2(i), std::abs(in2(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out2);
}


static void test_cuda_sum_reductions() {

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  const int num_rows = internal::random<int>(1024, 5*1024);
  const int num_cols = internal::random<int>(1024, 5*1024);

  Tensor<std::complex<float>, 2> in(num_rows, num_cols);
  in.setRandom();

  Tensor<std::complex<float>, 0> full_redux;
  full_redux = in.sum();

  std::size_t in_bytes = in.size() * sizeof(std::complex<float>);
  std::size_t out_bytes = full_redux.size() * sizeof(std::complex<float>);
  std::complex<float>* gpu_in_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(in_bytes));
  std::complex<float>* gpu_out_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(out_bytes));
  gpu_device.memcpyHostToDevice(gpu_in_ptr, in.data(), in_bytes);

  TensorMap<Tensor<std::complex<float>, 2> > in_gpu(gpu_in_ptr, num_rows, num_cols);
  TensorMap<Tensor<std::complex<float>, 0> > out_gpu(gpu_out_ptr);

  out_gpu.device(gpu_device) = in_gpu.sum();

  Tensor<std::complex<float>, 0> full_redux_gpu;
  gpu_device.memcpyDeviceToHost(full_redux_gpu.data(), gpu_out_ptr, out_bytes);
  gpu_device.synchronize();

  // Check that the CPU and GPU reductions return the same result.
  VERIFY_IS_APPROX(full_redux(), full_redux_gpu());

  gpu_device.deallocate(gpu_in_ptr);
  gpu_device.deallocate(gpu_out_ptr);
}

static void test_cuda_mean_reductions() {

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  const int num_rows = internal::random<int>(1024, 5*1024);
  const int num_cols = internal::random<int>(1024, 5*1024);

  Tensor<std::complex<float>, 2> in(num_rows, num_cols);
  in.setRandom();

  Tensor<std::complex<float>, 0> full_redux;
  full_redux = in.mean();

  std::size_t in_bytes = in.size() * sizeof(std::complex<float>);
  std::size_t out_bytes = full_redux.size() * sizeof(std::complex<float>);
  std::complex<float>* gpu_in_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(in_bytes));
  std::complex<float>* gpu_out_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(out_bytes));
  gpu_device.memcpyHostToDevice(gpu_in_ptr, in.data(), in_bytes);

  TensorMap<Tensor<std::complex<float>, 2> > in_gpu(gpu_in_ptr, num_rows, num_cols);
  TensorMap<Tensor<std::complex<float>, 0> > out_gpu(gpu_out_ptr);

  out_gpu.device(gpu_device) = in_gpu.mean();

  Tensor<std::complex<float>, 0> full_redux_gpu;
  gpu_device.memcpyDeviceToHost(full_redux_gpu.data(), gpu_out_ptr, out_bytes);
  gpu_device.synchronize();

  // Check that the CPU and GPU reductions return the same result.
  VERIFY_IS_APPROX(full_redux(), full_redux_gpu());

  gpu_device.deallocate(gpu_in_ptr);
  gpu_device.deallocate(gpu_out_ptr);
}

static void test_cuda_product_reductions() {

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  const int num_rows = internal::random<int>(1024, 5*1024);
  const int num_cols = internal::random<int>(1024, 5*1024);

  Tensor<std::complex<float>, 2> in(num_rows, num_cols);
  in.setRandom();

  Tensor<std::complex<float>, 0> full_redux;
  full_redux = in.prod();

  std::size_t in_bytes = in.size() * sizeof(std::complex<float>);
  std::size_t out_bytes = full_redux.size() * sizeof(std::complex<float>);
  std::complex<float>* gpu_in_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(in_bytes));
  std::complex<float>* gpu_out_ptr = static_cast<std::complex<float>*>(gpu_device.allocate(out_bytes));
  gpu_device.memcpyHostToDevice(gpu_in_ptr, in.data(), in_bytes);

  TensorMap<Tensor<std::complex<float>, 2> > in_gpu(gpu_in_ptr, num_rows, num_cols);
  TensorMap<Tensor<std::complex<float>, 0> > out_gpu(gpu_out_ptr);

  out_gpu.device(gpu_device) = in_gpu.prod();

  Tensor<std::complex<float>, 0> full_redux_gpu;
  gpu_device.memcpyDeviceToHost(full_redux_gpu.data(), gpu_out_ptr, out_bytes);
  gpu_device.synchronize();

  // Check that the CPU and GPU reductions return the same result.
  VERIFY_IS_APPROX(full_redux(), full_redux_gpu());

  gpu_device.deallocate(gpu_in_ptr);
  gpu_device.deallocate(gpu_out_ptr);
}


EIGEN_DECLARE_TEST(test_cxx11_tensor_complex)
{
  CALL_SUBTEST(test_cuda_nullary());
  CALL_SUBTEST(test_cuda_sum_reductions());
  CALL_SUBTEST(test_cuda_mean_reductions());
  CALL_SUBTEST(test_cuda_product_reductions());
}
