#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <vector>
#include "lite/backends/cuda/cuda_utils.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_group_padding_compute.h"

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {
using Tensor = lite::Tensor;

template <typename Dtype>
__global__ void ker_search_group_padding(Dtype* out_emb_padding_data,
                                         Dtype* out_padding_data,
                                         const Dtype* in_data,
                                         const uint64_t* offset,
                                         const int seq_num,
                                         const int max_len,
                                         const int emb_size,
                                         const Dtype pad_id,
                                         const int count) {
  CUDA_KERNEL_LOOP(tid, count) {
    int emb_id = tid % emb_size;
    int word_id = tid / emb_size;
    int seq_id = word_id / max_len;
    int word_id_in_seq = word_id % max_len;
    int cur_len = offset[seq_id + 1] - offset[seq_id];
    if (word_id_in_seq < cur_len) {
      out_emb_padding_data[tid] =
          in_data[(offset[seq_id] + word_id_in_seq) * emb_size + emb_id];
    } else {
      out_emb_padding_data[tid] = 0.f;
      if (emb_id == 0) {
        out_padding_data[word_id] = pad_id;
      }
    }
  }
}

void SearchGroupPaddingCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto hip_stream = ctx.exec_stream();

  const Tensor* x = param.x;
  Tensor* out_emb_padding = param.out_emb_padding;
  Tensor* out_new = param.out_new;
  Tensor* out_padding = param.out_padding;
  const float pad_id = static_cast<float>(param.pad_id);
  const float* in_data = x->data<float>();
  const auto& in_seq_offset = x->lod()[0];
  int batch = in_seq_offset.size() - 1;
  int max_seq = 0;
  for (int i = 0; i < batch; ++i) {
    if (in_seq_offset[i + 1] - in_seq_offset[i] > max_seq) {
      max_seq = in_seq_offset[i + 1] - in_seq_offset[i];
    }
  }
  std::vector<size_t> new_offset;
  new_offset.resize(batch + 1);
  for (int i = 0; i < batch + 1; ++i) {
    new_offset[i] = i * max_seq;
  }
  std::vector<int64_t> x_dims = x->dims().Vectorize();
  LoD out_emb_padding_lod;
  out_emb_padding_lod.push_back(new_offset);
  out_emb_padding->set_lod(out_emb_padding_lod);
  out_emb_padding->Resize({batch * max_seq, x_dims[1]});
  float* out_emb_padding_data =
      out_emb_padding->mutable_data<float>(TARGET(kCUDA));

  LoD out_new_lod;
  out_new_lod.push_back(in_seq_offset);
  out_new->set_lod(out_new_lod);
  out_new->Resize({x_dims[0], 1});
  float* out_new_data = out_new->mutable_data<float>(TARGET(kCUDA));

  LoD out_padding_lod;
  out_padding_lod.push_back(new_offset);
  out_padding->set_lod(out_padding_lod);
  out_padding->Resize({batch * max_seq, 1});
  float* out_padding_data = out_padding->mutable_data<float>(TARGET(kCUDA));

  const int count = out_emb_padding->numel();
  const auto& out_emb_padding_seq_offset = out_emb_padding->lod()[0];
  int max_len = out_emb_padding_seq_offset[1];
  int seq_num = out_emb_padding_seq_offset.size() - 1;
  int emb_size = x->dims()[1];
  _in_seq_offset.Resize({seq_num + 1, 1, 1, 1});
  uint64_t* offset_data = _in_seq_offset.mutable_data<uint64_t>(TARGET(kCUDA));

  TargetWrapperCuda::MemcpyAsync(offset_data,
                                 in_seq_offset.data(),
                                 sizeof(uint64_t) * in_seq_offset.size(),
                                 IoDirection::HtoD,
                                 hip_stream);

  TargetWrapperCuda::MemsetSync(
      out_new_data, 0, out_new->dims()[0] * out_new->dims()[1] * sizeof(float));
  TargetWrapperCuda::MemsetSync(
      out_padding_data,
      0,
      out_padding->dims()[0] * out_padding->dims()[1] * sizeof(float));

  ker_search_group_padding<
      float><<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
      out_emb_padding_data,
      out_padding_data,
      in_data,
      offset_data,
      seq_num,
      max_len,
      emb_size,
      pad_id,
      count);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(search_group_padding,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::SearchGroupPaddingCompute,
                     def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out_emb_padding",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .BindOutput("Out_new",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .BindOutput("Out_padding",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();
