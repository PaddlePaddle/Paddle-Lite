#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_seq_fc_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename dtype>
__global__ void add_bias(int n,
                         int output_size,
                         const dtype* bias,
                         dtype* dout) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int bias_index = index % output_size;
  if (index < n) {
    dout[index] = dout[index] + bias[bias_index];
  }
}

void SearchSeqFcCompute::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<float, float>);
}

void SearchSeqFcCompute::Run() {
  auto& param = this->Param<param_t>();
  CHECK(ctx_) << "running context should be set first";
  auto& cuda_ctx = ctx_->template As<HIPContext>();
  auto hip_stream = cuda_ctx.exec_stream();

  auto x = param.x;
  auto w = param.w;
  auto b = param.b;
  auto out = param.out;
  auto out_size = param.out_size;
  const auto x_dims = x->dims();
  const auto w_dims = w->dims();
  const auto out_dims = out->dims();
  CHECK_EQ(x_dims.size(), 2) << "The Input(X) should be 2-D tensor.";
  CHECK_EQ(w_dims.size(), 2) << "W should be 2-D tensor.";
  CHECK_EQ(out_dims.size(), 2) << "The Output(Out) should be 2-D tensor.";
  CHECK_EQ(x_dims[1], w_dims[1]) << "Wrong shape: x_dims[1] != w_dims[1]";
  CHECK_EQ(w_dims[0], out_size) << "Wrong shape: w_dims[0] != out_size";
  CHECK_EQ(out_dims[0], x_dims[0]) << "Wrong shape: out_dims[0] != x_dims[0]";
  CHECK_EQ(out_dims[1], out_size) << "Wrong shape: out_dims[1] != out_size";
  int M = x_dims[0];
  int K = x_dims[1];
  int N = w_dims[0];
  auto x_data = x->data<float>();
  auto w_data = w->data<float>();
  auto out_data = out->mutable_data<float>(TARGET(kCUDA));

  CHECK(gemm_impl_->init(false, true, M, N, K, &cuda_ctx));
  gemm_impl_->run(1.0f, 0.0f, x_data, w_data, out_data, &cuda_ctx);

  if (b != nullptr) {
    auto b_dims = b->dims();
    CHECK_EQ(b_dims.size(), 1) << "b should be 1-D tensor.";
    CHECK_EQ(b_dims[0], w_dims[0]) << "Wrong shape: b_dims[0] != w_dims[0]";
    auto b_data = b->mutable_data<float>();
    int total_size = M * N;
    add_bias<float><<<CUDA_GET_BLOCKS(total_size),
                      CUDA_NUM_THREADS,
                      0,
                      hip_stream>>>(total_size, N, b_data, out_data);
  }
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(search_seq_fc,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::SearchSeqFcCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
