/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <vector>
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/match_matrix_tensor_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {
using Tensor = lite::Tensor;

void MatchMatrixTensorCompute::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<float, float>);
}

void MatchMatrixTensorCompute::Run() {
  CHECK(ctx_) << "running context should be set first";
  auto& param = this->Param<param_t>();
  auto& context = this->ctx_->template As<HIPContext>();

  auto* x = param.x;
  auto* w = param.w;
  auto* y = param.y;
  auto* out = param.out;
  auto* tmp = param.tmp;
  int dim_t = param.dim_t;
  int dim_in = x->dims()[1];

  const auto& offset_l = x->lod()[0];
  const auto& offset_r = y->lod()[0];

  std::vector<size_t> top_offset;
  int top_size = 0;
  top_offset.push_back(top_size);
  for (size_t b = 0; b < x->lod()[0].size() - 1; b++) {
    int len_l = offset_l[b + 1] - offset_l[b];
    int len_r = offset_r[b + 1] - offset_r[b];
    top_size += dim_t * len_l * len_r;
    top_offset.push_back(top_size);
  }

  auto* bottom_l_data = x->data<float>();
  auto* bottom_r_data = y->data<float>();
  auto* t_data = w->data<float>();
  auto* out_data = out->mutable_data<float>(TARGET(kCUDA));
  auto* bottom_l_trans_data = tmp->mutable_data<float>(TARGET(kCUDA));

  gemm_impl_->init(
      false, false, x->dims()[0], dim_t * dim_in, dim_in, &context);
  gemm_impl_->run(
      1.0f, 0.0f, bottom_l_data, t_data, bottom_l_trans_data, &context);

  for (size_t b = 0; b < x->lod()[0].size() - 1; b++) {
    for (int t = 0; t < dim_t; t++) {
      int len_l = offset_l[b + 1] - offset_l[b];
      int len_r = offset_r[b + 1] - offset_r[b];
      auto* top_data = out_data + top_offset[b] + t * len_l * len_r;
      const auto* l_t_data =
          bottom_l_trans_data + offset_l[b] * dim_t * dim_in + t * dim_in;
      const auto* r_data = bottom_r_data + offset_r[b] * dim_in;

      gemm_impl_->init(false,
                       true,
                       len_l,
                       len_r,
                       dim_in,
                       dim_t * dim_in,
                       dim_in,
                       len_r,
                       &context);
      gemm_impl_->run(1.0f, 0.0f, l_t_data, r_data, top_data, &context);
    }
  }

  int batch_size = x->lod()[0].size() - 1;
  int lod_lv1_size = batch_size * dim_t;
  int lod_lv2_size = x->lod()[0].back() * dim_t;
  std::vector<size_t> out_lod0(batch_size + 1, 0);
  std::vector<size_t> out_lod1(lod_lv1_size + 1, 0);
  std::vector<size_t> out_lod2(lod_lv2_size + 1, 0);
  for (int i = 0; i < batch_size; i++) {
    out_lod0[i + 1] = out_lod0[i] + dim_t;
    int len_l = offset_l[i + 1] - offset_l[i];

    for (int j = 0; j < dim_t; j++) {
      out_lod1[i * dim_t + j + 1] = out_lod1[i * dim_t + j] + len_l;
      int len_r = offset_r[i + 1] - offset_r[i];

      for (int k = 0; k < len_l; k++) {
        out_lod2[offset_l[i] * dim_t + j * len_l + k + 1] =
            out_lod2[offset_l[i] * dim_t + j * len_l + k] + len_r;
      }
    }
  }

  LoD out_lod;
  out_lod.push_back(top_offset);
  out_lod.push_back(offset_l);
  out_lod.push_back(offset_r);
  out->set_lod(out_lod);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(match_matrix_tensor,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::MatchMatrixTensorCompute,
                     def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("W",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .BindOutput("Tmp",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();
