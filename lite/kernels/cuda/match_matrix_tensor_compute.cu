#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include <algorithm>
#include <vector>

#include "lite/backends/cuda/cuda_utils.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/match_matrix_tensor_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename dtype>
__global__ void ReorganizeOutput(const dtype* src,
                                 dtype* dst,
                                 const int count,
                                 const int len_l,
                                 const int len_r,
                                 const int dim_t) {
  CUDA_KERNEL_LOOP(tid, count) {
    int r_id = tid % len_r;
    int l_id = tid / len_r % len_l;
    int t_id = tid / len_r / len_l % dim_t;
    dst[tid] = src[r_id * len_l * dim_t + l_id * dim_t + t_id];
  }
}

template <typename dtype>
__global__ void ReorganizeOutputFuseRelu(const dtype* src,
                                         dtype* dst,
                                         const int count,
                                         const int len_l,
                                         const int len_r,
                                         const int dim_t) {
  CUDA_KERNEL_LOOP(tid, count) {
    int r_id = tid % len_r;
    int l_id = tid / len_r % len_l;
    int t_id = tid / len_r / len_l % dim_t;
    dst[tid] = src[r_id * len_l * dim_t + l_id * dim_t + t_id] > 0
                   ? src[r_id * len_l * dim_t + l_id * dim_t + t_id]
                   : 0;
  }
}

template <typename dtype>
__global__ void PaddingOutNotSameL(const dtype* src,
                                   const int* offset,
                                   const int seq_num_r,
                                   const int max_len_r,
                                   const int dim_t,
                                   const int len_l,
                                   const int count,
                                   dtype* dst) {
  CUDA_KERNEL_LOOP(tid, count) {
    int batch_id = tid / (dim_t * len_l * max_len_r);
    int r_id = tid % max_len_r;
    int l_id = tid / max_len_r % len_l;
    int t_id = tid / max_len_r / len_l % dim_t;

    int cur_len = offset[batch_id + 1] - offset[batch_id];
    if (r_id < cur_len) {
      dst[tid] = src[(offset[batch_id] + r_id) * (len_l * dim_t) + t_id +
                     l_id * dim_t];
    } else {
      dst[tid] = 0.f;
    }
  }
}

template <typename dtype>
__global__ void PaddingOutNotSameLFuseRelu(const dtype* src,
                                           const int* offset,
                                           const int seq_num_r,
                                           const int max_len_r,
                                           const int dim_t,
                                           const int len_l,
                                           const int count,
                                           dtype* dst) {
  CUDA_KERNEL_LOOP(tid, count) {
    int batch_id = tid / (dim_t * len_l * max_len_r);
    int r_id = tid % max_len_r;
    int l_id = tid / max_len_r % len_l;
    int t_id = tid / max_len_r / len_l % dim_t;

    int cur_len = offset[batch_id + 1] - offset[batch_id];
    if (r_id < cur_len) {
      dtype val = src[(offset[batch_id] + r_id) * (len_l * dim_t) +
                      l_id * dim_t + t_id];
      dst[tid] = val > 0.f ? val : 0.f;
    } else {
      dst[tid] = 0.f;
    }
  }
}

template <typename dtype>
__global__ void PaddingOut(const dtype* src,
                           const int* offset,
                           const int batch_r,
                           const int max_len_r,
                           const int tl,
                           const int count,
                           const bool fuse_relu,
                           dtype* dst) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_num = blockDim.x * gridDim.x;
  for (tid = threadIdx.x + blockIdx.x * blockDim.x; tid < count;
       tid += thread_num) {
    int seq_id = tid / (tl * max_len_r);
    int tl_id = (tid / (max_len_r)) % tl;
    int r_id = tid % max_len_r;
    int cur_len = offset[seq_id + 1] - offset[seq_id];
    if (r_id < cur_len) {
      if (fuse_relu) {
        dst[tid] = src[(offset[seq_id] + r_id) * tl + tl_id] > 0
                       ? src[(offset[seq_id] + r_id) * tl + tl_id]
                       : 0;
      } else {
        dst[tid] = src[(offset[seq_id] + r_id) * tl + tl_id];
      }
    } else {
      dst[tid] = 0.f;
    }
  }
}

template <typename T, PrecisionType PType>
void MatchMatrixTensorCompute<T, PType>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<T, T>);
}

template <typename T, PrecisionType PType>
void MatchMatrixTensorCompute<T, PType>::Run() {
  auto& param = this->template Param<param_t>();
  auto& context = this->ctx_->template As<HIPContext>();
  auto stream = context.exec_stream();

  auto* x = param.x;
  auto* w = param.w;
  auto* y = param.y;
  auto* out = param.out;

  int dim_t = param.dim_t;
  int dim_in = x->dims()[1];
  bool fuse_relu = param.fuse_relu;

  bool is_x_lod_same_len{true};
  bool is_l_same{true};

  const auto& offset_l = x->lod()[0];
  const auto& offset_r = y->lod()[0];
  std::vector<int> offset_r_int(offset_r.size());
  std::transform(offset_r.begin(),
                 offset_r.end(),
                 offset_r_int.begin(),
                 [](int64_t x) -> int { return static_cast<int>(x); });

  int batch = offset_r.size() - 1;
  int batch_l = x->dims()[0];
  int batch_r = offset_r[offset_r.size() - 1];
  int len_l = offset_l[1] - offset_l[0];
  int max_len_l = len_l;
  for (int i = 1; i < offset_l.size() - 1; i++) {
    int cur_len = offset_l[i + 1] - offset_l[i];
    if (cur_len != len_l) {
      is_x_lod_same_len = false;
      max_len_l = cur_len > max_len_l ? cur_len : max_len_l;
    }
  }
  int max_len_r = 0;
  for (int i = 0; i < offset_r.size() - 1; ++i) {
    int cur_len = offset_r[i + 1] - offset_r[i];
    max_len_r = cur_len > max_len_r ? cur_len : max_len_r;
  }

  const T* input_l = x->template data<T>();
  const T* r_data = y->template data<T>();
  const T* weight_data = w->template data<T>();

  // compare the mean value of each sequence to determine whether the input is
  // same.
  if (is_x_lod_same_len) {
    thrust::device_ptr<T> dev_ptr(const_cast<T*>(input_l));
    T seq_sum = thrust::reduce(dev_ptr,
                               dev_ptr + len_l * dim_in,
                               static_cast<T>(0),
                               thrust::plus<T>());
    T all_sum = thrust::reduce(dev_ptr,
                               dev_ptr + x->dims()[0] * dim_in,
                               static_cast<T>(0),
                               thrust::plus<T>());
    // TODO(wilber) consider for half.
    if (abs(all_sum / batch - seq_sum) > 1e-5) {
      is_l_same = false;
    }
  } else {
    is_l_same = false;
  }

  if (is_l_same) {
    input_l_transform_.Resize({1, dim_t, dim_in, len_l});
    input_l_transform_reorganize_.Resize({1, dim_t, len_l, dim_in});
    param.tmp->Resize({batch, max_len_r, dim_t, len_l});
    out->Resize({batch, dim_t, len_l, max_len_r});

    offset_r_.Resize({static_cast<int64_t>(offset_r.size())});
    TargetWrapperCuda::MemcpyAsync(offset_r_.mutable_data<int>(TARGET(kCUDA)),
                                   &offset_r_int[0],
                                   sizeof(int) * offset_r.size(),
                                   IoDirection::HtoD,
                                   stream);

    T* input_l_transform = input_l_transform_.mutable_data<T>(TARGET(kCUDA));
    T* input_l_transform_reorganize =
        input_l_transform_reorganize_.mutable_data<T>(TARGET(kCUDA));
    T* output_tmp = param.tmp->template mutable_data<T>(TARGET(kCUDA));
    T* out_data = out->template mutable_data<T>(TARGET(kCUDA));

    gemm_impl_->init(true, true, dim_t * dim_in, len_l, dim_in, &context);
    gemm_impl_->run(
        1.0f, 0.0f, weight_data, input_l, input_l_transform, &context);
    trans_.transpose(input_l_transform_reorganize,
                     input_l_transform,
                     input_l_transform_.dims().Vectorize(),
                     {0, 1, 3, 2},
                     &stream);

    gemm_impl_->init(false, true, batch_r, dim_t * len_l, dim_in, &context);
    gemm_impl_->run(
        1.0f, 0.0f, r_data, input_l_transform_reorganize, output_tmp, &context);
    int batch = offset_r.size() - 1;
    int count = batch * max_len_r * dim_t * len_l;
    const int blocks = 512;
    const int grids = (count + blocks - 1) / blocks;
    PaddingOut<T><<<grids, blocks, 0, stream>>>(param.tmp->template data<T>(),
                                                offset_r_.data<int>(),
                                                batch,
                                                max_len_r,
                                                dim_t * len_l,
                                                count,
                                                fuse_relu,
                                                out_data);
    CUDA_POST_KERNEL_CHECK;
    out->set_lod(y->lod());
  } else if (is_x_lod_same_len) {
    input_l_transform_.Resize({dim_t, dim_in, batch_l});
    input_l_transform_reorganize_.Resize({batch_l, dim_t, dim_in});
    param.tmp->Resize({batch_r, dim_t, len_l});
    out->Resize({batch, dim_t, len_l, max_len_r});

    offset_r_.Resize({static_cast<int64_t>(offset_r.size())});
    TargetWrapperCuda::MemcpyAsync(offset_r_.mutable_data<int>(TARGET(kCUDA)),
                                   &offset_r_int[0],
                                   sizeof(int) * offset_r.size(),
                                   IoDirection::HtoD,
                                   stream);

    T* input_l_transform = input_l_transform_.mutable_data<T>(TARGET(kCUDA));
    T* input_l_transform_reorganize =
        input_l_transform_reorganize_.mutable_data<T>(TARGET(kCUDA));
    T* output_tmp = param.tmp->template mutable_data<T>(TARGET(kCUDA));
    T* out_data = out->template mutable_data<T>(TARGET(kCUDA));

    gemm_impl_->init(true, true, dim_t * dim_in, batch_l, dim_in, &context);
    gemm_impl_->run(
        1.0f, 0.0f, weight_data, input_l, input_l_transform, &context);
    trans_.transpose(input_l_transform_reorganize,
                     input_l_transform,
                     input_l_transform_.dims().Vectorize(),
                     {2, 0, 1},
                     &stream);

    auto* tmp_out = output_tmp;
    for (int i = 0; i < batch; ++i) {
      int len_r = offset_r[i + 1] - offset_r[i];
      auto tmp_input_r = r_data + offset_r[i] * dim_in;
      auto tmp_input_l =
          input_l_transform_reorganize + i * len_l * dim_t * dim_in;
      gemm_impl_->init(false, true, len_r, len_l * dim_t, dim_in, &context);
      gemm_impl_->run(1.0f, 0.0f, tmp_input_r, tmp_input_l, tmp_out, &context);
      tmp_out += len_r * dim_t * len_l;
    }
    int count = batch * dim_t * len_l * max_len_r;
    if (fuse_relu) {
      PaddingOutNotSameLFuseRelu<
          T><<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          param.tmp->template data<T>(),
          offset_r_.data<int>(),
          batch,
          max_len_r,
          dim_t,
          len_l,
          count,
          out_data);
    } else {
      PaddingOutNotSameL<
          T><<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          param.tmp->template data<T>(),
          offset_r_.data<int>(),
          batch,
          max_len_r,
          dim_t,
          len_l,
          count,
          out_data);
    }
    CUDA_POST_KERNEL_CHECK;
    out->set_lod(y->lod());
  } else {
    // naive method.
    int out_len = 0;
    for (size_t i = 0; i < batch; i++) {
      int len_l = offset_l[i + 1] - offset_l[i];
      int len_r = offset_r[i + 1] - offset_r[i];
      out_len += len_l * len_r * dim_t;
    }
    out->Resize({out_len});
    T* out_data = out->template mutable_data<T>(TARGET(kCUDA));
    param.tmp->Resize({max_len_r, max_len_l, dim_t});
    T* out_tmp_data = param.tmp->template mutable_data<T>(TARGET(kCUDA));

    input_l_transform_.Resize({dim_t, dim_in, batch_l});
    input_l_transform_reorganize_.Resize({batch_l, dim_t, dim_in});
    T* input_l_transform = input_l_transform_.mutable_data<T>(TARGET(kCUDA));
    T* input_l_transform_reorganize =
        input_l_transform_reorganize_.mutable_data<T>(TARGET(kCUDA));

    std::vector<int> offset_l_int(offset_l.size());
    std::transform(offset_l.begin(),
                   offset_l.end(),
                   offset_l_int.begin(),
                   [](int64_t x) -> int { return static_cast<int>(x); });
    offset_l_.Resize({static_cast<int64_t>(offset_l.size())});
    TargetWrapperCuda::MemcpyAsync(offset_l_.mutable_data<int>(TARGET(kCUDA)),
                                   &offset_l_int[0],
                                   sizeof(int) * offset_l.size(),
                                   IoDirection::HtoD,
                                   stream);

    gemm_impl_->init(true, true, dim_t * dim_in, batch_l, dim_in, &context);
    gemm_impl_->run(
        1.0f, 0.0f, weight_data, input_l, input_l_transform, &context);
    trans_.transpose(input_l_transform_reorganize,
                     input_l_transform,
                     input_l_transform_.dims().Vectorize(),
                     {2, 0, 1},
                     &stream);

    auto* l_data = input_l_transform_reorganize;

    for (size_t i = 0; i < batch; ++i) {
      int len_l = offset_l[i + 1] - offset_l[i];
      int len_r = offset_r[i + 1] - offset_r[i];
      int out_seq_count = len_r * len_l * dim_t;
      gemm_impl_->init(false, true, len_r, len_l * dim_t, dim_in, &context);
      gemm_impl_->run(1.0f,
                      0.0f,
                      r_data,
                      l_data,
                      param.tmp->template mutable_data<T>(TARGET(kCUDA)),
                      &context);
      if (fuse_relu) {
        ReorganizeOutputFuseRelu<
            T><<<CUDA_GET_BLOCKS(out_seq_count), CUDA_NUM_THREADS, 0, stream>>>(
            param.tmp->template data<T>(),
            out_data,
            out_seq_count,
            len_l,
            len_r,
            dim_t);
      } else {
        ReorganizeOutput<
            T><<<CUDA_GET_BLOCKS(out_seq_count), CUDA_NUM_THREADS, 0, stream>>>(
            param.tmp->template data<T>(),
            out_data,
            out_seq_count,
            len_l,
            len_r,
            dim_t);
      }
      r_data += len_r * dim_in;
      l_data += dim_t * len_l * dim_in;
      out_data += out_seq_count;
    }
    CUDA_POST_KERNEL_CHECK;

    std::vector<std::vector<uint64_t>> lod;
    lod.push_back(x->lod()[0]);
    lod.push_back(y->lod()[0]);
    lod.push_back({dim_t});
    out->set_lod(lod);
  }
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

using MMTFp32 =
    paddle::lite::kernels::cuda::MatchMatrixTensorCompute<float,
                                                          PRECISION(kFloat)>;

REGISTER_LITE_KERNEL(match_matrix_tensor, kCUDA, kFloat, kNCHW, MMTFp32, def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("W",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .BindOutput("Tmp",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();
