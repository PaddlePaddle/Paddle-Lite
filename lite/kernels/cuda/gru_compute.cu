// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <string>

#include "lite/backends/cuda/cuda_utils.h"
#include "lite/backends/cuda/math/bias.h"
#include "lite/backends/cuda/math/gru_forward.h"
#include "lite/backends/cuda/math/sequence2batch.h"
#include "lite/backends/cuda/target_wrapper.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/gru_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename T>
struct GRUMetaValue {
  T* gate_weight;
  T* state_weight;
  T* gate_value;
  T* reset_output_value;
  T* output_value;
  T* prev_out_value;
};

template <typename T>
struct GRUUnitFunctor {
  static void compute(GRUMetaValue<T> value,
                      int frame_size,
                      int batch_size,
                      const lite::cuda::math::ActivationType& active_node,
                      const lite::cuda::math::ActivationType& active_gate,
                      bool origin_mode,
                      lite::cuda::math::Gemm<T, T>* blas,
                      HIPContext* context) {
    dim3 threads, grids;
    if (batch_size == 1) {
      int frame_per_block = frame_size <= 1024 ? frame_size : 1024;
      int frame_blocks = (frame_size + 1024 - 1) / 1024;
      threads = dim3(frame_per_block, 1);
      grids = dim3(frame_blocks, 1);
    } else {
      threads = dim3(32, 32);
      grids = dim3((frame_size + 32 - 1) / 32, (batch_size + 32 - 1) / 32);
    }

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size * 2,
                       frame_size,
                       frame_size,
                       frame_size * 2,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.prev_out_value,
                value.gate_weight,
                value.gate_value,
                context);
    }
    CUDA_POST_KERNEL_CHECK;

    lite::cuda::math::GruForwardResetOutput<
        T><<<grids, threads, 0, context->exec_stream()>>>(
        value.gate_value,
        value.reset_output_value,
        value.prev_out_value,
        frame_size,
        batch_size,
        active_gate,
        batch_size == 1);
    CUDA_POST_KERNEL_CHECK;

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.reset_output_value,
                value.state_weight,
                value.gate_value + frame_size * 2,
                context);
    }
    CUDA_POST_KERNEL_CHECK;

    lite::cuda::math::GruForwardFinalOutput<
        T><<<grids, threads, 0, context->exec_stream()>>>(value.gate_value,
                                                          value.prev_out_value,
                                                          value.output_value,
                                                          frame_size,
                                                          batch_size,
                                                          active_node,
                                                          origin_mode,
                                                          batch_size == 1);
    CUDA_POST_KERNEL_CHECK;
  }
};

template struct GRUUnitFunctor<float>;

template <typename T, PrecisionType PType>
void GRUCompute<T, PType>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<T, T>);
}

template <typename T, PrecisionType PType>
void GRUCompute<T, PType>::Run() {
  auto& context = this->ctx_->template As<HIPContext>();
  auto stream = context.exec_stream();
  auto& param = this->template Param<param_t>();

  auto* input = param.input;
  lite::Tensor* h0{nullptr};
  if (param.h0) {
    h0 = const_cast<lite::Tensor*>(param.h0);
  }
  lite::Tensor* bias{nullptr};
  if (param.bias) {
    bias = const_cast<lite::Tensor*>(param.bias);
  }
  auto* weight = param.weight;
  auto* weight_data = const_cast<T*>(weight->template data<T>());
  auto* batch_gate = param.batch_gate;
  auto* batch_reset_hidden_prev = param.batch_reset_hidden_prev;
  auto* batch_hidden = param.batch_hidden;
  auto* hidden = param.hidden;
  auto* batch_reset_hidden_prev_data =
      batch_reset_hidden_prev->template mutable_data<T>(TARGET(kCUDA));
  hidden->template mutable_data<T>(TARGET(kCUDA));
  auto* batch_gate_data = batch_gate->template mutable_data<T>(TARGET(kCUDA));
  auto* batch_hidden_data =
      batch_hidden->template mutable_data<T>(TARGET(kCUDA));
  bool is_reverse = param.is_reverse;
  auto active_node = lite::cuda::math::GetActiveType(param.activation);
  auto active_gate = lite::cuda::math::GetActiveType(param.gate_activation);
  bool origin_mode = param.origin_mode;

  auto hidden_dims = hidden->dims();
  int frame_size = hidden_dims[1];

  lite::cuda::math::LoDTensor2BatchFunctor<T> batch_func;
  batch_func(*input, batch_gate, is_reverse, stream);

  if (bias) {
    lite::cuda::math::RowwiseAdd<T> add_bias;
    add_bias(batch_gate_data,
             bias->template data<T>(),
             batch_gate_data,
             frame_size,
             batch_gate->numel(),
             stream);
  }
  GRUMetaValue<T> gru_value;
  gru_value.gate_weight = weight_data;
  gru_value.state_weight = weight_data + 2 * frame_size * frame_size;

  if (h0) {
    // Since the batch computing for GRU reorders the input sequences
    // according to their length. The initialized cell state also needs
    // to reorder.
    ordered_h0_.Resize(h0->dims());
    lite::cuda::math::CopyMatrixRowsFunctor<T> row_shuffle;
    row_shuffle(*h0, &ordered_h0_, batch_gate->lod()[2], true, stream);
    gru_value.prev_out_value = ordered_h0_.mutable_data<T>(TARGET(kCUDA));
  } else {
    gru_value.prev_out_value = nullptr;
  }
  auto batch_starts = batch_gate->lod()[0];
  size_t num_batch = batch_starts.size() - 1;
  for (size_t n = 0; n < num_batch; ++n) {
    int bstart = static_cast<int>(batch_starts[n]);
    int bend = static_cast<int>(batch_starts[n + 1]);
    int cur_batch_size = bend - bstart;

    gru_value.output_value = batch_hidden_data + bstart * frame_size;
    gru_value.gate_value = batch_gate_data + bstart * frame_size * 3;
    gru_value.reset_output_value =
        batch_reset_hidden_prev_data + bstart * frame_size;

    GRUUnitFunctor<T>::compute(gru_value,
                               frame_size,
                               cur_batch_size,
                               active_node,
                               active_gate,
                               origin_mode,
                               gemm_impl_.get(),
                               &context);
    gru_value.prev_out_value = gru_value.output_value;
  }

  lite::cuda::math::Batch2LoDTensorFunctor<T> to_seq;
  batch_hidden->set_lod(batch_gate->lod());
  to_seq(*batch_hidden, hidden, stream);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

using GRUFp32 =
    paddle::lite::kernels::cuda::GRUCompute<float, PRECISION(kFloat)>;

REGISTER_LITE_KERNEL(gru, kCUDA, kFloat, kNCHW, GRUFp32, def)
    .BindInput("Input", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("H0", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Weight", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Bias", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchGate", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchResetHiddenPrev", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchHidden", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Hidden", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
