// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "lite/kernels/cuda/gru_compute.h"

#include <string>
#include <vector>

#include "lite/backends/cuda/cuda_utils.h"
#include "lite/backends/cuda/math/bias.h"
#include "lite/backends/cuda/math/gru_forward.h"
#include "lite/backends/cuda/math/sequence2batch.h"
#include "lite/backends/cuda/target_wrapper.h"
#include "lite/core/op_registry.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename T>
struct GRUMetaValue {
  T* gate_weight;
  T* state_weight;
  T* gate_value;
  T* reset_output_value;
  T* output_value;
  T* prev_out_value;
};

template <typename T>
struct GRUUnitFunctor {
  static void compute(GRUMetaValue<T> value,
                      int frame_size,
                      int batch_size,
                      const lite::cuda::math::ActivationType& active_node,
                      const lite::cuda::math::ActivationType& active_gate,
                      bool origin_mode,
                      lite::cuda::math::Gemm<T, T>* blas,
                      HIPContext* context) {
    dim3 threads, grids;
    if (batch_size == 1) {
      if (lite::TargetWrapperCuda::GetComputeCapability() >= 70) {
        if (frame_size < 16) {
          constexpr int tiled_size = 8;
          int frame_blocks = (frame_size * 2 + tiled_size - 1) / tiled_size;
          threads = dim3(tiled_size, 1);
          grids = dim3(frame_blocks, 1);
          lite::cuda::math::FastCollectiveGruGate<
              T,
              tiled_size><<<grids, threads, 0, context->exec_stream()>>>(
              value.gate_value,
              value.prev_out_value,
              value.gate_weight,
              value.reset_output_value,
              frame_size,
              active_gate);
          frame_blocks = (frame_size + tiled_size - 1) / tiled_size;
          grids = dim3(frame_blocks, 1);
          lite::cuda::math::FastCollectiveGruOut<
              T,
              tiled_size><<<grids, threads, 0, context->exec_stream()>>>(
              value.state_weight,
              value.prev_out_value,
              value.output_value,
              value.gate_value,
              value.reset_output_value,
              frame_size,
              active_node,
              origin_mode);
        } else {
          constexpr int tiled_size = 16;
          int frame_blocks = (frame_size * 2 + tiled_size - 1) / tiled_size;
          threads = dim3(tiled_size, 1);
          grids = dim3(frame_blocks, 1);
          lite::cuda::math::FastCollectiveGruGate<
              T,
              tiled_size><<<grids, threads, 0, context->exec_stream()>>>(
              value.gate_value,
              value.prev_out_value,
              value.gate_weight,
              value.reset_output_value,
              frame_size,
              active_gate);
          frame_blocks = (frame_size + tiled_size - 1) / tiled_size;
          grids = dim3(frame_blocks, 1);
          lite::cuda::math::FastCollectiveGruOut<
              T,
              tiled_size><<<grids, threads, 0, context->exec_stream()>>>(
              value.state_weight,
              value.prev_out_value,
              value.output_value,
              value.gate_value,
              value.reset_output_value,
              frame_size,
              active_node,
              origin_mode);
        }
        return;
      } else {
        int frame_per_block = frame_size <= 1024 ? frame_size : 1024;
        int frame_blocks = (frame_size + 1024 - 1) / 1024;
        threads = dim3(frame_per_block, 1);
        grids = dim3(frame_blocks, 1);
      }
    } else {
      threads = dim3(32, 32);
      grids = dim3((frame_size + 32 - 1) / 32, (batch_size + 32 - 1) / 32);
    }

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size * 2,
                       frame_size,
                       frame_size,
                       frame_size * 2,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.prev_out_value,
                value.gate_weight,
                value.gate_value,
                context);
    }

    lite::cuda::math::GruForwardResetOutput<
        T><<<grids, threads, 0, context->exec_stream()>>>(
        value.gate_value,
        value.reset_output_value,
        value.prev_out_value,
        frame_size,
        batch_size,
        active_gate,
        batch_size != 1);
    CUDA_POST_KERNEL_CHECK;

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.reset_output_value,
                value.state_weight,
                value.gate_value + frame_size * 2,
                context);
    }

    lite::cuda::math::GruForwardFinalOutput<
        T><<<grids, threads, 0, context->exec_stream()>>>(value.gate_value,
                                                          value.prev_out_value,
                                                          value.output_value,
                                                          frame_size,
                                                          batch_size,
                                                          active_node,
                                                          origin_mode,
                                                          batch_size != 1);
    CUDA_POST_KERNEL_CHECK;
  }
};

template struct GRUUnitFunctor<float>;

template <>
struct GRUUnitFunctor<half> {
  static void compute(GRUMetaValue<half> value,
                      int frame_size,
                      int batch_size,
                      const lite::cuda::math::ActivationType& active_node,
                      const lite::cuda::math::ActivationType& active_gate,
                      bool origin_mode,
                      lite::cuda::math::Gemm<half, half>* blas,
                      HIPContext* context) {
    dim3 threads, grids;
    if (batch_size == 1) {
      int frame_per_block = frame_size <= 1024 ? frame_size : 1024;
      int frame_blocks = (frame_size + 1024 - 1) / 1024;
      threads = dim3(frame_per_block, 1);
      grids = dim3(frame_blocks, 1);
    } else {
      threads = dim3(32, 32);
      grids = dim3((frame_size + 32 - 1) / 32, (batch_size + 32 - 1) / 32);
    }

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size * 2,
                       frame_size,
                       frame_size,
                       frame_size * 2,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.prev_out_value,
                value.gate_weight,
                value.gate_value,
                context);
    }

    lite::cuda::math::GruForwardResetOutput<
        half><<<grids, threads, 0, context->exec_stream()>>>(
        value.gate_value,
        value.reset_output_value,
        value.prev_out_value,
        frame_size,
        batch_size,
        active_gate,
        batch_size == 1);
    CUDA_POST_KERNEL_CHECK;

    if (value.prev_out_value) {
      CHECK(blas->init(false,
                       false,
                       batch_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size,
                       frame_size * 3,
                       context));
      blas->run(1.0f,
                1.0f,
                value.reset_output_value,
                value.state_weight,
                value.gate_value + frame_size * 2,
                context);
    }

    lite::cuda::math::GruForwardFinalOutput<
        half><<<grids, threads, 0, context->exec_stream()>>>(
        value.gate_value,
        value.prev_out_value,
        value.output_value,
        frame_size,
        batch_size,
        active_node,
        origin_mode,
        batch_size == 1);
    CUDA_POST_KERNEL_CHECK;
  }
};

template <typename T, PrecisionType PType>
void GRUCompute<T, PType>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<T, T>);
}

template <typename T, PrecisionType PType>
void GRUCompute<T, PType>::Run() {
  auto& context = this->ctx_->template As<HIPContext>();
  auto stream = context.exec_stream();
  auto& param = this->template Param<param_t>();

  auto* input = param.input;
  T* x_data =
      const_cast<lite::Tensor*>(input)->template mutable_data<T>(TARGET(kCUDA));
  lite::Tensor* h0{nullptr};
  if (param.h0) {
    h0 = const_cast<lite::Tensor*>(param.h0);
  }
  lite::Tensor* bias{nullptr};
  if (param.bias) {
    bias = const_cast<lite::Tensor*>(param.bias);
  }
  const lite::Tensor* weight = param.weight;
  T* weight_data = const_cast<T*>(weight->template data<T>());
  lite::Tensor* batch_gate = param.batch_gate;
  lite::Tensor* batch_reset_hidden_prev = param.batch_reset_hidden_prev;
  lite::Tensor* batch_hidden = param.batch_hidden;
  lite::Tensor* hidden = param.hidden;
  T* batch_reset_hidden_prev_data =
      batch_reset_hidden_prev->template mutable_data<T>(TARGET(kCUDA));
  T* out_data = hidden->template mutable_data<T>(TARGET(kCUDA));
  T* batch_gate_data = batch_gate->template mutable_data<T>(TARGET(kCUDA));
  T* batch_hidden_data = batch_hidden->template mutable_data<T>(TARGET(kCUDA));
  bool is_reverse = param.is_reverse;
  auto active_node = lite::cuda::math::GetActiveType(param.activation);
  auto active_gate = lite::cuda::math::GetActiveType(param.gate_activation);
  bool origin_mode = param.origin_mode;

  auto hidden_dims = hidden->dims();
  int frame_size = hidden_dims[1];

  LoD offset_vec_vec = input->lod();
  std::vector<int> offset(offset_vec_vec[offset_vec_vec.size() - 1].size());
  for (size_t i = 0; i < offset_vec_vec[offset_vec_vec.size() - 1].size();
       ++i) {
    offset[i] = static_cast<int>(offset_vec_vec[offset_vec_vec.size() - 1][i]);
  }
  bool need_process = seq_utils_.GetSortedMap(offset, stream);
  int emit_length = seq_utils_.GetEmitOffsetVec().size() - 1;
  auto emit_offset_vec = seq_utils_.GetEmitOffsetVec();
  if (need_process) {
    seq_utils_.Seq2SortedSeq(
        input->template data<T>(), batch_gate_data, 3 * frame_size, stream);
    x_data = batch_gate_data;
    out_data = batch_hidden_data;
  }

  if (bias) {
    // TODO(wilber): validate when bias is not nullptr
    lite::cuda::math::RowwiseAdd<T> add_bias;
    add_bias(x_data,
             bias->template data<T>(),
             x_data,
             frame_size,
             batch_gate->numel(),
             stream);
  }
  GRUMetaValue<T> gru_value;
  gru_value.gate_weight = weight_data;
  gru_value.state_weight = weight_data + 2 * frame_size * frame_size;

  if (h0) {
    // Since the batch computing for GRU reorders the input sequences
    // according to their length. The initialized cell state also needs
    // to reorder.
    // TODO(wilber): validate when h0 is not nullptr
    ordered_h0_.Resize(h0->dims());
    lite::cuda::math::CopyMatrixRowsFunctor<T> row_shuffle;
    row_shuffle(*h0, &ordered_h0_, batch_gate->lod()[2], true, stream);
    gru_value.prev_out_value = ordered_h0_.mutable_data<T>(TARGET(kCUDA));
  } else {
    gru_value.prev_out_value = nullptr;
  }
  for (size_t n = 0; n < emit_length; ++n) {
    int bstart = emit_offset_vec[n];
    int bend = emit_offset_vec[n + 1];
    int cur_batch_size = bend - bstart;

    gru_value.output_value = out_data + bstart * frame_size;
    gru_value.gate_value = x_data + bstart * frame_size * 3;
    gru_value.reset_output_value =
        batch_reset_hidden_prev_data + bstart * frame_size;

    GRUUnitFunctor<T>::compute(gru_value,
                               frame_size,
                               cur_batch_size,
                               active_node,
                               active_gate,
                               origin_mode,
                               gemm_impl_.get(),
                               &context);
    gru_value.prev_out_value = gru_value.output_value;
  }
  if (need_process) {
    seq_utils_.SortedSeq2Seq(batch_hidden_data,
                             hidden->mutable_data<T>(TARGET(kCUDA)),
                             frame_size,
                             stream);
  }
  hidden->set_lod(input->lod());
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

using GRUFp32 =
    paddle::lite::kernels::cuda::GRUCompute<float, PRECISION(kFloat)>;

using GRUFp16 = paddle::lite::kernels::cuda::GRUCompute<half, PRECISION(kFP16)>;

REGISTER_LITE_KERNEL(gru, kCUDA, kFloat, kNCHW, GRUFp32, def)
    .BindInput("Input", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("H0", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Weight", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Bias", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchGate", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchResetHiddenPrev", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("BatchHidden", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Hidden", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(gru, kCUDA, kFP16, kNCHW, GRUFp16, def)
    .BindInput("Input",
               {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("H0", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("Weight",
               {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("Bias", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindOutput("BatchGate",
                {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindOutput("BatchResetHiddenPrev",
                {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindOutput("BatchHidden",
                {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindOutput("Hidden",
                {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .Finalize();
