#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_fc_compute.h"
namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {
template <typename T>
static void anakin_NV_gemv(hipblasHandle_t handle,
                           const bool TransA,
                           const int M,
                           const int N,
                           const T alpha,
                           const T* A,
                           const T* x,
                           const T beta,
                           T* y);
template <>
void anakin_NV_gemv<float>(hipblasHandle_t handle,
                           const bool TransA,
                           const int M,
                           const int N,
                           const float alpha,
                           const float* A,
                           const float* x,
                           const float beta,
                           float* y) {
  LOG(INFO) << "1";
  hipblasOperation_t cuTransA = (TransA == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(
      hipblasSgemv(handle, cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1));
}
template <typename T>
static void anakin_NV_gemm(hipblasHandle_t handle,
                           const bool TransA,
                           const bool TransB,
                           const int M,
                           const int N,
                           const int K,
                           const T alpha,
                           const T* A,
                           const T* B,
                           const T beta,
                           T* C);

template <>
void anakin_NV_gemm<float>(hipblasHandle_t handle,
                           const bool TransA,
                           const bool TransB,
                           const int M,
                           const int N,
                           const int K,
                           const float alpha,
                           const float* A,
                           const float* B,
                           const float beta,
                           float* C) {
  LOG(INFO) << "1";
  // Note that cublas follows fortran order.
  int lda = (!TransA /* == CblasNoTrans*/) ? K : M;
  int ldb = (!TransB /* == CblasNoTrans*/) ? N : K;
  LOG(INFO) << "1";
  hipblasOperation_t cuTransA =
      (!TransA /* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  LOG(INFO) << "1";
  hipblasOperation_t cuTransB =
      (!TransB /* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  LOG(INFO) << "1";
  CUBLAS_CHECK(hipblasSgemm(handle,
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           &alpha,
                           B,
                           ldb,
                           A,
                           lda,
                           &beta,
                           C,
                           N));
  LOG(INFO) << "1";
}

template <>
void anakin_NV_gemm<char>(hipblasHandle_t handle,
                          const bool TransA,
                          const bool TransB,
                          const int M,
                          const int N,
                          const int K,
                          const char alpha,
                          const char* A,
                          const char* B,
                          const char beta,
                          char* C) {
  LOG(FATAL) << "int8 gemm is not implemented";
}

template <typename T>
static __global__ void add_bias(int n,
                                int output_size,
                                const T* bias,
                                T* dout) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int bias_index = index % output_size;
  if (index < n) {
    dout[index] = dout[index] + bias[bias_index];
  }
}

template <typename T>
void SearchFcCompute<T>::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();
  const Tensor* x_tensor = param.X;
  param.Out->Resize({x_tensor->dims()[0], param.out_size});
  _M = x_tensor->dims().count(0, 1);
  _K = x_tensor->dims().count(1, x_tensor->numel());
  _N = param.out_size;
  const T* din = x_tensor->data<T>();
  Tensor* out_tensor = param.Out;
  T* dout = out_tensor->mutable_data<T>(TARGET(kCUDA));
  const Tensor* w_tensor = param.W;
  const T* weight = w_tensor->data<T>();
  const Tensor* b_tensor = param.b;
  const T* bias = b_tensor->data<T>();
  hipblasCreate(&_handle);
  if (_M == 1 && _K > 50000) {
    anakin_NV_gemv<T>(_handle, false, _N, _K, (T)1, weight, din, (T)0, dout);
  } else {
    anakin_NV_gemm<T>(_handle,
                      false,
                      !_flag_trans_weights,
                      _M,
                      _N,
                      _K,
                      (T)1,
                      din,
                      weight,
                      (T)0,
                      dout);
  }
  int total_size = _M * _N;
  add_bias<T><<<CUDA_GET_BLOCKS(total_size), CUDA_NUM_THREADS, 0, stream>>>(
      total_size, _N, bias, dout);
}
}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(search_fc,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::SearchFcCompute<float>,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
