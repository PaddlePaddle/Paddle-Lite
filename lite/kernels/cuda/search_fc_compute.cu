#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_fc_compute.h"
namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename T>
static __global__ void add_bias(int n,
                                int output_size,
                                const T* bias,
                                T* dout) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int bias_index = index % output_size;
  if (index < n) {
    dout[index] = dout[index] + bias[bias_index];
  }
}

template <typename T>
void SearchFcCompute<T>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<float, float>);
}

template <typename T>
void SearchFcCompute<T>::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();
  const Tensor* x_tensor = param.X;
  param.Out->Resize({x_tensor->dims()[0], param.out_size});
  _M = x_tensor->dims().count(0, 1);
  _K = x_tensor->dims().count(1, x_tensor->numel());
  _N = param.out_size;
  const T* din = x_tensor->data<T>();
  Tensor* out_tensor = param.Out;
  T* dout = out_tensor->mutable_data<T>(TARGET(kCUDA));
  const Tensor* w_tensor = param.W;
  const T* weight = w_tensor->data<T>();
  const Tensor* b_tensor = param.b;
  const T* bias = b_tensor->data<T>();

  CHECK(gemm_impl_->init(false, true, _M, _N, _K, &ctx));
  gemm_impl_->run(1.0f, 0.0f, din, weight, dout, &ctx);

  int total_size = _M * _N;
  add_bias<T><<<CUDA_GET_BLOCKS(total_size), CUDA_NUM_THREADS, 0, stream>>>(
      total_size, _N, bias, dout);
}
}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(search_fc,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::SearchFcCompute<float>,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
