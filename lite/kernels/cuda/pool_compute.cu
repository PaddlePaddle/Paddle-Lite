#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <vector>
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/pool_compute.h"
#include "lite/utils/macros.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {
using Tensor = lite::Tensor;
using DDim = lite::DDim;

#define MAX_VAL(a, b) (((a) > (b)) ? (a) : (b))
#define MIN_VAL(a, b) (((a) < (b)) ? (a) : (b))

__global__ void max_pool_kernel(const float* input,
                                float* output,
                                const int spatial_in,
                                const int spatial_out,
                                const int in_h,
                                const int in_w,
                                const int out_h,
                                const int out_w,
                                const int pad_h,
                                const int pad_w,
                                const int win_h,
                                const int win_w,
                                const int stride_h,
                                const int stride_w,
                                const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int nc_id = gid / spatial_out;
    const int w_id = gid % spatial_out % out_w;
    const int h_id = gid % spatial_out / out_w;
    const int w_s = w_id * stride_w - pad_w;
    const int iw_s = MAX_VAL(w_s, 0);
    const int iw_e = MIN_VAL(w_s + win_w, in_w);
    const int w_loop = iw_e - iw_s;
    const int h_s = h_id * stride_h - pad_h;
    const int ih_s = MAX_VAL(h_s, 0);
    const int ih_e = MIN_VAL(h_s + win_h, in_h);
    const int h_loop = ih_e - ih_s;
    const float* in_p = input + nc_id * spatial_in + ih_s * in_w + iw_s;
    float max_val = -FLT_MAX;
    for (int i = 0; i < h_loop; ++i) {
      for (int j = 0; j < w_loop; ++j) {
        max_val = MAX_VAL(max_val, *(in_p + j));
      }
      in_p += in_w;
    }
    max_val = max_val == -FLT_MAX ? 0.f : max_val;
    output[nc_id * spatial_out + h_id * out_w + w_id] = max_val;
  }
}

__global__ void adaptive_max_pool_kernel(const float* input,
                                         float* output,
                                         const int spatial_in,
                                         const int spatial_out,
                                         const int in_h,
                                         const int in_w,
                                         const int out_h,
                                         const int out_w,
                                         const int pad_h,
                                         const int pad_w,
                                         const int win_h,
                                         const int win_w,
                                         const int stride_h,
                                         const int stride_w,
                                         const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int nc_id = gid / spatial_out;
    const int w_id = gid % spatial_out % out_w;
    const int h_id = gid % spatial_out / out_w;
    const int iw_s = floor(static_cast<double>(w_id * in_w) / out_w);
    const int iw_e = ceil(static_cast<double>((w_id + 1) * in_w) / out_w);
    const int w_loop = iw_e - iw_s;
    const int ih_s = floor(static_cast<double>(h_id * in_h) / out_h);
    const int ih_e = ceil(static_cast<double>((h_id + 1) * in_h) / out_h);
    const int h_loop = ih_e - ih_s;
    const float* in_p = input + nc_id * spatial_in + ih_s * in_w + iw_s;
    float max_val = -FLT_MAX;
    for (int i = 0; i < h_loop; ++i) {
      for (int j = 0; j < w_loop; ++j) {
        max_val = MAX_VAL(max_val, *(in_p + j));
      }
      in_p += in_w;
    }
    output[nc_id * spatial_out + h_id * out_w + w_id] = max_val;
  }
}

__global__ void avg_pool_kernel(const float* input,
                                float* output,
                                const int spatial_in,
                                const int spatial_out,
                                const int in_h,
                                const int in_w,
                                const int out_h,
                                const int out_w,
                                const int pad_h,
                                const int pad_w,
                                const int win_h,
                                const int win_w,
                                const int stride_h,
                                const int stride_w,
                                bool exclusive,
                                const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int nc_id = gid / spatial_out;
    const int w_id = gid % spatial_out % out_w;
    const int h_id = gid % spatial_out / out_w;
    const int w_s = w_id * stride_w - pad_w;
    const int iw_s = MAX_VAL(w_s, 0);
    const int iw_e = MIN_VAL(w_s + win_w, in_w);
    const int w_loop = iw_e - iw_s;
    const int h_s = h_id * stride_h - pad_h;
    const int ih_s = MAX_VAL(h_s, 0);
    const int ih_e = MIN_VAL(h_s + win_h, in_h);
    const int h_loop = ih_e - ih_s;
    const float* in_p = input + nc_id * spatial_in + ih_s * in_w + iw_s;
    float sum_val = 0.f;
    for (int i = 0; i < h_loop; ++i) {
      for (int j = 0; j < w_loop; ++j) {
        sum_val += *(in_p + j);
      }
      in_p += in_w;
    }
    int pool_size = exclusive ? h_loop * w_loop : win_w * win_h;
    pool_size = pool_size == 0 ? 1 : pool_size;
    output[nc_id * spatial_out + h_id * out_w + w_id] = sum_val / pool_size;
  }
}

__global__ void adaptive_avg_pool_kernel(const float* input,
                                         float* output,
                                         const int spatial_in,
                                         const int spatial_out,
                                         const int in_h,
                                         const int in_w,
                                         const int out_h,
                                         const int out_w,
                                         const int pad_h,
                                         const int pad_w,
                                         const int win_h,
                                         const int win_w,
                                         const int stride_h,
                                         const int stride_w,
                                         const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int nc_id = gid / spatial_out;
    const int w_id = gid % spatial_out % out_w;
    const int h_id = gid % spatial_out / out_w;
    const int iw_s = floor(static_cast<double>(w_id * in_w) / out_w);
    const int iw_e = ceil(static_cast<double>((w_id + 1) * in_w) / out_w);
    const int w_loop = iw_e - iw_s;
    const int ih_s = floor(static_cast<double>(h_id * in_h) / out_h);
    const int ih_e = ceil(static_cast<double>((h_id + 1) * in_h) / out_h);
    const int h_loop = ih_e - ih_s;
    const float* in_p = input + nc_id * spatial_in + ih_s * in_w + iw_s;
    float sum_val = 0.f;
    for (int i = 0; i < h_loop; ++i) {
      for (int j = 0; j < w_loop; ++j) {
        sum_val += *(in_p + j);
      }
      in_p += in_w;
    }
    int pool_size = h_loop * w_loop;
    pool_size = pool_size == 0 ? 1 : pool_size;
    output[nc_id * spatial_out + h_id * out_w + w_id] = sum_val / pool_size;
  }
}

__global__ void global_max_pool_kernel(const float* input,
                                       float* output,
                                       const int in_h,
                                       const int in_w,
                                       const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int spatial_in = in_h * in_w;
    const float* in_p = input + gid * spatial_in;
    int i = 0;
    float max_val = -0.f;
    // unroll 8
    for (; i < spatial_in - 7; i += 8) {
      max_val = MAX_VAL(max_val, *(in_p + 0));
      max_val = MAX_VAL(max_val, *(in_p + 1));
      max_val = MAX_VAL(max_val, *(in_p + 2));
      max_val = MAX_VAL(max_val, *(in_p + 3));
      max_val = MAX_VAL(max_val, *(in_p + 4));
      max_val = MAX_VAL(max_val, *(in_p + 5));
      max_val = MAX_VAL(max_val, *(in_p + 6));
      max_val = MAX_VAL(max_val, *(in_p + 7));
      in_p += 8;
    }
    for (; i < spatial_in; i++) {
      max_val = MAX_VAL(max_val, *in_p);
      in_p++;
    }
    output[gid] = max_val;
  }
}

__global__ void global_avg_pool_kernel(const float* input,
                                       float* output,
                                       const int in_h,
                                       const int in_w,
                                       const int total_threads) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < total_threads) {
    const int spatial_in = in_h * in_w;
    const float* in_p = input + gid * spatial_in;
    int i = 0;
    float sum_val = 0.f;
    // unroll 8
    for (; i < spatial_in - 7; i += 8) {
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
      sum_val += *in_p++;
    }
    for (; i < spatial_in; i++) {
      sum_val += *in_p++;
    }
    output[gid] = sum_val / spatial_in;
  }
}

void PoolCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();

  bool exclusive = param.exclusive;
  bool adaptive = param.adaptive;
  auto x_dims = param.x->dims();
  auto out_dims = param.output->dims();
  auto paddings = *param.paddings;
  const int in_h = x_dims[2];
  const int in_w = x_dims[3];
  const int out_h = out_dims[2];
  const int out_w = out_dims[3];
  const int spatial_in = in_h * in_w;
  const int spatial_out = out_h * out_w;
  const int win_h = param.ksize[0];
  const int win_w = param.ksize[1];
  const int stride_h = param.strides[0];
  const int stride_w = param.strides[1];
  const int pad_h = paddings[0];
  const int pad_w = paddings[2];
  const int total_threads = out_dims.production();
  const int threads = 512;
  const int blocks = (total_threads + threads - 1) / threads;
  auto input_data = param.x->data<float>();
  auto output_data = param.output->mutable_data<float>(TARGET(kCUDA));
  if (param.global_pooling) {
    if (param.pooling_type == "max") {
      global_max_pool_kernel<<<blocks, threads, 0, stream>>>(
          input_data, output_data, in_h, in_w, total_threads);
    } else {
      global_avg_pool_kernel<<<blocks, threads, 0, stream>>>(
          input_data, output_data, in_h, in_w, total_threads);
    }
  } else {
    if (!adaptive) {
      if (param.pooling_type == "max") {
        max_pool_kernel<<<blocks, threads, 0, stream>>>(input_data,
                                                        output_data,
                                                        spatial_in,
                                                        spatial_out,
                                                        in_h,
                                                        in_w,
                                                        out_h,
                                                        out_w,
                                                        pad_h,
                                                        pad_w,
                                                        win_h,
                                                        win_w,
                                                        stride_h,
                                                        stride_w,
                                                        total_threads);
      } else {
        avg_pool_kernel<<<blocks, threads, 0, stream>>>(input_data,
                                                        output_data,
                                                        spatial_in,
                                                        spatial_out,
                                                        in_h,
                                                        in_w,
                                                        out_h,
                                                        out_w,
                                                        pad_h,
                                                        pad_w,
                                                        win_h,
                                                        win_w,
                                                        stride_h,
                                                        stride_w,
                                                        exclusive,
                                                        total_threads);
      }
    } else {
      if (param.pooling_type == "max") {
        adaptive_max_pool_kernel<<<blocks, threads, 0, stream>>>(input_data,
                                                                 output_data,
                                                                 spatial_in,
                                                                 spatial_out,
                                                                 in_h,
                                                                 in_w,
                                                                 out_h,
                                                                 out_w,
                                                                 pad_h,
                                                                 pad_w,
                                                                 win_h,
                                                                 win_w,
                                                                 stride_h,
                                                                 stride_w,
                                                                 total_threads);
      } else {
        adaptive_avg_pool_kernel<<<blocks, threads, 0, stream>>>(input_data,
                                                                 output_data,
                                                                 spatial_in,
                                                                 spatial_out,
                                                                 in_h,
                                                                 in_w,
                                                                 out_h,
                                                                 out_w,
                                                                 pad_h,
                                                                 pad_w,
                                                                 win_h,
                                                                 win_w,
                                                                 stride_h,
                                                                 stride_w,
                                                                 total_threads);
      }
    }
  }
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(FATAL) << hipGetErrorString(error);
}

inline int PoolOutputSize(
    int input_size, int filter_size, int padding, int stride, bool ceil_mode) {
  int output_size;
  if (!ceil_mode) {
    output_size = (input_size - filter_size + 2 * padding) / stride + 1;
  } else {
    output_size =
        (input_size - filter_size + 2 * padding + stride - 1) / stride + 1;
  }
  return output_size;
}

void PoolComputeNHWC::PrepareForRun() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  pool_impl_.reset(new lite::cuda::math::CudnnPool2DNHWC<PRECISION(kFloat)>);
  pool_impl_->init(param, &ctx);
}

void PoolComputeNHWC::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();
  const auto x_dims = param.x->dims();
  std::vector<int>& ksize = param.ksize;
  if (param.global_pooling) {
    ksize.resize(static_cast<size_t>(x_dims.size()) - 2);
    for (size_t i = 0; i < ksize.size(); ++i) {
      (*param.paddings)[i] = 0;
      ksize[i] = static_cast<int>(x_dims[i + 1]);
    }
  }

  std::vector<int64_t> output_shape({x_dims[0]});
  if (param.adaptive) {
    output_shape.insert(
        output_shape.end(), param.ksize.begin(), param.ksize.end());
  } else {
    for (size_t i = 0; i < param.ksize.size(); ++i) {
      output_shape.push_back(PoolOutputSize(x_dims[i + 1],
                                            param.ksize[i],
                                            (*param.paddings)[i],
                                            param.strides[i],
                                            param.ceil_mode));
    }
  }
  output_shape.push_back(x_dims[3]);
  param.output->Resize(lite::DDim(output_shape));

  pool_impl_->run(param);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(FATAL) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(
    pool2d, kCUDA, kFloat, kNCHW, paddle::lite::kernels::cuda::PoolCompute, def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();

REGISTER_LITE_KERNEL(pool2d,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::PoolComputeNHWC,
                     def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();
