#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/tanh_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename T>
__global__ void TanhKernel(const int num, const T* input, T* output) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
    output[index] = tanh(input[index]);
  }
}

void TanhCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();

  int num = static_cast<int>(param.X->numel());
  auto input = param.X->data<float>();
  auto output = param.Out->mutable_data<float>(TARGET(kCUDA));

  const int threads = 512;
  const int blocks = (num + threads - 1) / threads;
  TanhKernel<float><<<blocks, threads, 0, stream>>>(num, input, output);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(ERROR) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(
    tanh, kCUDA, kFloat, kNCHW, paddle::lite::kernels::cuda::TanhCompute, def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
