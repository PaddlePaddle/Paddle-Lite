#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "lite/core/op_registry.h"
#include "lite/core/type_system.h"
#include "lite/kernels/cuda/calib_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

__device__ __forceinline__ int8_t float2int8(float x) {
  x = fmaxf(x, INT8_MIN);
  x = fminf(x, INT8_MAX);
  return __float2int_rn(x);
}

__global__ void Fp32ToInt8Kernel(const int num,
                                 const float scale,
                                 const float* input,
                                 int8_t* output) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
    output[index] = float2int8(input[index] / scale);
  }
}

__global__ void Int8ToFp32Kernel(const int num,
                                 const float scale,
                                 const int8_t* input,
                                 float* output) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
    output[index] = input[index] * scale;
  }
}

void CalibComputeFp32ToInt8::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->As<HIPContext>();
  auto stream = ctx.exec_stream();

  auto scale = param.scale;
  const auto* din = param.input->data<float>();
  auto* dout = param.output->mutable_data<int8_t>(TARGET(kCUDA));
  int num = static_cast<int>(param.input->numel());
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;
  Fp32ToInt8Kernel<<<blocks, threads, 0, stream>>>(num, scale, din, dout);
  hipError_t error = hipGetLastError();
  CHECK(error == hipSuccess) << hipGetErrorString(error);
}

void CalibComputeInt8ToFp32::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->As<HIPContext>();
  auto stream = ctx.exec_stream();

  auto scale = param.scale;
  const auto* din = param.input->data<int8_t>();
  auto* dout = param.output->mutable_data<float>(TARGET(kCUDA));
  int num = static_cast<int>(param.input->numel());
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;
  Int8ToFp32Kernel<<<blocks, threads, 0, stream>>>(num, scale, din, dout);
  hipError_t error = hipGetLastError();
  CHECK(error == hipSuccess) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(calib,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::CalibComputeFp32ToInt8,
                     fp32_to_int8)
    .BindInput("Input",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kAny))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kInt8),
                                       DATALAYOUT(kAny))})
    .Finalize();

REGISTER_LITE_KERNEL(calib,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::CalibComputeInt8ToFp32,
                     int8_to_fp32)
    .BindInput("Input",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kInt8),
                                      DATALAYOUT(kAny))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kAny))})
    .Finalize();

REGISTER_LITE_KERNEL(calib_once,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::CalibComputeFp32ToInt8,
                     fp32_to_int8)
    .BindInput("Input",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kAny))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kInt8),
                                       DATALAYOUT(kAny))})
    .Finalize();
REGISTER_LITE_KERNEL(calib_once,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::CalibComputeInt8ToFp32,
                     int8_to_fp32)
    .BindInput("Input",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kInt8),
                                      DATALAYOUT(kAny))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kAny))})
    .Finalize();
