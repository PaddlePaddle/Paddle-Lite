#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <limits>
#include <vector>
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/sequence_topk_avg_pooling_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename Dtype>
__global__ void topk_avg_pooling_kernel_by_row_improve(
    Dtype *output_data,
    const Dtype *input,
    const int *gpu_input_offset_l,
    const int *gpu_input_offset_r,
    const int row_max,
    const int col_max,
    const int topk_size,
    const int *topks,
    const int feat_map_num) {
  int row =
      gpu_input_offset_l[blockIdx.x + 1] - gpu_input_offset_l[blockIdx.x];  // 8
  int col = gpu_input_offset_r[blockIdx.x + 1] -
            gpu_input_offset_r[blockIdx.x];  // 30

  int max_k = topks[topk_size - 1];
  max_k = max_k < col ? max_k : col;

  extern __shared__ Dtype smem[];  // H*W

  const Dtype *fm_row_in_data = input +
                                blockIdx.x * row_max * feat_map_num * col_max +
                                blockIdx.y * row_max * col_max;

  for (int i = threadIdx.x; i < row * col_max; i += blockDim.x) {
    smem[i] = fm_row_in_data[i];
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < row; idx += blockDim.x) {
    Dtype *fm_row_out_data =
        output_data +
        (gpu_input_offset_l[blockIdx.x] + idx) * feat_map_num * topk_size +
        blockIdx.y * topk_size;

    Dtype *smem_start_col = smem + idx * col_max;

    int counter = max_k;  // topk_size;
    Dtype last_max_val = -20000.0;
    while (counter) {
      Dtype max_val = -10000.0;
      int max_pos = 0;
      int m = 0;
      for (; m < col; m++) {
        Dtype cur_data = smem_start_col[m];
        if (cur_data > max_val) {
          max_val = cur_data;
          max_pos = m;
          last_max_val = max_val;
        }
      }
      if (max_val < -9999.0) {  // == -10000.0
        max_val = last_max_val;
      }
      smem_start_col[max_pos] = 10000000.0;
      int i = max_k - counter;
      for (int c = 0; c < topk_size; c++) {
        if (i <= topks[c] - 1) {
          fm_row_out_data[c] += max_val;
        }
      }
      counter--;
    }
    __syncthreads();
    // compute avg
    for (int i = 0; i < topk_size; i++) {
      fm_row_out_data[i] = fm_row_out_data[i] / topks[i];
    }
  }
}

template <typename T>
void SequenceTopkAvgPoolingCompute<T>::Run() {
  auto &param = this->Param<param_t>();
  auto &ctx = this->ctx_->template As<HIPContext>();
  auto hip_stream = ctx.exec_stream();

  int topk_num = param.topks.size();
  lite::DDim top_ks_shape(std::vector<int64_t>{topk_num, 1, 1, 1});
  _top_ks.Resize(top_ks_shape);
  hipMemcpyAsync(_top_ks.mutable_data<int>(TARGET(kCUDA)),
                  &param.topks[0],
                  sizeof(int) * topk_num,
                  hipMemcpyHostToDevice,
                  hip_stream);

  int width_offset_len = param.X->lod()[0].size();
  lite::DDim width_offset_shape(
      std::vector<int64_t>{width_offset_len, 1, 1, 1});
  _width_offset.Resize(width_offset_shape);
  hipMemcpyAsync(_width_offset.mutable_data<int>(TARGET(kCUDA)),
                  &(param.X->lod()[0][0]),
                  sizeof(int) * width_offset_len,
                  hipMemcpyHostToDevice,
                  hip_stream);

  int height_offset_len = param.ROW->lod()[0].size();
  lite::DDim height_offset_shape(
      std::vector<int64_t>{height_offset_len, 1, 1, 1});
  _height_offset.Resize(height_offset_shape);
  hipMemcpyAsync(_height_offset.mutable_data<int>(TARGET(kCUDA)),
                  &(param.ROW->lod()[0][0]),
                  sizeof(int) * height_offset_len,
                  hipMemcpyHostToDevice,
                  hip_stream);

  const Tensor *x_tensor = param.X;
  Tensor *out_tensor = param.Out;
  const T *in_data = x_tensor->data<T>();
  T *out_data = out_tensor->mutable_data<T>(TARGET(kCUDA));
  TargetWrapperCuda::MemsetAsync(out_tensor->mutable_data<T>(TARGET(kCUDA)),
                                 0,
                                 sizeof(T) * out_tensor->numel(),
                                 hip_stream);

  auto x_dims = x_tensor->dims();
  int num = x_dims[0];
  int channel = x_dims[1];
  int height = x_dims[2];
  int width = x_dims[3];

  const int *height_offset = _height_offset.data<int>();
  const int *width_offset = _width_offset.data<int>();

  int feat_map_size = height * width;
  dim3 blocks(num, channel);
  dim3 threads(32, 1);
  topk_avg_pooling_kernel_by_row_improve<
      T><<<blocks, threads, feat_map_size * sizeof(T), hip_stream>>>(
      out_data,
      in_data,
      height_offset,
      width_offset,
      height,
      width,
      param.topks.size(),
      _top_ks.data<int>(),
      param.channel_num);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(
    sequence_topk_avg_pooling,
    kCUDA,
    kFloat,
    kNCHW,
    paddle::lite::kernels::cuda::SequenceTopkAvgPoolingCompute<float>,
    def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("ROW",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("COLUMN",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .BindOutput("pos",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();
