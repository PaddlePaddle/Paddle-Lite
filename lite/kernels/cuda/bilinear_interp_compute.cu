#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <vector>
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/bilinear_interp_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {
using Tensor = lite::Tensor;

template <typename T>
__global__ void BilinearInterp(const T* in,
                               const size_t in_img_h,
                               const size_t in_img_w,
                               const size_t input_h,
                               const size_t input_w,
                               T* out,
                               const size_t out_img_h,
                               const size_t out_img_w,
                               const size_t output_h,
                               const size_t output_w,
                               const size_t num_channels,
                               const float ratio_h,
                               const float ratio_w,
                               const bool align_corners,
                               const int align_mode) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id = out_id_w / out_img_size;
    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int out_img_idx = tid % out_img_w;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                          in_img_idy * in_img_w + in_img_idx];

    // bilinear interpolation
    out[out_id_h * output_w + out_id_w] =
        h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
        h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                    w1lambda * in_pos[h_id * in_img_w + w_id]);
  }
}

void BilinearInterpCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();

  Tensor* input = param.X;
  Tensor* output = param.Out;
  Tensor* out_size = param.OutSize;

  auto* input_data = input->data<float>();

  const int n = input->dims()[0];
  const int c = input->dims()[1];
  const int in_h = input->dims()[2];
  const int in_w = input->dims()[3];

  int out_h = param.out_h;
  int out_w = param.out_w;
  float scale = param.scale;
  bool align_corners = param.align_corners;
  if (scale > 0) {
    out_h = static_cast<int>(in_h * scale);
    out_w = static_cast<int>(in_w * scale);
  }

  if (out_size != nullptr) {
    Tensor sizes;
    float* size_data = sizes.mutable_data<float>();
    float* outsize_data = out_size->mutable_data<float>(TARGET(kCUDA));
    hipMemcpy(
        size_data, outsize_data, sizeof(float) * 2, hipMemcpyDeviceToHost);
    out_h = static_cast<int>(size_data[0]);
    out_w = static_cast<int>(size_data[1]);
  }

  auto output_data = output->mutable_data<float>(TARGET(kCUDA));

  if (in_h == out_h && in_w == out_w) {
    hipMemcpy(output_data,
               input_data,
               sizeof(float) * n * c * in_h * in_w,
               hipMemcpyHostToDevice);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixel_num = n * out_chw;
  int threads = 512;
  int blocks = (pixel_num + threads - 1) / threads;
  blocks = blocks > 8 ? 8 : blocks;
  int align_mode = param.align_mode;

  BilinearInterp<<<blocks, threads, 0, stream>>>(input_data,
                                                 in_h,
                                                 in_w,
                                                 n,
                                                 in_chw,
                                                 output_data,
                                                 out_h,
                                                 out_w,
                                                 n,
                                                 out_chw,
                                                 c,
                                                 ratio_h,
                                                 ratio_w,
                                                 align_corners,
                                                 align_mode);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(bilinear_interp,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::BilinearInterpCompute,
                     def)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindInput("OutSize",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNCHW))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNCHW))})
    .Finalize();
