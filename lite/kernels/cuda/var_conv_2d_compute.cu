/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <memory>
#include <vector>
#include "lite/backends/cuda/math/gemm.h"
#include "lite/core/op_registry.h"
#include "lite/core/target_wrapper.h"
#include "lite/core/tensor.h"
#include "lite/kernels/cuda/var_conv_2d_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

inline int ConvOutputSize(int input_size,
                          int filter_size,
                          int dilation,
                          int pad_left,
                          int pad_right,
                          int stride) {
  const int dkernel = dilation * (filter_size - 1) + 1;
  int output_size =
      (input_size + (pad_left + pad_right) - dkernel) / stride + 1;

  return output_size;
}

void VarConv2DCompute::PrepareForRun() {
  auto& context = this->ctx_->template As<HIPContext>();
  auto stream = context.exec_stream();
  auto& param = this->Param<param_t>();
  conv_param_.x = const_cast<lite::Tensor*>(param.X);
  conv_param_.var_length = true;

  conv_param_.paddings.reset(new std::vector<int>);
  conv_param_.paddings->push_back(static_cast<int>(param.kernel_h / 2));
  conv_param_.paddings->push_back(static_cast<int>(param.kernel_h / 2));
  conv_param_.paddings->push_back(static_cast<int>(param.kernel_w / 2));
  conv_param_.paddings->push_back(static_cast<int>(param.kernel_w / 2));
  conv_param_.dilations.reset(new std::vector<int>);
  conv_param_.dilations->push_back(1);
  conv_param_.dilations->push_back(1);
  conv_param_.strides[0] = param.stride_h;
  conv_param_.strides[1] = param.stride_w;
  conv_param_.filter = const_cast<lite::Tensor*>(param.W);
  conv_param_.filter->Resize({param.output_channel,
                              param.input_channel,
                              param.kernel_h,
                              param.kernel_w});

  conv_param_.output = param.Out;
  std::vector<int64_t> output_shape(
      {conv_param_.x->dims()[0], param.output_channel});
  for (size_t i = 0; i < conv_param_.strides.size(); ++i) {
    output_shape.push_back(
        ConvOutputSize(conv_param_.x->dims()[i + 2],
                       conv_param_.filter->dims()[i + 2],
                       (*conv_param_.dilations.get())[i],
                       (*conv_param_.paddings.get())[i * 2],
                       (*conv_param_.paddings.get())[i * 2 + 1],
                       conv_param_.strides[i]));
  }
  if (param.fuse_relu) {
    conv_param_.activation_param.has_active = true;
    conv_param_.activation_param.active_type = lite_api::ActivationType::kRelu;
  }
  conv_param_.output->Resize({output_shape});
  conv_impl_.reset(new lite::cuda::math::CudnnConv2D<PRECISION(kFloat)>);
  conv_impl_->init(conv_param_, &context);
}

void VarConv2DCompute::Run() {
  auto& context = this->ctx_->template As<HIPContext>();
  auto stream = context.exec_stream();
  auto& param = this->Param<param_t>();

  param.Out->set_lod(param.X->lod());
  std::vector<int64_t> output_shape(
      {conv_param_.x->dims()[0], param.output_channel});
  for (size_t i = 0; i < conv_param_.strides.size(); ++i) {
    output_shape.push_back(
        ConvOutputSize(conv_param_.x->dims()[i + 2],
                       conv_param_.filter->dims()[i + 2],
                       (*conv_param_.dilations.get())[i],
                       (*conv_param_.paddings.get())[i * 2],
                       (*conv_param_.paddings.get())[i * 2 + 1],
                       conv_param_.strides[i]));
  }
  conv_param_.output->Resize({output_shape});
  conv_impl_->create(conv_param_, &context);
  conv_impl_->run(conv_param_);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(var_conv_2d,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::VarConv2DCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Col", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
