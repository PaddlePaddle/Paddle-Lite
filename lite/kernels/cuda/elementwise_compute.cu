/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <map>
#include <vector>
#include "lite/backends/cuda/math/elementwise.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/elementwise_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

inline DDim trim_trailing_singular_dims(const DDim& dims) {
  // Remove trailing dimensions of size 1 for y
  auto actual_dims_size = dims.size();
  for (; actual_dims_size != 0; --actual_dims_size) {
    if (dims[actual_dims_size - 1] != 1) break;
  }

  std::vector<int64_t> trim_dims;
  trim_dims.resize(actual_dims_size);
  for (int i = 0; i < actual_dims_size; ++i) {
    trim_dims[i] = dims[i];
  }
  if (trim_dims.size() == 0) {
    return DDim();
  }
  return DDim(trim_dims);
}

inline bool is_broadcast(const DDim& x_dims,
                         const DDim& y_dims,
                         int axis,
                         int* pre,
                         int* n,
                         int* post) {
  if (axis < 0) {
    axis = x_dims.size() - y_dims.size();
  }
  DDim y_dim_trim = trim_trailing_singular_dims(y_dims);
  axis = (y_dim_trim.size() == 0) ? x_dims.size() : axis;
  if (x_dims.size() == y_dim_trim.size()) {
    return false;
  }
  *pre = 1;
  *n = 1;
  *post = 1;
  for (int i = 0; i < axis; ++i) {
    (*pre) *= x_dims[i];
  }
  for (int i = 0; i < y_dim_trim.size(); ++i) {
    CHECK_EQ(x_dims[i + axis], y_dim_trim[i])
        << "Broadcast dimension mismatch.";
    (*n) *= y_dim_trim[i];
  }
  for (int i = axis + y_dim_trim.size(); i < x_dims.size(); ++i) {
    (*post) *= x_dims[i];
  }
  return true;
}

#define ELEMENTWISE_COMPUTE(OP, WITH_RELU)                           \
  auto& param = this->Param<param_t>();                              \
  auto& ctx = this->ctx_->template As<HIPContext>();                \
  auto stream = ctx.exec_stream();                                   \
  const lite::Tensor* x = param.X;                                   \
  const lite::Tensor* y = param.Y;                                   \
  lite::Tensor* out = param.Out;                                     \
  int axis = param.axis;                                             \
  auto* x_data = x->data<float>();                                   \
  auto* y_data = y->data<float>();                                   \
  auto out_data = out->mutable_data<float>(TARGET(kCUDA));           \
  int pixel_num = x->numel();                                        \
  int pre = 1;                                                       \
  int n = pixel_num;                                                 \
  int post = 1;                                                      \
  if (WITH_RELU) {                                                   \
    if (is_broadcast(x->dims(), y->dims(), axis, &pre, &n, &post)) { \
      lite::cuda::math::elementwise_relu(                            \
          x_data, y_data, out_data, pre, n, post, OP, stream);       \
    } else {                                                         \
      lite::cuda::math::elementwise_relu(                            \
          x_data, y_data, out_data, 1, pixel_num, 1, OP, stream);    \
    }                                                                \
  } else {                                                           \
    if (is_broadcast(x->dims(), y->dims(), axis, &pre, &n, &post)) { \
      lite::cuda::math::elementwise(                                 \
          x_data, y_data, out_data, pre, n, post, OP, stream);       \
    } else {                                                         \
      lite::cuda::math::elementwise(                                 \
          x_data, y_data, out_data, 1, pixel_num, 1, OP, stream);    \
    }                                                                \
  }

#define ELEMENTWISE_COMPUTE_NHWC(OP, WITH_RELU)                      \
  std::map<int, int> pos_map = {{0, 0}, {1, 3}, {2, 1}, {3, 2}};     \
  auto& param = this->Param<param_t>();                              \
  auto& ctx = this->ctx_->template As<HIPContext>();                \
  auto stream = ctx.exec_stream();                                   \
  const lite::Tensor* x = param.X;                                   \
  const lite::Tensor* y = param.Y;                                   \
  lite::Tensor* out = param.Out;                                     \
  int axis = param.axis;                                             \
  if (axis < 0) axis = x->dims().size() - y->dims().size();          \
  CHECK(axis >= 0) << "invalid axis of elementwise op";              \
  axis = pos_map[axis];                                              \
  auto* x_data = x->data<float>();                                   \
  auto* y_data = y->data<float>();                                   \
  auto out_data = out->mutable_data<float>(TARGET(kCUDA));           \
  int pixel_num = x->numel();                                        \
  int pre = 1;                                                       \
  int n = pixel_num;                                                 \
  int post = 1;                                                      \
  if (WITH_RELU) {                                                   \
    if (is_broadcast(x->dims(), y->dims(), axis, &pre, &n, &post)) { \
      lite::cuda::math::elementwise_relu(                            \
          x_data, y_data, out_data, pre, n, post, OP, stream);       \
    } else {                                                         \
      lite::cuda::math::elementwise_relu(                            \
          x_data, y_data, out_data, 1, pixel_num, 1, OP, stream);    \
    }                                                                \
  } else {                                                           \
    if (is_broadcast(x->dims(), y->dims(), axis, &pre, &n, &post)) { \
      lite::cuda::math::elementwise(                                 \
          x_data, y_data, out_data, pre, n, post, OP, stream);       \
    } else {                                                         \
      lite::cuda::math::elementwise(                                 \
          x_data, y_data, out_data, 1, pixel_num, 1, OP, stream);    \
    }                                                                \
  }

void ElementwiseAddCompute::Run() {
  ELEMENTWISE_COMPUTE(lite::cuda::math::BinaryOperation::kADD, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseAddComputeNHWC::Run() {
  ELEMENTWISE_COMPUTE_NHWC(lite::cuda::math::BinaryOperation::kADD, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseSubCompute::Run() {
  ELEMENTWISE_COMPUTE(lite::cuda::math::BinaryOperation::kSUB, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseSubComputeNHWC::Run() {
  ELEMENTWISE_COMPUTE_NHWC(lite::cuda::math::BinaryOperation::kSUB, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseMulCompute::Run() {
  ELEMENTWISE_COMPUTE(lite::cuda::math::BinaryOperation::kMUL, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseMulComputeNHWC::Run() {
  ELEMENTWISE_COMPUTE_NHWC(lite::cuda::math::BinaryOperation::kMUL, false)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseAddReluCompute::Run() {
  ELEMENTWISE_COMPUTE(lite::cuda::math::BinaryOperation::kADD, true)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseAddReluComputeNHWC::Run() {
  ELEMENTWISE_COMPUTE_NHWC(lite::cuda::math::BinaryOperation::kADD, true)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseMulReluCompute::Run() {
  ELEMENTWISE_COMPUTE(lite::cuda::math::BinaryOperation::kMUL, true)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

void ElementwiseMulReluComputeNHWC::Run() {
  ELEMENTWISE_COMPUTE_NHWC(lite::cuda::math::BinaryOperation::kMUL, true)
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(elementwise_add,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::ElementwiseAddCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(elementwise_sub,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::ElementwiseSubCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(elementwise_add,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::ElementwiseAddComputeNHWC,
                     nhwc_format)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();

REGISTER_LITE_KERNEL(elementwise_sub,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::ElementwiseSubComputeNHWC,
                     nhwc_format)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();

REGISTER_LITE_KERNEL(elementwise_mul,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::ElementwiseMulCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(elementwise_mul,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::ElementwiseMulComputeNHWC,
                     nhwc_format)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();

REGISTER_LITE_KERNEL(fusion_elementwise_add_activation,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::ElementwiseAddReluCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(fusion_elementwise_add_activation,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::ElementwiseAddReluComputeNHWC,
                     nhwc_format)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();

REGISTER_LITE_KERNEL(fusion_elementwise_mul_activation,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::ElementwiseMulReluCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(fusion_elementwise_mul_activation,
                     kCUDA,
                     kFloat,
                     kNHWC,
                     paddle::lite::kernels::cuda::ElementwiseMulReluComputeNHWC,
                     nhwc_format)
    .BindInput("X",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindInput("Y",
               {LiteType::GetTensorTy(TARGET(kCUDA),
                                      PRECISION(kFloat),
                                      DATALAYOUT(kNHWC))})
    .BindOutput("Out",
                {LiteType::GetTensorTy(TARGET(kCUDA),
                                       PRECISION(kFloat),
                                       DATALAYOUT(kNHWC))})
    .Finalize();
