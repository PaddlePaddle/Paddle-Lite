#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "lite/core/op_registry.h"
#include "lite/core/target_wrapper.h"
#include "lite/kernels/cuda/attention_padding_mask_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

#define CUDA_NUM_THREADS 256

inline int CUDA_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void ker_attention_padding_mask(T* out_data,
                                           const T* attn_data,
                                           const int* src_offset,
                                           const int attn_seq_num,
                                           const int attn_seq_len,
                                           const int src_seq_num,
                                           const int src_seq_len,
                                           const T* pad_begin_data,
                                           const T mask,
                                           const int count) {
  CUDA_KERNEL_LOOP(tid, count) {
    int src_word_id = tid % src_seq_len;
    int tmp_tid = tid / src_seq_len;
    int attn_seq_id = tmp_tid / attn_seq_len;
    int attn_word_id = tmp_tid % attn_seq_len;
    int src_seq_id = attn_seq_id % src_seq_num;
    int cur_len = src_offset[src_seq_id + 1] - src_offset[src_seq_id];

    int k = static_cast<int>(pad_begin_data[src_seq_id]);
    if (k < cur_len &&
        tid >= src_seq_len * (attn_seq_len * attn_seq_id + attn_word_id) + k &&
        tid < src_seq_len * (attn_seq_len * attn_seq_id + attn_word_id) +
                  cur_len) {
      out_data[tid] = mask;
    } else {
      out_data[tid] = attn_data[tid];
    }
  }
}

template <typename Dtype>
__global__ void ker_find_begin_data(int count,
                                    Dtype* out,
                                    const Dtype* src,
                                    const Dtype pad_data,
                                    const int offset_len) {
  CUDA_KERNEL_LOOP(tid, count) {
    int index = offset_len - 1;
    const Dtype* src_data = src + offset_len * tid;
    for (; index >= 0 && pad_data == src_data[index]; --index) {
    }
    out[tid] = index + 1;
  }
}

void AttentionPaddingMaskCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();
  auto stream = ctx.exec_stream();

  auto attn = param.X;
  auto src = param.Y;
  const int count = attn->numel();
  auto attn_offset = attn->lod()[0];
  auto src_offset = src->lod()[0];
  const int attn_seq_num = attn_offset.size() - 1;
  const int attn_seq_len = attn_offset[1];
  const int src_seq_num = src_offset.size() - 1;
  const int src_seq_len = count / attn->dims()[0];

  auto out = param.Out;
  out->Resize(attn->dims());
  out->set_lod(attn->lod());

  auto attn_data = attn->data<float>();
  auto out_data = out->mutable_data<float>(TARGET(kCUDA));

  param.pad_begin->Resize({static_cast<int64_t>(src_seq_num)});
  auto pad_begin_cuda_data =
      param.pad_begin->mutable_data<float>(TARGET(kCUDA));
  ker_find_begin_data<
      float><<<CUDA_GET_BLOCKS(src_seq_num), CUDA_NUM_THREADS, 0, stream>>>(
      src_seq_num,
      pad_begin_cuda_data,
      src->data<float>(),
      static_cast<float>(param.pad_id),
      static_cast<int>(src->lod()[0][1]));

  std::vector<int> src_offset_cpu(src_offset.size(), 0);
  for (int i = 0; i < src_offset.size(); i++) {
    src_offset_cpu[i] = src_offset[i];
  }

  src_offset_cuda.Resize({static_cast<int64_t>(src_offset.size())});
  auto src_offset_cuda_data = src_offset_cuda.mutable_data<int>(TARGET(kCUDA));
  TargetWrapperCuda::MemcpyAsync(src_offset_cuda_data,
                                 src_offset_cpu.data(),
                                 sizeof(int) * src_offset.size(),
                                 IoDirection::HtoD,
                                 stream);

  ker_attention_padding_mask<
      float><<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
      out_data,
      attn_data,
      src_offset_cuda_data,
      attn_seq_num,
      attn_seq_len,
      src_seq_num,
      src_seq_len,
      pad_begin_cuda_data,
      param.mask,
      count);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(ERROR) << hipGetErrorString(error);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(search_attention_padding_mask,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::AttentionPaddingMaskCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("Y", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("pad_begin", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();
