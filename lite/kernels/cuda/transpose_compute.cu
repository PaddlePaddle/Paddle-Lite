/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <vector>
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/transpose_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

void TransposeCompute::Run() {
  auto& param = this->Param<param_t>();
  auto& ctx = this->ctx_->template As<HIPContext>();

  const lite::Tensor* X = param.x;
  lite::Tensor* Out = param.output;
  std::vector<int> axes = param.axis;

  const float* in = X->data<float>();
  float* out = Out->mutable_data<float>(TARGET(kCUDA));

  int ndim = X->dims().size();
  std::vector<int64_t> dims = X->dims().data();

  // NCHW -> NHWC
  if (axes.size() == 4 && axes[0] == 0 && axes[1] == 2 && axes[2] == 3 &&
      axes[3] == 1) {
    lite::cuda::math::NCHW2NHWC(
        dims[0], dims[1], dims[2] * dims[3], in, out, &ctx);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
    return;
  }

  // NHWC -> NCHW
  if (axes.size() == 4 && axes[0] == 0 && axes[1] == 3 && axes[2] == 1 &&
      axes[3] == 2) {
    lite::cuda::math::NHWC2NCHW(
        dims[0], dims[3], dims[1] * dims[2], in, out, &ctx);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) LOG(INFO) << hipGetErrorString(error);
    return;
  }

  lite::cuda::math::Transpose(dims, axes, in, out, &ctx);
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

REGISTER_LITE_KERNEL(transpose,
                     kCUDA,
                     kFloat,
                     kNCHW,
                     paddle::lite::kernels::cuda::TransposeCompute,
                     def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

// REGISTER_LITE_KERNEL(transpose2,
//                      kCUDA,
//                      kFloat,
//                      kNCHW,
//                      paddle::lite::kernels::cuda::TransposeCompute,
//                      def)
//     .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
//     .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
//     .BindOutput("XShape", {LiteType::GetTensorTy(TARGET(kCUDA))})
//     .Finalize();
