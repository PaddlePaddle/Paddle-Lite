#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>

#include "lite/backends/cuda/math/gru_forward.h"
#include "lite/core/device_info.h"

namespace paddle {
namespace lite {
namespace cuda {
namespace math {

/*
 * threads(frame_per_block, batch_per_block)
 * grid(frame_blocks, batch_blocks)
 */
template <typename T>
__global__ void GruForwardResetOutput(
    T* gate_value,
    T* reset_output_value,
    T* prev_output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_gate,
    bool is_batch) {
  const int frame_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (frame_idx >= frame_size) return;

  int batch_idx = 0;
  if (is_batch) {
    batch_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_idx >= batch_size) return;
    gate_value += batch_idx * 3 * frame_size;
    reset_output_value += batch_idx * frame_size;
  }
  T prev_out = 0;
  T reset_out_val;
  T update_gate_value = gate_value[frame_idx + frame_size * 0];
  T reset_gate_value = gate_value[frame_idx + frame_size * 1];

  if (prev_output_value) {
    if (is_batch) {
      prev_output_value += batch_idx * frame_size;
    }
    prev_out = prev_output_value[frame_idx];
  }

  if (active_gate == lite::cuda::math::ActivationType::kSigmoid) {
    update_gate_value = Sigmoid(update_gate_value);
    reset_gate_value = Sigmoid(reset_gate_value);
  } else if (active_gate == lite::cuda::math::ActivationType::kReLU) {
    update_gate_value = ReLU(update_gate_value);
    reset_gate_value = ReLU(reset_gate_value);
  } else if (active_gate == lite::cuda::math::ActivationType::kTanh) {
    update_gate_value = Tanh(update_gate_value);
    reset_gate_value = Tanh(reset_gate_value);
  }

  reset_out_val = prev_out * reset_gate_value;

  gate_value[frame_idx + frame_size * 0] = update_gate_value;
  gate_value[frame_idx + frame_size * 1] = reset_gate_value;
  reset_output_value[frame_idx] = reset_out_val;
}

template <>
__global__ void GruForwardResetOutput(
    half* gate_value,
    half* reset_output_value,
    half* prev_output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_gate,
    bool is_batch) {
  const int frame_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (frame_idx >= frame_size) return;

  int batch_idx = 0;
  if (is_batch) {
    batch_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_idx >= batch_size) return;
    gate_value += batch_idx * 3 * frame_size;
    reset_output_value += batch_idx * frame_size;
  }
  half prev_out = 0;
  half reset_out_val;
  half update_gate_value = gate_value[frame_idx + frame_size * 0];
  half reset_gate_value = gate_value[frame_idx + frame_size * 1];

  if (prev_output_value) {
    if (is_batch) {
      prev_output_value += batch_idx * frame_size;
    }
    prev_out = prev_output_value[frame_idx];
  }

  if (active_gate == ActivationType::kSigmoid) {
    update_gate_value = Sigmoid(update_gate_value);
    reset_gate_value = Sigmoid(reset_gate_value);
  } else if (active_gate == ActivationType::kReLU) {
    update_gate_value = ReLU(update_gate_value);
    reset_gate_value = ReLU(reset_gate_value);
  } else if (active_gate == ActivationType::kTanh) {
    update_gate_value = Tanh(update_gate_value);
    reset_gate_value = Tanh(reset_gate_value);
  }
#if __CUDA_ARCH__ >= 530
  reset_out_val = __hmul(prev_out, reset_gate_value);
#else
  reset_out_val =
      __float2half(__half2float(prev_out) * __half2float(reset_gate_value));
#endif

  gate_value[frame_idx + frame_size * 0] = update_gate_value;
  gate_value[frame_idx + frame_size * 1] = reset_gate_value;
  reset_output_value[frame_idx] = reset_out_val;
}

/*
 * threads(frame_per_block, batch_per_block)
 * grid(frame_blocks, batch_blocks)
 */
template <typename T>
__global__ void GruForwardFinalOutput(
    T* gate_value,
    T* prev_output_value,
    T* output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_node,
    bool origin_mode,
    bool is_batch) {
  const int frame_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (frame_idx >= frame_size) return;
  int batch_idx = 0;
  if (is_batch) {
    batch_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_idx >= batch_size) {
      return;
    }
    gate_value += batch_idx * 3 * frame_size;
    output_value += batch_idx * frame_size;
  }

  T output;
  T prev_out = 0;
  T update_gate_value = gate_value[frame_idx + frame_size * 0];
  T state_frame_value = gate_value[frame_idx + frame_size * 2];

  if (prev_output_value) {
    if (is_batch) prev_output_value += batch_idx * frame_size;
    prev_out = prev_output_value[frame_idx];
  }

  if (active_node == lite::cuda::math::ActivationType::kSigmoid) {
    state_frame_value = Sigmoid(state_frame_value);
  } else if (active_node == lite::cuda::math::ActivationType::kReLU) {
    state_frame_value = ReLU(state_frame_value);
  } else if (active_node == lite::cuda::math::ActivationType::kTanh) {
    state_frame_value = Tanh(state_frame_value);
  }

  if (origin_mode) {
    output = update_gate_value * prev_out + state_frame_value -
             update_gate_value * state_frame_value;
  } else {
    output = prev_out - update_gate_value * prev_out +
             update_gate_value * state_frame_value;
  }

  gate_value[frame_idx + frame_size * 2] = state_frame_value;
  output_value[frame_idx] = output;
}

template <>
__global__ void GruForwardFinalOutput(
    half* gate_value,
    half* prev_output_value,
    half* output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_node,
    bool origin_mode,
    bool is_batch) {
  const int frame_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (frame_idx >= frame_size) return;
  int batch_idx = 0;
  if (is_batch) {
    batch_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_idx >= batch_size) {
      return;
    }
    gate_value += batch_idx * 3 * frame_size;
    output_value += batch_idx * frame_size;
  }

  half output;
  half prev_out = 0;
  half update_gate_value = gate_value[frame_idx + frame_size * 0];
  half state_frame_value = gate_value[frame_idx + frame_size * 2];

  if (prev_output_value) {
    if (is_batch) prev_output_value += batch_idx * frame_size;
    prev_out = prev_output_value[frame_idx];
  }

  if (active_node == lite::cuda::math::ActivationType::kSigmoid) {
    state_frame_value = Sigmoid(state_frame_value);
  } else if (active_node == lite::cuda::math::ActivationType::kReLU) {
    state_frame_value = ReLU(state_frame_value);
  } else if (active_node == lite::cuda::math::ActivationType::kTanh) {
    state_frame_value = Tanh(state_frame_value);
  }

  if (origin_mode) {
#if __CUDA_ARCH__ >= 530
    output =
        __hsub(__hadd(__hmul(update_gate_value, prev_out), state_frame_value),
               __hmul(update_gate_value, state_frame_value));
#else
    output = __float2half(
        __half2float(update_gate_value) * __half2float(prev_out) +
        __half2float(state_frame_value) -
        __half2float(update_gate_value) * __half2float(state_frame_value));
#endif
  } else {
#if __CUDA_ARCH__ >= 530
    output = prev_out - update_gate_value * prev_out +
             update_gate_value * state_frame_value;
    output = __hadd(__hsub(prev_out, __hmul(update_gate_value, prev_out)),
                    __hmul(update_gate_value, state_frame_value));
#else
    output = __float2half(
        __half2float(prev_out) -
        __half2float(update_gate_value) * __half2float(prev_out) +
        __half2float(update_gate_value) * __half2float(state_frame_value));
#endif
  }

  gate_value[frame_idx + frame_size * 2] = state_frame_value;
  output_value[frame_idx] = output;
}

template __global__ void GruForwardFinalOutput<float>(
    float* gate_value,
    float* prev_output_value,
    float* output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_node,
    bool origin_mode,
    bool is_batch);

template __global__ void GruForwardResetOutput<float>(
    float* gate_value,
    float* reset_output_value,
    float* prev_output_value,
    int frame_size,
    int batch_size,
    lite::cuda::math::ActivationType active_gate,
    bool is_batch);

}  // namespace math
}  // namespace cuda
}  // namespace lite
}  // namespace paddle
