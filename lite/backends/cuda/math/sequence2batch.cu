#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>

#include "lite/backends/cuda/cuda_utils.h"
#include "lite/backends/cuda/math/sequence2batch.h"
#include "lite/backends/cuda/math/utils.h"

namespace paddle {
namespace lite {
namespace cuda {
namespace math {

template <typename T>
__global__ void CopyMatrixRowsKernel(const T* src,
                                     T* dst,
                                     const uint64_t* index,
                                     int height,
                                     int width,
                                     bool is_src_index) {
  CUDA_KERNEL_LOOP(tid, height * width) {
    int row = tid / width;
    int idx = tid % width;
    int src_row = is_src_index ? index[row] : row;
    int dst_row = is_src_index ? row : index[row];
    dst[dst_row * width + idx] = src[src_row * width + idx];
  }
}

template <typename T>
void CopyMatrixRowsFunctor<T>::operator()(
    const lite::Tensor& src,
    lite::Tensor* dst,
    const std::vector<uint64_t>& index_lod,
    bool is_src_index,
    const hipStream_t& stream) {
  auto src_dims = src.dims();
  auto dst_dims = dst->dims();
  CHECK_EQ(src_dims.size(), 2) << "The src must be matrix with rank 2.";
  CHECK_EQ(dst_dims.size(), 2) << "The dst must be matrix with rank 2.";
  CHECK_EQ(src_dims[1], dst_dims[1])
      << "The width of src and dst must be same.";
  int height = dst_dims[0];
  int width = dst_dims[1];
  const auto* src_data = src.data<T>();
  auto* dst_data = dst->template mutable_data<T>(TARGET(kCUDA));

  index_tensor_.Resize({static_cast<int64_t>(index_lod.size())});
  auto* index_tensor_data = index_tensor_.mutable_data<uint64_t>(TARGET(kCUDA));
  TargetWrapperCuda::MemcpyAsync(index_tensor_data,
                                 index_lod.data(),
                                 sizeof(uint64_t) * index_lod.size(),
                                 IoDirection::HtoD,
                                 stream);
  CopyMatrixRowsKernel<
      T><<<CUDA_GET_BLOCKS(height * width), CUDA_NUM_THREADS, 0, stream>>>(
      src_data, dst_data, index_tensor_data, height, width, is_src_index);
  CUDA_POST_KERNEL_CHECK;
}

template class CopyMatrixRowsFunctor<float>;
template class CopyMatrixRowsFunctor<half>;

template class LoDTensor2BatchFunctor<float>;
template class LoDTensor2BatchFunctor<half>;

template class Batch2LoDTensorFunctor<float>;
template class Batch2LoDTensorFunctor<half>;

}  // namespace math
}  // namespace cuda
}  // namespace lite
}  // namespace paddle
