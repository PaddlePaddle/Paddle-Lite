#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>

#include "lite/backends/cuda/cuda_utils.h"
#include "lite/backends/cuda/math/sequence2batch.h"
#include "lite/backends/cuda/math/utils.h"

namespace paddle {
namespace lite {
namespace cuda {
namespace math {

template <typename T>
__global__ void CopyMatrixRowsKernel(const T* src,
                                     T* dst,
                                     const uint64_t* index,
                                     int height,
                                     int width,
                                     bool is_src_index) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int row_id = blockDim.y * gridDim.x + idy;
  if (row_id < height) {
    int src_idx = is_src_index ? index[row_id] : row_id;
    int dst_idx = is_src_index ? row_id : index[row_id];
    const T* src_data = src + src_idx * width;
    T* dst_data = dst + dst_idx * width;
    for (int i = idx; i < width; i += blockDim.x) {
      dst_data[i] = src_data[i];
    }
  }
}

template <typename T>
void CopyMatrixRowsFunctor<T>::operator()(
    const lite::Tensor& src,
    lite::Tensor* dst,
    const std::vector<uint64_t>& index_lod,
    bool is_src_index,
    const hipStream_t& stream) {
  auto src_dims = src.dims();
  auto dst_dims = dst->dims();
  CHECK_EQ(src_dims.size(), 2) << "The src must be matrix with rank 2.";
  CHECK_EQ(dst_dims.size(), 2) << "The dst must be matrix with rank 2.";
  CHECK_EQ(src_dims[1], dst_dims[1])
      << "The width of src and dst must be same.";
  int height = dst_dims[0];
  int width = dst_dims[1];
  const auto* src_data = src.data<T>();
  auto* dst_data = dst->template mutable_data<T>(TARGET(kCUDA));

  index_tensor_.Resize({static_cast<int64_t>(index_lod.size())});
  auto* index_tensor_data = index_tensor_.mutable_data<uint64_t>(TARGET(kCUDA));
  TargetWrapperCuda::MemcpyAsync(index_tensor_data,
                                 index_lod.data(),
                                 sizeof(uint64_t) * index_lod.size(),
                                 IoDirection::HtoD,
                                 stream);
  dim3 threads(128, 8);
  dim3 grids((height + threads.y - 1) / threads.y);
  CopyMatrixRowsKernel<T><<<grids, threads, 0, stream>>>(
      src_data, dst_data, index_tensor_data, height, width, true);
  CUDA_POST_KERNEL_CHECK;
}

template class CopyMatrixRowsFunctor<float>;
template class LoDTensor2BatchFunctor<float>;
template class Batch2LoDTensorFunctor<float>;

}  // namespace math
}  // namespace cuda
}  // namespace lite
}  // namespace paddle
