#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>
#include "lite/backends/cuda/math/activation.h"
#include "lite/backends/cuda/math/utils.h"

namespace paddle {
namespace lite {
namespace cuda {
namespace math {

template <typename T>
__global__ void relu_kernel(const int num,
                            const T alpha,
                            const T* input,
                            T* output) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
#if __CUDA_ARCH__ >= 350
    output[index] = __ldg(input + index) >= 0 ? __ldg(input + index)
                                              : __ldg(input + index) * alpha;
#else
    output[index] = input[index] >= 0 ? input[index] : input[index] * alpha;
#endif
  }
}

__global__ void bias_relu_int8_nhwc4_kernel(int num,
                                            const float4* in,
                                            const float4* bias,
                                            float4* out,
                                            int N,
                                            int K,
                                            int H,
                                            int W,
                                            const float4* scale,
                                            float alpha) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int bias_idx = tid % K;
    const float4 bias_ptr = bias[bias_idx];
    const float4 scale_ptr = scale[bias_idx];
    const float4 in_ptr = in[tid];

    float4 packed_val;
    packed_val.x = in_ptr.x * scale_ptr.x + bias_ptr.x;
    packed_val.x = fmaxf(packed_val.x * alpha, packed_val.x);
    packed_val.y = in_ptr.y * scale_ptr.y + bias_ptr.y;
    packed_val.y = fmaxf(packed_val.y * alpha, packed_val.y);
    packed_val.z = in_ptr.z * scale_ptr.z + bias_ptr.z;
    packed_val.z = fmaxf(packed_val.z * alpha, packed_val.z);
    packed_val.w = in_ptr.w * scale_ptr.w + bias_ptr.w;
    packed_val.w = fmaxf(packed_val.w * alpha, packed_val.w);
    out[tid] = packed_val;
  }
}

__global__ void bias_relu_int8_nhwc4_kernel(int num,
                                            const float4* in,
                                            const float4* bias,
                                            char4* out,
                                            int N,
                                            int K,
                                            int H,
                                            int W,
                                            const float4* scale,
                                            float alpha) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int bias_idx = tid % K;
    const float4 bias_ptr = bias[bias_idx];
    const float4 scale_ptr = scale[bias_idx];
    const float4 in_ptr = in[tid];

    float4 packed_val;
    char4 result_val;
    packed_val.x = in_ptr.x * scale_ptr.x + bias_ptr.x;
    result_val.x =
        from_float<int8_t>(fmaxf(packed_val.x * alpha, packed_val.x));
    packed_val.y = in_ptr.y * scale_ptr.y + bias_ptr.y;
    result_val.y =
        from_float<int8_t>(fmaxf(packed_val.y * alpha, packed_val.y));
    packed_val.z = in_ptr.z * scale_ptr.z + bias_ptr.z;
    result_val.z =
        from_float<int8_t>(fmaxf(packed_val.z * alpha, packed_val.z));
    packed_val.w = in_ptr.w * scale_ptr.w + bias_ptr.w;
    result_val.w =
        from_float<int8_t>(fmaxf(packed_val.w * alpha, packed_val.w));

    out[tid] = result_val;
  }
}

__global__ void relu_int8_nhwc4_kernel(int num,
                                       const float4* in,
                                       float4* out,
                                       int N,
                                       int K,
                                       int H,
                                       int W,
                                       const float4* scale,
                                       float alpha) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int scale_idx = tid % K;
    const float4 scale_ptr = scale[scale_idx];
    const float4 in_ptr = in[tid];

    float4 packed_val;
    packed_val.x = in_ptr.x * scale_ptr.x;
    packed_val.x = fmaxf(packed_val.x * alpha, packed_val.x);
    packed_val.y = in_ptr.y * scale_ptr.y;
    packed_val.y = fmaxf(packed_val.y * alpha, packed_val.y);
    packed_val.z = in_ptr.z * scale_ptr.z;
    packed_val.z = fmaxf(packed_val.z * alpha, packed_val.z);
    packed_val.w = in_ptr.w * scale_ptr.w;
    packed_val.w = fmaxf(packed_val.w * alpha, packed_val.w);
    out[tid] = packed_val;
  }
}

__global__ void relu_int8_nhwc4_kernel(int num,
                                       const float4* in,
                                       char4* out,
                                       int N,
                                       int K,
                                       int H,
                                       int W,
                                       const float4* scale,
                                       float alpha) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int scale_idx = tid % K;
    const float4 scale_ptr = scale[scale_idx];
    const float4 in_ptr = in[tid];

    float4 packed_val;
    char4 result_val;
    packed_val.x = in_ptr.x * scale_ptr.x;
    result_val.x =
        from_float<int8_t>(fmaxf(packed_val.x * alpha, packed_val.x));
    packed_val.y = in_ptr.y * scale_ptr.y;
    result_val.y =
        from_float<int8_t>(fmaxf(packed_val.y * alpha, packed_val.y));
    packed_val.z = in_ptr.z * scale_ptr.z;
    result_val.z =
        from_float<int8_t>(fmaxf(packed_val.z * alpha, packed_val.z));
    packed_val.w = in_ptr.w * scale_ptr.w;
    result_val.w =
        from_float<int8_t>(fmaxf(packed_val.w * alpha, packed_val.w));

    out[tid] = result_val;
  }
}

template <>
void bias_relu_int8_nhwc4<float>(int num,
                                 const void* in,
                                 const void* bias,
                                 void* out,
                                 int N,
                                 int K,
                                 int H,
                                 int W,
                                 const void* scale,
                                 float alpha,
                                 hipStream_t stream) {
  int thread = 256;
  int block = (num + thread - 1) / thread;
  bias_relu_int8_nhwc4_kernel<<<block, thread, 0, stream>>>(
      num,
      static_cast<const float4*>(in),
      static_cast<const float4*>(bias),
      static_cast<float4*>(out),
      N,
      K,
      H,
      W,
      static_cast<const float4*>(scale),
      alpha);
}

template <>
void bias_relu_int8_nhwc4<int8_t>(int num,
                                  const void* in,
                                  const void* bias,
                                  void* out,
                                  int N,
                                  int K,
                                  int H,
                                  int W,
                                  const void* scale,
                                  float alpha,
                                  hipStream_t stream) {
  int thread = 256;
  int block = (num + thread - 1) / thread;
  bias_relu_int8_nhwc4_kernel<<<block, thread, 0, stream>>>(
      num,
      static_cast<const float4*>(in),
      static_cast<const float4*>(bias),
      static_cast<char4*>(out),
      N,
      K,
      H,
      W,
      static_cast<const float4*>(scale),
      alpha);
}

template <>
void relu_int8_nhwc4<float>(int num,
                            const void* in,
                            void* out,
                            int N,
                            int K,
                            int H,
                            int W,
                            const void* scale,
                            float alpha,
                            hipStream_t stream) {
  int thread = 256;
  int block = (num + thread - 1) / thread;
  relu_int8_nhwc4_kernel<<<block, thread, 0, stream>>>(
      num,
      static_cast<const float4*>(in),
      static_cast<float4*>(out),
      N,
      K,
      H,
      W,
      static_cast<const float4*>(scale),
      alpha);
}

template <>
void relu_int8_nhwc4<int8_t>(int num,
                             const void* in,
                             void* out,
                             int N,
                             int K,
                             int H,
                             int W,
                             const void* scale,
                             float alpha,
                             hipStream_t stream) {
  int thread = 256;
  int block = (num + thread - 1) / thread;
  relu_int8_nhwc4_kernel<<<block, thread, 0, stream>>>(
      num,
      static_cast<const float4*>(in),
      static_cast<char4*>(out),
      N,
      K,
      H,
      W,
      static_cast<const float4*>(scale),
      alpha);
}

template <typename T>
void relu(int num, const T* din, T* dout, float alpha, hipStream_t stream) {
  int thread = 256;
  int block = (num + thread - 1) / thread;
  relu_kernel<<<block, thread, 0, stream>>>(num, alpha, din, dout);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) std::cout << hipGetErrorString(error);
}
template void relu(int, const float*, float*, float, hipStream_t);

}  // namespace math
}  // namespace cuda
}  // namespace lite
}  // namespace paddle
