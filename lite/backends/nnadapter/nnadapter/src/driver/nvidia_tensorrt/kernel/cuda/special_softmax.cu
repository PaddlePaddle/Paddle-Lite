#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cmath>
#include "driver/nvidia_tensorrt/kernel/cuda/special_softmax.h"
#include "driver/nvidia_tensorrt/kernel/cuda/yolo.h"
namespace nnadapter {
namespace nvidia_tensorrt {
namespace cuda {

__global__ void yolo_tensor_bbox_num(const float* input,
                                     int* bbox_count,
                                     const uint gridSize,
                                     const uint numOutputClasses,
                                     const uint numBBoxes,
                                     float prob_thresh) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;
  if (x_id == 0 && y_id == 0 && z_id == 0) printf("%f\n", *input);

  if ((x_id >= gridSize) || (y_id >= gridSize) || (z_id >= numBBoxes)) {
    return;
  }

  const int numGridCells = gridSize * gridSize;
  const int bbindex = y_id * gridSize + x_id;

  // objectness
  float objectness =
      input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];
  if (objectness < prob_thresh) {
    return;
  }

  atomicAdd(bbox_count, 1);
}

inline __device__ void correct_yolo_box(float& x,
                                        float& y,
                                        float& w,
                                        float& h,
                                        float pic_w,
                                        float pic_h,
                                        float netw,
                                        float neth) {
  int new_w = 0;
  int new_h = 0;
  if ((netw / pic_w) < (neth / pic_h)) {
    new_w = netw;
    new_h = (pic_h * netw) / pic_w;
  } else {
    new_h = neth;
    new_w = (pic_w * neth) / pic_h;
  }

  x = (x - (netw - new_w) / 2.) / new_w;
  y = (y - (neth - new_h) / 2.) / new_h;
  w /= (float)new_w;
  h /= (float)new_h;
}

__global__ void yolo_tensor_parse_kernel(const float* input,
                                         const float* ImShape_data,
                                         const float* ScaleFactor_data,
                                         float* output,
                                         int* bbox_index,
                                         const uint gridSize,
                                         const uint numOutputClasses,
                                         const uint numBBoxes,
                                         const uint netw,
                                         const uint neth,
                                         int* biases,
                                         float prob_thresh) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if ((x_id >= gridSize) || (y_id >= gridSize) || (z_id >= numBBoxes)) {
    return;
  }

  const float pic_h = ImShape_data[0] / ScaleFactor_data[0];
  const float pic_w = ImShape_data[1] / ScaleFactor_data[1];

  const int numGridCells = gridSize * gridSize;
  const int bbindex = y_id * gridSize + x_id;

  // objectness
  float objectness =
      input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

  if (objectness < prob_thresh) {
    return;
  }

  int cur_bbox_index = atomicAdd(bbox_index, 1);
  int tensor_index = cur_bbox_index * (5 + numOutputClasses);

  // x
  float x = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)];
  x = (float)((x + (float)x_id) * (float)netw) / (float)gridSize;

  // y
  float y = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)];
  y = (float)((y + (float)y_id) * (float)neth) / (float)gridSize;

  // w
  float w = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)];
  w = w * biases[2 * z_id];

  // h
  float h = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)];
  h = h * biases[2 * z_id + 1];

  correct_yolo_box(x, y, w, h, pic_w, pic_h, netw, neth);

  output[tensor_index] = objectness;
  output[tensor_index + 1] = x;
  output[tensor_index + 2] = y;
  output[tensor_index + 3] = w;
  output[tensor_index + 4] = h;

  // Probabilities of classes
  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob =
        input[bbindex +
              numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] *
        objectness;
    output[tensor_index + 5 + i] = prob < prob_thresh ? 0. : prob;
  }
}

void yolo_tensor_parse_cuda(
    const float* yolo_layer_tensor,  // [in] YOLO layer tensor input
    const float* image_shape_data,
    const float* image_scale_data,
    float** bboxes_tensor_ptr,  // [out] Bounding boxes output tensor
    int& bbox_count_max_alloc,  // [in/out] maximum bounding box number
                                // allocated in dev
    int& bbox_count_host,  // [in/out] bounding boxes number recorded in host
    int* bbox_count_device_ptr,  // [in/out] bounding boxes number calculated in
                                 // device side
    int* bbox_index_device_ptr,  // [in] bounding box index for kernel threads
                                 // shared access
    int gridSize,
    int numOutputClasses,
    int numBBoxes,
    int netw,
    int neth,
    int* biases_device,
    float prob_thresh) {
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSize / threads_per_block.x) + 1,
                        (gridSize / threads_per_block.y) + 1,
                        (numBBoxes / threads_per_block.z) + 1);

  // evaluated how many boxes will be choosed
  int bbox_count = 0;
  hipMemcpy(
      bbox_count_device_ptr, &bbox_count, sizeof(int), hipMemcpyHostToDevice);
  yolo_tensor_bbox_num<<<number_of_blocks, threads_per_block, 0>>>(
      yolo_layer_tensor,
      bbox_count_device_ptr,
      gridSize,
      numOutputClasses,
      numBBoxes,
      prob_thresh);
  hipMemcpy(
      &bbox_count, bbox_count_device_ptr, sizeof(int), hipMemcpyDeviceToHost);

  // Record actual bbox number
  bbox_count_host = bbox_count;

  // Obtain previous allocated bbox tensor in device side
  float* bbox_tensor = *bboxes_tensor_ptr;
  // Update previous maximum bbox number
  if (bbox_count > bbox_count_max_alloc) {
    printf(
        "Bbox tensor expanded: %d -> %d!\n", bbox_count_max_alloc, bbox_count);
    hipFree(bbox_tensor);
    hipMalloc(&bbox_tensor,
               bbox_count * (5 + numOutputClasses) * sizeof(float));
    bbox_count_max_alloc = bbox_count;
    *bboxes_tensor_ptr = bbox_tensor;
  }

  // now generate the boxes
  int bbox_index = 0;
  hipMemcpy(
      bbox_index_device_ptr, &bbox_index, sizeof(int), hipMemcpyHostToDevice);
  yolo_tensor_parse_kernel<<<number_of_blocks, threads_per_block, 0>>>(
      yolo_layer_tensor,
      image_shape_data,
      image_scale_data,
      bbox_tensor,
      bbox_index_device_ptr,
      gridSize,
      numOutputClasses,
      numBBoxes,
      netw,
      neth,
      biases_device,
      prob_thresh);
}

int SpecialSoftmaxKernel::Run(
    core::Operation* operation,
    std::map<core::Operand*, std::shared_ptr<Tensor>>* operand_map) {
  auto& input_operands = operation->input_operands;
  std::vector<const float*> boxes_input;
  std::vector<std::vector<int32_t>> boxes_input_dims;
  for (int i = 0; i < 3; i++) {
    auto input_tensor = operand_map->at(operation->input_operands[i]);
    const float* input = reinterpret_cast<const float*>(input_tensor->Data());
    boxes_input.push_back(input);
    boxes_input_dims.push_back(input_tensor->Dims());
  }
  auto image_shape_tensor = operand_map->at(input_operands[3]);
  auto image_scale_tensor = operand_map->at(input_operands[4]);
  const float* image_shape_data =
      reinterpret_cast<const float*>(image_shape_tensor->Data());
  const float* image_scale_data =
      reinterpret_cast<const float*>(image_scale_tensor->Data());

  auto boxes_scores_tensor = operand_map->at(operation->output_operands[0]);
  auto boxes_num_tensor = operand_map->at(operation->output_operands[1]);
  /* anchors */
  auto anchors_operand0 = input_operands[5];
  auto anchors_operand1 = input_operands[6];
  auto anchors_operand2 = input_operands[7];
  auto anchors_count0 = anchors_operand0->length / sizeof(int32_t);
  auto anchors_count1 = anchors_operand1->length / sizeof(int32_t);
  auto anchors_count2 = anchors_operand2->length / sizeof(int32_t);
  auto anchors_data0 = reinterpret_cast<int32_t*>(anchors_operand0->buffer);
  auto anchors_data1 = reinterpret_cast<int32_t*>(anchors_operand1->buffer);
  auto anchors_data2 = reinterpret_cast<int32_t*>(anchors_operand2->buffer);
  auto anchors =
      std::vector<int32_t>(anchors_count0 + anchors_count1 + anchors_count2);

  memcpy(&anchors[0], anchors_data0, anchors_count0 * sizeof(int));
  memcpy(&anchors[anchors_count0], anchors_data1, anchors_count1 * sizeof(int));
  memcpy(&anchors[anchors_count0 + anchors_count1],
         anchors_data2,
         anchors_count2 * sizeof(int));
  // memcpy anchors to gpu memory
  int* d_anchors;
  hipMalloc(&d_anchors, anchors.size() * sizeof(int));
  hipMemcpy(d_anchors,
             anchors.data(),
             anchors.size() * sizeof(int),
             hipMemcpyHostToDevice);
  int* dev_anchors_ptr[3];
  dev_anchors_ptr[0] = d_anchors;
  dev_anchors_ptr[1] = dev_anchors_ptr[0] + anchors_count0;
  dev_anchors_ptr[2] = dev_anchors_ptr[1] + anchors_count1;
  int anchors_num[3] = {
      anchors_count0 / 2, anchors_count1 / 2, anchors_count2 / 2};
  /* various attrs */
  int class_num = *reinterpret_cast<int*>(input_operands[8]->buffer);
  float conf_thresh = *reinterpret_cast<float*>(input_operands[9]->buffer);
  int downsample_ratio0 = *reinterpret_cast<int*>(input_operands[10]->buffer);
  int downsample_ratio1 = *reinterpret_cast<int*>(input_operands[11]->buffer);
  int downsample_ratio2 = *reinterpret_cast<int*>(input_operands[12]->buffer);
  int downsample_ratio[3] = {
      downsample_ratio0, downsample_ratio1, downsample_ratio2};
  // clip_bbox and scale_x_y is not used now!
  bool clip_bbox = *reinterpret_cast<bool*>(input_operands[13]->buffer);
  float scale_x_y = *reinterpret_cast<float*>(input_operands[14]->buffer);
  // attrs with NMS
  float nms_thresh = *reinterpret_cast<float*>(input_operands[15]->buffer);

  // other attrs
  int batch = image_shape_tensor->Dims()[0];

  TensorInfo* ts_info = new TensorInfo[batch * boxes_input.size()];

  for (int i = 0; i < batch * boxes_input.size(); i++) {
    hipMalloc(
        &ts_info[i].bboxes_dev_ptr,
        ts_info[i].bbox_count_max_alloc * (5 + class_num) * sizeof(float));
    ts_info[i].bboxes_host_ptr =
        (float*)malloc(ts_info[i].bbox_count_max_alloc * sizeof(float));
    hipMalloc(&ts_info[i].bbox_count_device_ptr, sizeof(int));
  }

  // box index counter in gpu memory
  // *bbox_index_device_ptr used by atomicAdd
  int* bbox_index_device_ptr;
  hipMalloc(&bbox_index_device_ptr, sizeof(int));

  std::vector<float> result;

  for (int batch_id = 0; batch_id < batch; batch_id++) {
    for (int input_id = 0; input_id < boxes_input.size(); input_id++) {
      int c = boxes_input_dims[input_id][1];
      int h = boxes_input_dims[input_id][2];
      int w = boxes_input_dims[input_id][3];
      int ts_id = batch_id * boxes_input.size() + input_id;
      int bbox_count_max_alloc = ts_info[ts_id].bbox_count_max_alloc;
      yolo_tensor_parse_cuda(
          boxes_input[input_id] + batch_id * c * h * w,
          image_shape_data + batch_id * 2,
          image_scale_data + batch_id * 2,
          &(ts_info[ts_id].bboxes_dev_ptr),  // output in gpu memory,
                                             // here we must use
                                             // 2-level
          // pointer, because we may re-malloc this area
          bbox_count_max_alloc,  // bbox_count_alloc_ptr boxes we pre-allocate
          ts_info[ts_id].bbox_count_host,        // record bbox numbers
          ts_info[ts_id].bbox_count_device_ptr,  // for atomicAdd
          bbox_index_device_ptr,                 // for atomicAdd
          h,
          class_num,
          anchors_num[input_id],
          downsample_ratio[input_id] * h,
          downsample_ratio[input_id] * w,
          dev_anchors_ptr[input_id],
          conf_thresh);

      // batch info update
      if (bbox_count_max_alloc > ts_info[ts_id].bbox_count_max_alloc) {
        ts_info[ts_id].bbox_count_max_alloc = bbox_count_max_alloc;
        ts_info[ts_id].bboxes_host_ptr = (float*)realloc(
            ts_info[ts_id].bboxes_host_ptr,
            bbox_count_max_alloc * (5 + class_num) * sizeof(float));
      }
      // we need copy bbox_count_host boxes to cpu memory
      hipMemcpy(
          ts_info[ts_id].bboxes_host_ptr,
          ts_info[ts_id].bboxes_dev_ptr,
          ts_info[ts_id].bbox_count_host * (5 + class_num) * sizeof(float),
          hipMemcpyDeviceToHost);

      // copy bbox_count_host boxes to cpu memory
      for (int i = 0; i < ts_info[ts_id].bbox_count_host * (5 + class_num); i++)
        result.push_back(ts_info[ts_id].bboxes_host_ptr[i]);
    }
  }
  boxes_scores_tensor->Resize(
      {(int)(result.size() / (5 + class_num)), 5 + class_num});
  float* boxes_scores_data =
      reinterpret_cast<float*>(boxes_scores_tensor->Data(false));
  memcpy(boxes_scores_data, result.data(), result.size() * sizeof(float));

  boxes_num_tensor->Resize({batch});
  int* boxes_num_data = reinterpret_cast<int*>(boxes_num_tensor->Data(false));

  /* ----------------------- NMS  -----------------------------  */

  for (int batch_id = 0; batch_id < batch; batch_id++) {
    std::vector<detection> bbox_det_vec;

    for (int input_id = 0; input_id < boxes_input.size(); input_id++) {
      int ts_id = batch_id * boxes_input.size() + input_id;
      int bbox_count = ts_info[ts_id].bbox_count_host;

      if (bbox_count <= 0) {
        continue;
      }

      float* bbox_host_ptr = ts_info[ts_id].bboxes_host_ptr;

      for (int bbox_index = 0; bbox_index < bbox_count; ++bbox_index) {
        detection bbox_det;
        memset(&bbox_det, 0, sizeof(detection));
        bbox_det.objectness = bbox_host_ptr[bbox_index * (5 + class_num) + 0];
        bbox_det.bbox.x = bbox_host_ptr[bbox_index * (5 + class_num) + 1];
        bbox_det.bbox.y = bbox_host_ptr[bbox_index * (5 + class_num) + 2];
        bbox_det.bbox.w = bbox_host_ptr[bbox_index * (5 + class_num) + 3];
        bbox_det.bbox.h = bbox_host_ptr[bbox_index * (5 + class_num) + 4];
        bbox_det.classes = class_num;
        bbox_det.prob = (float*)malloc(class_num * sizeof(float));
        int max_prob_class_index = -1;
        float max_class_prob = 0.0;
        for (int class_index = 0; class_index < class_num; class_index++) {
          float prob =
              bbox_host_ptr[bbox_index * (5 + class_num) + 5 + class_index];
          bbox_det.prob[class_index] = prob;
          if (prob > max_class_prob) {
            max_class_prob = prob;
            max_prob_class_index = class_index;
          }
        }
        bbox_det.max_prob_class_index = max_prob_class_index;
        bbox_det.sort_class = max_prob_class_index;
        bbox_det_vec.push_back(bbox_det);
      }
    }
    post_nms(bbox_det_vec, nms_thresh, class_num);
    for (int i = 0; i < bbox_det_vec.size(); i++)
    free(bbox_det_vec[i].prob);
  
    boxes_num_data[batch_id] = 1;  // bbox_det_vec.size();
  }
  /* ----------------------- NMS END  -----------------------------  */

  /* ----------------------- for  -----------------------------  */
  struct Num {
    float data[85];
  };

  std::vector<Num> tmp_result;

  for (int i = 0; i < result.size(); i += (5 + class_num)) {
    Num ele;

    for (int j = 0; j < (5 + class_num); j++) {
      ele.data[j] = result[i + j];
    }
    tmp_result.push_back(ele);
  }

  std::sort(tmp_result.begin(), tmp_result.end(), [](Num a, Num b) {
    return a.data[0] > b.data[0];
  });
  result.clear();
  for (int i = 0; i < tmp_result.size(); i++)
    for (int j = 0; j < 85; j++) result.push_back(tmp_result[i].data[j]);

  memcpy(boxes_scores_data, result.data(), result.size() * sizeof(float));

  FILE* f = fopen("/zhoukangkang/lite_jetson_yolo_head/cuda.txt", "w");
  for (int i = 0; i < result.size(); i++) fprintf(f, "%f\n", result[i]);
  fclose(f);
  /*-------------------------------------------------------------------------*/

  hipFree(bbox_index_device_ptr);

  for (int i = 0; i < batch * boxes_input.size(); i++) {
    hipFree(ts_info[i].bboxes_dev_ptr);
    hipFree(ts_info[i].bbox_count_device_ptr);
    free(ts_info[i].bboxes_host_ptr);
  }

  return NNADAPTER_NO_ERROR;
}

}  // namespace cuda
}  // namespace nvidia_tensorrt
}  // namespace nnadapter
