#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "driver/nvidia_tensorrt/converter/plugin/swish.h"

namespace nnadapter {
namespace nvidia_tensorrt {

template <typename T>
__device__ inline T MathExp(T a);

template <>
__device__ inline float MathExp<float>(float a) {
  return expf(a);
}

template <typename T>
__global__ void SwishKernel(int num, const T* input, T* output, T beta) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
    output[index] =
        input[index] / (static_cast<T>(1.0) + MathExp<T>(-beta * input[index]));
  }
}

int SwishPlugin::enqueue(int batch_size,
#if TENSORRT_VERSION_GE(8, 0, 0, 0)
                         void const* const* inputs,
                         void* const* outputs,
#else
                         const void* const* inputs,
                         void** outputs,
#endif
                         void* workspace,
                         hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_dims_[0];
  int num = batch_size;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;
  const float* input = static_cast<const float*>(inputs[0]);
  float* output = static_cast<float*>(outputs[0]);
  SwishKernel<float><<<blocks, threads, 0, stream>>>(num, input, output, beta_);
  return 0;
}

REGISTER_NNADAPTER_TENSORRT_PLUGIN(SwishPlugin,
                                   SwishPluginCreator,
                                   "swish_plugin");

}  // namespace nvidia_tensorrt
}  // namespace nnadapter
