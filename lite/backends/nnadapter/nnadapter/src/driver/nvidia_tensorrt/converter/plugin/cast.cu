#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "driver/nvidia_tensorrt/converter/plugin/cast.h"

namespace nnadapter {
namespace nvidia_tensorrt {

CastPluginDynamic::CastPluginDynamic(nvinfer1::DataType intype,
                                     nvinfer1::DataType outtype)
    : intype_(intype), outtype_(outtype) {}

CastPluginDynamic::CastPluginDynamic(const void* serial_data,
                                     size_t serial_length) {
  Deserialize(&serial_data, &serial_length, &intype_);
  Deserialize(&serial_data, &serial_length, &outtype_);
}

nvinfer1::IPluginV2DynamicExt* CastPluginDynamic::clone() const noexcept {
  return new CastPluginDynamic(intype_, outtype_);
}

template <typename Tin, typename Tout, unsigned TPB>
__global__ void cast_kernel(int n, const Tin* input, Tout* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < n) {
    output[idx] = static_cast<Tout>(input[idx]);
  }
}

int32_t CastPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) noexcept {
  auto input_dims = input_desc[0].dims;
  int num = 1;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  const int block_size = 256;
  const int grid_size = (num + block_size - 1) / block_size;

  if (intype_ == nvinfer1::DataType::kINT32 &&
      outtype_ == nvinfer1::DataType::kFLOAT) {  // int32->float32
    const int32_t* input = static_cast<const int32_t*>(inputs[0]);
    float* output = static_cast<float*>(outputs[0]);
    cast_kernel<int32_t,
                float,
                block_size><<<grid_size, block_size, 0, stream>>>(
        num, input, output);
  } else if (intype_ == nvinfer1::DataType::kFLOAT &&
             outtype_ == nvinfer1::DataType::kINT32) {  // float32->int32
    const float* input = static_cast<const float*>(inputs[0]);
    int32_t* output = static_cast<int32_t*>(outputs[0]);
    cast_kernel<float,
                int32_t,
                block_size><<<grid_size, block_size, 0, stream>>>(
        num, input, output);
  } else {
    NNADAPTER_LOG(FATAL) << "cast nvidia_tensorrt doesn't support this cast \n";
  }

  return 0;
}

size_t CastPluginDynamic::getSerializationSize() const noexcept {
  return SerializedSize(outtype_) + SerializedSize(intype_);
}

void CastPluginDynamic::serialize(void* buffer) const noexcept {
  Serialize(&buffer, outtype_);
  Serialize(&buffer, intype_);
}

bool CastPluginDynamic::supportsFormatCombination(
    int32_t pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int32_t nb_inputs,
    int32_t nb_outputs) noexcept {
  NNADAPTER_CHECK_LT(pos, nb_inputs + nb_outputs);
  NNADAPTER_CHECK(in_out);
  return true;
}

REGISTER_NNADAPTER_TENSORRT_PLUGIN(CastPluginDynamic,
                                   CastPluginDynamicCreator,
                                   "cast_plugin_dynamic");

}  // namespace nvidia_tensorrt
}  // namespace nnadapter
