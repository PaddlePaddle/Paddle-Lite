#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <limits>
#include "driver/nvidia_tensorrt/converter/plugin/arg_max.h"

namespace nnadapter {
namespace nvidia_tensorrt {

ArgMaxPluginDynamic::ArgMaxPluginDynamic() {}

ArgMaxPluginDynamic::ArgMaxPluginDynamic(int axis) : _axis(axis) {}

ArgMaxPluginDynamic::ArgMaxPluginDynamic(const void* serial_data,
                                         size_t serial_length) {
  Deserialize(&serial_data, &serial_length, &_axis);
}

nvinfer1::IPluginV2DynamicExt* ArgMaxPluginDynamic::clone() const noexcept {
  return new ArgMaxPluginDynamic(_axis);
}

template <typename InType, typename OutType, unsigned TPB>
__global__ void arg_max_kernel(const InType* input,
                               OutType* output,
                               int pre,
                               int axis_num,
                               int post,
                               const InType init) {
  int height = pre * post;
  int width = axis_num;
  int post_size = post;
  __shared__ int block_pair_idx[TPB];
  __shared__ InType block_pair_val[TPB];

  for (int idx = blockIdx.x; idx < height; idx += gridDim.x) {
    int kv_pair_idx = -1;
    InType kv_pair_val = init;
    int h = idx / post_size;
    int w = idx % post_size;

    for (int k = threadIdx.x; k < width; k += blockDim.x) {
      int index = h * width * post_size + k * post_size + w;
      if (input[index] > kv_pair_val) {
        kv_pair_val = input[index];
        kv_pair_idx = k;
      }
    }
    block_pair_idx[threadIdx.x] = kv_pair_idx;
    block_pair_val[threadIdx.x] = kv_pair_val;
    __syncthreads();

    if (0 == threadIdx.x) {
      int kv_pair_idx_1 = -1;
      InType kv_pair_val_1 = init;
      for (int i = 0; i < TPB; i++) {
        if (block_pair_val[i] > kv_pair_val_1) {
          kv_pair_idx_1 = block_pair_idx[i];
          kv_pair_val_1 = block_pair_val[i];
        }
      }
      output[idx] = static_cast<OutType>(kv_pair_idx_1);
    }
    __syncthreads();
  }
}

int32_t ArgMaxPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) noexcept {
  auto input_dims = input_desc[0].dims;
  auto axis_num = input_dims.d[_axis];
  int pre = 1;
  int post = 1;
  for (int i = 0; i < _axis; i++) {
    pre *= input_dims.d[i];
  }
  for (int i = _axis + 1; i < input_dims.nbDims; i++) {
    post *= input_dims.d[i];
  }
  const int block_size = 128;
  const int grid_size = (axis_num + block_size - 1) / block_size;

  if (input_desc[0].type == nvinfer1::DataType::kFLOAT) {
    const float* input = static_cast<const float*>(inputs[0]);
    int* output = static_cast<int*>(outputs[0]);
    auto init = std::numeric_limits<float>::lowest();
    arg_max_kernel<float,
                   int,
                   block_size><<<grid_size, block_size, 0, stream>>>(
        input, output, pre, axis_num, post, init);
  } else {
    NNADAPTER_LOG(FATAL)
        << "ArgMax only support float-input and int-output for now.";
  }
  return 0;
}

size_t ArgMaxPluginDynamic::getSerializationSize() const noexcept {
  return SerializedSize(_axis);
}

void ArgMaxPluginDynamic::serialize(void* buffer) const noexcept {
  Serialize(&buffer, _axis);
}

REGISTER_NNADAPTER_TENSORRT_PLUGIN(ArgMaxPluginDynamic,
                                   ArgMaxPluginDynamicCreator,
                                   "arg_max_plugin_dynamic");

}  // namespace nvidia_tensorrt
}  // namespace nnadapter
