#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "driver/nvidia_tensorrt/converter/plugin/fill.h"

namespace nnadapter {
namespace nvidia_tensorrt {

FillPluginDynamic::FillPluginDynamic(float value,
                                     bool is_value_tensor,
                                     std::vector<int32_t> shape)
    : value_(value), is_value_tensor_(is_value_tensor), shape_(shape) {}

FillPluginDynamic::FillPluginDynamic(const void* serial_data,
                                     size_t serial_length) {
  Deserialize(&serial_data, &serial_length, &value_);
  Deserialize(&serial_data, &serial_length, &is_value_tensor_);
  Deserialize(&serial_data, &serial_length, &shape_);
}

nvinfer1::IPluginV2DynamicExt* FillPluginDynamic::clone() const TRT_NOEXCEPT {
  return new FillPluginDynamic(value_, is_value_tensor_, shape_);
}

nvinfer1::DimsExprs FillPluginDynamic::getOutputDimensions(
    int32_t output_index,
    const nvinfer1::DimsExprs* inputs,
    int32_t nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  NNADAPTER_CHECK_EQ(output_index, 0);
  NNADAPTER_CHECK(inputs);
  NNADAPTER_CHECK_GE(nb_inputs, 1);
  nvinfer1::DimsExprs outdims;
  outdims.nbDims = shape_.size();
  for (int i = 0; i < shape_.size(); i++) {
    outdims.d[i] = expr_builder.constant(shape_[i]);
  }
  return outdims;
}

template <typename T, unsigned TPB>
__global__ void fill_kernel_value(int n, T* output, T value) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < n) {
    output[idx] = value;
  }
}

template <typename T, unsigned TPB>
__global__ void fill_kernel_value_tensor(int n, T* output, const T* value) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < n) {
    output[idx] = *value;
  }
}

int32_t FillPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    void const* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  auto output_dims = output_desc[0].dims;
  int num = 1;
  for (int i = 0; i < output_dims.nbDims; i++) {
    num *= output_dims.d[i];
  }
  const int block_size = 256;
  const int grid_size = (num + block_size - 1) / block_size;

  float* output = static_cast<float*>(outputs[0]);
  if (is_value_tensor_)
    fill_kernel_value_tensor<float,
                             block_size><<<grid_size, block_size, 0, stream>>>(
        num, output, (static_cast<const float*>(inputs[0])));
  else
    fill_kernel_value<float, block_size><<<grid_size, block_size, 0, stream>>>(
        num, output, value_);

  return 0;
}

size_t FillPluginDynamic::getSerializationSize() const TRT_NOEXCEPT {
  return SerializedSize(value_) + SerializedSize(is_value_tensor_) +
         SerializedSize(shape_);
}

void FillPluginDynamic::serialize(void* buffer) const TRT_NOEXCEPT {
  Serialize(&buffer, value_);
  Serialize(&buffer, is_value_tensor_);
  Serialize(&buffer, shape_);
}

bool FillPluginDynamic::supportsFormatCombination(
    int32_t pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int32_t nb_inputs,
    int32_t nb_outputs) TRT_NOEXCEPT {
  NNADAPTER_CHECK_LT(pos, nb_inputs + nb_outputs);
  NNADAPTER_CHECK(in_out);
  return true;
}

REGISTER_NNADAPTER_TENSORRT_PLUGIN(FillPluginDynamic,
                                   FillPluginDynamicCreator,
                                   "fill_plugin_dynamic");

}  // namespace nvidia_tensorrt
}  // namespace nnadapter
